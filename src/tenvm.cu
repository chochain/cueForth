#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - eForth Vritual Machine implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "tenvm.h"

#if T4_ENABLE_OBJ
__GPU__ void
TensorVM::tprint(DU d) {
    if (IS_OBJ(d)) { fout << d; mmu.mark_free(d); }
    else fout << " " << d;                  /// eForth has a space prefix
}
__GPU__ void TensorVM::add_to_tensor(DU n) {
    DU *d = (DU*)mmu.du2ten(top).data;
    d[ten_off++] = n;
}
///
/// tensor methods
///
__GPU__ void
TensorVM::texp() {
    if (!IS_TEN(top)) {                      /// * scalar
        top = EXP(top); NO_OBJ(top);         /// * mask off object-bit if any
        return;
    }
    Tensor &A = mmu.du2ten(top);
    Tensor &B = mmu.copy(A);
    DU *d = (DU*)B.data;
    for (int i=0; i < B.size; i++, d++) {    ///> TODO: CDP
        *d = EXP(*d);
    }
    PUSH(B);
}
__GPU__ void
TensorVM::tadd(tensor_op op, bool sub) {
    auto drop = [this](Tensor &X) { POP(); mmu.free(X); };
    
    bool s0 = !IS_TEN(top), s1 = !IS_TEN(ss[-1]);
    
    if (s0 && s1) {
        top = sub ? ss.pop() - top : ss.pop() + top;
        return;
    }
    if (s0 || s1) {                            ///> tensor +- scalar
        Tensor &A = mmu.du2ten(s0 ? ss[-1] : top);
        DU     n  = s0 ? top : ss[-1];
        Tensor &C = mmu.tensor(A.H(), A.W());
        Tensor::add(A, n, C, sub);
        if (s1 && sub) C.scale(-1.0);          /// negate
        if (op==DROP) { drop(A); POP(); }      /// TODO: in-place
        PUSH(C);
        return;
    }
    Tensor &A = mmu.du2ten(ss[-1]);            ///> tensor +- tensor
    Tensor &B = mmu.du2ten(top);
    U16 h = A.H(), w = A.W();
    if (h == B.H() && w == B.W()) {
        Tensor &C = mmu.tensor(h, w);
        Tensor::add(A, B, C, sub);
        if (op==DROP) { drop(B); drop(A); }    /// TODO: in-place 
        PUSH(C);
    }
}
/**
  TODO: Matrix product of two Tensors.
  The behavior depends on the dimensionality of the Tensors as follows:
  - DONE: If both arguments are 2-dimensional, the matrix-matrix product is returned.
  - DONE: If both Tensors are 1-dimensional, the dot product (scalar) is returned.
  - TODO: If the first argument is 2-dimensional and the second argument is 1-dimensional,
    the matrix-vector product is returned.
  - TODO: If the first argument is 1-dimensional and the second argument is 2-dimensional,
    a 1 is prepended to its dimension for the purpose of the matrix multiply.
    After the matrix multiply, the prepended dimension is removed.
  - TODO: If both arguments are at least 1-dimensional and at least one argument is
    N-dimensional (where N > 2), then a batched matrix multiply is returned.  If the first
    argument is 1-dimensional, a 1 is prepended to its dimension for the purpose of the
    batched matrix multiply and removed after.  If the second argument is 1-dimensional, a
    1 is appended to its dimension for the purpose of the batched matrix multiple and removed after.
    The non-matrix (i.e. batch) dimensions are broadcasted (and thus
    must be broadcastable).  For example, if tensor1 is a (j x 1 x n x m) Tensor
    and tensor2 is a (k x m x p) Tensor, the returned tensor will be an (j x k x n x p) Tensor.
*/
__GPU__ void
TensorVM::tmul(tensor_op op) {                        ///< tensor multiplication
    auto drop = [this](Tensor &X) { POP(); mmu.free(X); };
    
    bool s0 = !IS_TEN(top), s1 = !IS_TEN(ss[-1]);     /// * scalar check
    if (s0 && s1) { top *= ss.pop(); return; }        /// * scalar * scalar

    Tensor &A = mmu.du2ten(s1 ? top : ss[-1]);
    if (s0 || s1) {                                   /// * tensor * scalar
        Tensor &C = mmu.copy(A);                      /// * hard copy A tensor
        DU     k  = s0 ? top : ss[-1];
        VLOG2("T%d=%p * %f => A'=%p\n", A.rank, &A, k, &C);
        mmu.ten2du(C.scale(k));                       /// * resultant tensor on TOS
        if (op==DROP) { drop(A); POP(); }             /// TODO: in-place
        PUSH(C);
        return;
    }
    Tensor &B = mmu.du2ten(top);                      /// tensor * tensor
    U16 m = A.H(), ka = A.W(), kb = B.H(), n = B.W();
    VLOG2("A[%d,%d]=%p x B[%d,%d]=%p ", m, ka, &A, kb, n, &B);
    if (A.rank==1 && B.rank==1 && A.size==B.size) {   /// * vector x vector
        DU d = A.dot(B);                              /// * inner product
        if (op==DROP) { drop(B); drop(A); }
        PUSH(d);                                      /// * dot product on TOS
        VLOG2(" => %f\n", top);
    }
    else if (B.rank==1) {                             /// * tensor x vector
        Tensor &C = mmu.tensor(ka);
        Tensor::mm(A, B, C);
        if (op==DROP) { drop(B); drop(A); }           /// TODO: in-place
        PUSH(C);                                      /// * resultant tensor on TOS
        VLOG2("=> C[%d]=%p\n", C.H(), &C);
    }
    else if (ka == kb) {                              /// * tensor x tensor
        Tensor &C = mmu.tensor(m, n);
        Tensor::mm(A, B, C);
        if (op==DROP) { drop(B); drop(A); }           /// TODO: in-place
        PUSH(C);                                      /// * resultant tensor on TOS
        VLOG2("=> C[%d,%d]=%p\n", C.H(), C.W(), &C);
    }
    else ERROR("dim?");
}
__GPU__ void
TensorVM::tdiv(tensor_op op) {                        ///< tensor division
    auto drop = [this](Tensor &X) { POP(); mmu.free(X); };
        
    bool s0 = !IS_TEN(top), s1 = !IS_TEN(ss[-1]);
    if (s0 && s1) {
        top = ss.pop() / top;                         /// * scalar / scalar
        NO_OBJ(top);
        return;
    }
    else if (s0) {                                     /// * tensor / scaler
        Tensor &A = mmu.du2ten(ss[-1]);
        Tensor &C = mmu.copy(A);                       /// * hard copy A tensor
        VLOG2("A[%d,%d]=%p / %f => A'=%p\n", A.H(), A.W(), &A, top, &C);
        if (op==DROP) drop(A);                         /// TODO: in-place
        top = mmu.ten2du(C.scale(1.0/top));            /// * resultant tensor on TOS
        return;
    }
    /// tensor / tensor i.e. C = A * inv(B)
    Tensor &A  = mmu.du2ten(ss[-1]);
    Tensor &B  = mmu.du2ten(top);
    U16 m = A.H(), ka = A.W(), kb = B.H(), n = B.W();
    if (kb != n || ka != kb) { ERROR("dim?"); return; }/// * B square?
        
    tinv();                                            /// * top = inverse(B)
    Tensor &Bi = mmu.du2ten(POP());
    Tensor &C  = mmu.tensor(m, n);
    VLOG2("A[%d,%d]=%p / B[%d,%d]=%p => C=%p\n", m, ka, &A, kb, n, &B, &C);
    Tensor::mm(A, Bi, C);
    
    /// free matrices if desired
    mmu.free(Bi);                                      /// * drop Bi
    if (op==DROP) { drop(B); drop(A); }                /// TODO: in-place
    
    PUSH(C);                                           /// * put result on TOS
}
///
/// matrix inversion GauseJordan (with Pivot)
///
__GPU__ void
TensorVM::tinv() {
    if (!IS_TEN(top)) { ERROR("tensor?"); return; }
    Tensor &A = mmu.du2ten(top);
    Tensor &I = mmu.tensor(A.H(), A.W()).identity();
    Tensor &C = mmu.copy(A);
    Tensor::inverse(C, I);
    mmu.free(C);
    PUSH(I);
}
///
/// LU conversion (no Pivot)
///
__GPU__ void
TensorVM::tlu() {
    if (!IS_TEN(top)) { ERROR("tensor?"); return; }
    Tensor &A  = mmu.du2ten(top);
    Tensor &LU = mmu.copy(A);             /// * hardcopy original matrix
    Tensor::lu(LU);                       /// * decompose A to LU
    PUSH(LU);
}
///
/// matrix determinant
///
__GPU__ void
TensorVM::tdet() {
    if (!IS_TEN(top)) { ERROR("tensor?"); return; }
    Tensor &A  = mmu.du2ten(top);
    Tensor &LU = mmu.copy(A);             /// * hardcopy original matrix
    Tensor &P  = mmu.tensor(A.H());       /// * dummy
    Tensor::plu(LU, P);                   /// * decompose A to LU
    mmu.free(P);
    PUSH(LU.det());                       /// * return determinant on TOS
}
__GPU__ void
TensorVM::ttrans() {
    if (!IS_TEN(top)) { ERROR("tensor?"); return; }
    Tensor &A = mmu.du2ten(top);
    U16 h = A.H(), w = A.W();
    Tensor &B = mmu.tensor(w, h);
    VLOG2("A[%d,%d]=%p => B[%d,%d]=%p", h, w, &A, B.H(), B.W(), &B);
    Tensor::transpose(A, B);
    PUSH(B);
}
__GPU__ void
TensorVM::solve() {
    if (!IS_TEN(ss[-1]) || !IS_TEN(top)) { ERROR("tensor?"); return; }
    Tensor &B = mmu.du2ten(ss[-1]);      /// B vector
    Tensor &A = mmu.du2ten(top);         /// A linear equations
    U16 m = A.H(), k = A.W(), n = B.W();
    if (m==k && B.rank==1 && k==B.H()) {
        tinv();                          /// * inverse (i.e. A^-1)
        Tensor &Ai = mmu.du2ten(POP());  /// * pop off A^-1
        Tensor &X  = mmu.tensor(k);      /// resultant vector
        Tensor::mm(Ai, B, X);            /// X = A^-1 x B
        PUSH(X);                         /// * put resultant on TOS
        mmu.free(Ai);                    /// * release inverse matrix
    }
    else ERROR("B A or dim?");
}
__GPU__ void
TensorVM::gemm() {                       ///< blas GEMM
    Tensor &C = mmu.du2ten(top);
    Tensor &B = mmu.du2ten(ss[-1]);
    Tensor &A = mmu.du2ten(ss[-2]);
    DU     b  = ss[-3];
    DU     a  = ss[-4];
    U16 m = A.H(), k = A.W(), n = B.W();
    if (k == B.H() && m == C.H() && n == C.W()) {
        Tensor &D = mmu.copy(C);         /// * hard copy C tensor
        Tensor::gemm(A, B, D, a, b);
        PUSH(D);
    }
    else ERROR("dim?");
}
///
/// Tensor specific dictionary constructor
///
__GPU__ void
TensorVM::init_t() {
    const Code prim[] = {       /// singleton, build once only
    ///@defgroup Tensor creation ops
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("vector",                        ///< allocate a vector
        IU sz = POPi;
        PUSH(mmu.tensor(sz))),
    CODE("matrix",                       ///< allocate a matrix
        IU w = POPi; IU h = POPi;
        PUSH(mmu.tensor(h, w))),
    CODE("tensor",                       ///< allocate a NHWC tensor
        IU c = POPi; IU w = POPi; IU h = POPi; IU n = POPi;
        PUSH(mmu.tensor(n, h, w, c))),
    CODE("vector{",                      ///< create a vector with literals
        IU sz = POPi;
        PUSH(mmu.tensor(sz));
        ten_off = 0; ten_lvl = 1),
    CODE("matrix{",                      ///< create a matrix with literals
        IU w = POPi; IU h = POPi;
        PUSH(mmu.tensor(h, w));
        ten_off = 0; ten_lvl = 1),
    CODE("copy",    PUSH(mmu.copy(top))),
    ///@}
    ///@defgroup Tensor shape ops
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("flatten",                      ///< reshape as a vector (1-D array)
        Tensor &t = mmu.du2ten(top);
        t.reshape(t.size)),
    CODE("reshape2",                     ///< reshape as matrix(h,w)
        IU w = POPi; IU h = POPi;
        mmu.du2ten(top).reshape(h, w)),
    CODE("reshape4",                     ///< reshape as Tensor(NHWC)
        IU c = POPi; IU w = POPi; IU h = POPi; IU n = POPi;
        mmu.du2ten(top).reshape(n, h, w, c)),
    ///@}
    ///@defgroup Tensor fill ops
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("={",                          ///< (n -- ) or ( -- )
         ten_off = IS_TEN(top) ? 0 : POPi;
         ten_lvl = IS_TEN(top) ? 1 : 0),
    CODE("zeros", if (IS_TEN(top)) mmu.du2ten(top).fill(0)),
    CODE("ones",  if (IS_TEN(top)) mmu.du2ten(top).fill(1)),
    CODE("full",  if (IS_TEN(ss[-1])) { DU d = POP(); mmu.du2ten(top).fill(d); }),
    CODE("eye",   if (IS_TEN(top)) mmu.du2ten(top).identity()),
    CODE("rand",  top = mmu.rand(top, UNIFORM)),  ///< uniform randomize a tensor or number
    CODE("randn", top = mmu.rand(top, NORMAL)),   ///< normal dist. randomize a tensor
    ///@}
    ///@defgrup Tensor slice and dice
    ///@{
    CODE("sum",
        if (IS_TEN(top)) {
            DU d =  mmu.du2ten(top).sum();
            PUSH(d);
        }),
    CODE("{",   if (IS_TEN(top) && ten_lvl > 0) ++ten_lvl),
    CODE("}",   if (IS_TEN(top) && ten_lvl > 0) --ten_lvl),
    CODE("slice",
         IU y1 = POPi; IU y0 = POPi; IU x1 = POPi; IU x0 = POPi;
         if (IS_TEN(top)) {
             Tensor &t0 = mmu.du2ten(top);
             Tensor &t1 = mmu.slice(t0, x0, x1, y0, y1);
             PUSH(t1);
         }),
    ///@}
    ///@defgroup Tensor matrix ops (destructive, as in Forth)
    ///@{
    CODE("+=",        tadd(DROP)),
    CODE("*=",        tmul(DROP)),
    CODE("-=",        tadd(DROP, true)),
    CODE("/=",        tdiv(DROP)),
    ///@defgroup Tensor matrix ops
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("exp",       texp()),     ///< (A -- A A')    matrix exponential
    CODE("inverse",   tinv()),     ///< (A -- A Ai')   matrix inversion (GaussJordan)
    CODE("det",       tdet()),     ///< (A -- A d)     matrix determinant
    CODE("lu",        tlu()),      ///< (A -- A A')    LU decomposition
    CODE("luinv",                  ///< (A -- A A')    inverse an LU matrix
         if (!IS_TEN(top)) return;
         Tensor &t0 = mmu.du2ten(top);
         Tensor::inverse(t0)),
    CODE("upper", if (!IS_TEN(top)) return;
         Tensor &t0 = mmu.du2ten(top);
         Tensor &t1 = mmu.copy(t0);
         t1.triu();
         PUSH(t1)),
    CODE("lower", if (!IS_TEN(top)) return;
         Tensor &t0 = mmu.du2ten(top);
         Tensor &t1 = mmu.copy(t0);
         t1.tril();
         PUSH(t1)),
    CODE("transpose", ttrans()),   ///< (A -- A At)    matrix transpose
    CODE("matmul",    tmul(KEEP)), ///< (A B -- A B C) matrix multiplication
    CODE("solve",     solve()),    ///< (B A -- B A X) solve linear equations AX = B
    CODE("gemm",      gemm()),     ///< (a b A B C -- a b A B C') GEMM (C updated)
    ///@}
    };
    const Code old[] = {
    ///@defgroup redefined tensor ops
    ///@{
    CODE(".",   tprint(POP())),
    CODE("+",   tadd(KEEP)),
    CODE("*",   tmul(KEEP)),
    CODE("-",   tadd(KEEP, true)),
    CODE("/",   tdiv(KEEP)),
    ///@}
    CODE("boot", mmu.clear(FIND("gemm") + 1))
    };
    ForthVM::init_f();

    mmu.append(prim, sizeof(prim)/sizeof(Code));    /// * append tensor words
    mmu.merge(old,  sizeof(old)/sizeof(Code));      /// * update existed words
    mmu.status();

    status = VM_RUN;
};
///
/// override with tensor handler
///
__GPU__ int
TensorVM::number(char *str) {
    char *p;
    DU n = (STRCHR(idiom, '.'))
        ? STRTOF(idiom, &p)
        : STRTOL(idiom, &p, radix);
    if (*p != '\0') return 0;
    NO_OBJ(n);                           /// * mask out object bit
    if (compile) {                       /// * add literal when in compile mode
        VLOG2("%f\n", n);
        add_w(DOLIT);                    ///> dovar (+parameter field)
        add_du(n);                       ///> store literal
    }
    else if (ten_lvl > 0) {              /// * append literal into tensor storage
        VLOG2("T[%d]=%f\n", ten_off, n);
        add_to_tensor(n);
    }
    else {                               ///> or, add value onto data stack
        VLOG2("ss.push(%08x)\n", *(U32*)&n);
        PUSH(n);
    }
}
#endif  // T4_ENABLE_OBJ
//=======================================================================================
