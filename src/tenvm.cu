#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - eForth Vritual Machine implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "tenvm.h"

#if T4_ENABLE_OBJ
__GPU__ void
TensorVM::tprint(DU d) {
    if (IS_OBJ(d)) { fout << d; mmu.mark_free(d); }
    else fout << " " << d;                  /// eForth has a space prefix
}
__GPU__ void TensorVM::add_to_tensor(DU n) {
    DU *d = (DU*)mmu.du2ten(top).data;
    d[ten_off++] = n;
}
///
/// tensor methods
///
__GPU__ void
TensorVM::ssop(t4_mat_op op) {
    switch (op) {
    case ADD: top += ss.pop();      break;
    case SUB: top = ss.pop() - top; break;
    case MUL: top *= ss.pop();      break;
    case DIV: top = DIV(ss.pop(), top); SCALAR(top); break;
    }
    VLOG2(" => %f\n", top);
}
/**
  TODO: Matrix product of two Tensors.
  The behavior depends on the dimensionality of the Tensors as follows:
  - DONE: If both arguments are 2-dimensional, the matrix-matrix product is returned.
  - DONE: If both Tensors are 1-dimensional, the dot product (scalar) is returned.
  - TODO: If the first argument is 2-dimensional and the second argument is 1-dimensional,
    the matrix-vector product is returned.
  - TODO: If the first argument is 1-dimensional and the second argument is 2-dimensional,
    a 1 is prepended to its dimension for the purpose of the matrix multiply.
    After the matrix multiply, the prepended dimension is removed.
  - TODO: If both arguments are at least 1-dimensional and at least one argument is
    N-dimensional (where N > 2), then a batched matrix multiply is returned.  If the first
    argument is 1-dimensional, a 1 is prepended to its dimension for the purpose of the
    batched matrix multiply and removed after.  If the second argument is 1-dimensional, a
    1 is appended to its dimension for the purpose of the batched matrix multiple and removed after.
    The non-matrix (i.e. batch) dimensions are broadcasted (and thus
    must be broadcastable).  For example, if tensor1 is a (j x 1 x n x m) Tensor
    and tensor2 is a (k x m x p) Tensor, the returned tensor will be an (j x k x n x p) Tensor.
*/
__GPU__ void
TensorVM::tsop(t4_mat_op op, t4_drop_opt x, bool swap) {
    auto drop = [this](Tensor &A) { POP(); mmu.free(A); };
    
    Tensor &A = mmu.du2ten(swap ? top : ss[-1]);
    DU     n  = swap ? ss[-1] : top;
    Tensor &C = mmu.tensor(A.H(), A.W());
    if (swap && (op==DIV || op==SUB)) {     /// * op(scaler, tensor)
        Tensor &B = mmu.tensor(A.size);     /// * working tensor
        B.map(FILL, n);                     /// * broadcast
        Tensor::mat(op, B, A, C);           /// * Hadamard ops
        mmu.free(B);                        /// * free working tensor
    }
    else Tensor::mat(op, A, n, C);          /// * broadcast_op(tensor, scalar)
    
    if (x==DROP) { drop(A); POP(); }        /// TODO: in-place
    PUSH(C);
    VLOG2("=> C[%d,%d]=%p\n", C.H(), C.W(), &C);
}
__GPU__ void
TensorVM::tmat(t4_mat_op op, t4_drop_opt x) {
    bool s0 = !IS_TEN(top), s1 = !IS_TEN(ss[-1]); /// * scalar flags
    if (s0 && s1) return ssop(op);          ///> op(scalar, scalar)
    if (s0 || s1) return tsop(op, x, s1);   ///> op(tensor, scalar)
    ///
    /// op(tensor, tensor)
    ///
    auto free = [this](Tensor &A, Tensor &B) {
        POP(); mmu.free(B); POP(); mmu.free(A);
    };
    Tensor &A = mmu.du2ten(ss[-1]);
    Tensor &B = mmu.du2ten(top);
    U16 m = A.H(), n = A.W();               /// * get matrix dimensions
    if (m == B.H() && n == B.W()) {         ///> op(tensor,tensor) (Hadamard)
        Tensor &C = mmu.tensor(m, n);
        Tensor::mat(op, A, B, C);
        if (x==DROP) free(A, B);            /// TODO: in-place 
        PUSH(C);
        VLOG2("=> C[%d,%d]=%p\n", C.H(), C.W(), &C);
        return;
    }
    if (B.rank != 1 || op != MUL) { ERROR("mul?"); return; }
    ///
    /// broadcast_op(tensor, tensor)
    ///
    if (A.rank==1 && A.size==B.size) {      ///> dot(vector, vector)
        DU d = A.dot(B);                    /// * inner product
        if (x==DROP) free(A, B);
        PUSH(d);                            /// * dot product on TOS
        VLOG2(" => %f\n", top);
    }
    else if (n==B.size) {                   ///> inner(tensor, vector)
        Tensor &C = mmu.tensor(n);
        Tensor::mm(A, B, C);
        if (x==DROP) free(A, B);            /// TODO: in-place
        PUSH(C);                            /// * resultant tensor on TOS
        VLOG2("=> C[%d]=%p\n", C.H(), &C);
    }
    else ERROR("dim?");
}
__GPU__ void
TensorVM::tmul(t4_drop_opt x) {                       ///< tensor multiplication
    auto drop = [this](Tensor &X) { POP(); mmu.free(X); };
    
    bool s0 = !IS_TEN(top), s1 = !IS_TEN(ss[-1]);     /// * scalar check
    if (s0 || s1) return;                             /// * matrix-matrix only

    Tensor &A = mmu.du2ten(ss[-1]);                   /// tensor @ tensor
    Tensor &B = mmu.du2ten(top);
    U16 m = A.H(), ka = A.W(), kb = B.H(), n = B.W();
    VLOG2("A[%d,%d]=%p @ B[%d,%d]=%p ", m, ka, &A, kb, n, &B);
    if (A.rank==2 && B.rank==2 && ka == kb) {         /// * tensor x tensor
        Tensor &C = mmu.tensor(m, n);
        Tensor::mm(A, B, C);
        if (x==DROP) { drop(B); drop(A); }            /// TODO: in-place
        PUSH(C);                                      /// * resultant tensor on TOS
        VLOG2("=> C[%d,%d]=%p\n", C.H(), C.W(), &C);
    }
    else ERROR("dim?");
}
__GPU__ void
TensorVM::tdiv(t4_drop_opt x) {                       ///< tensor division
    auto drop = [this](Tensor &X) { POP(); mmu.free(X); };
        
    bool s0 = !IS_TEN(top), s1 = !IS_TEN(ss[-1]);
    if (s0 || s1) return;
    
    /// tensor / tensor i.e. C = A * inv(B)
    Tensor &A  = mmu.du2ten(ss[-1]);
    Tensor &B  = mmu.du2ten(top);
    U16 m = A.H(), ka = A.W(), kb = B.H(), n = B.W();
    if (kb != n || ka != kb) { ERROR("dim?"); return; }/// * B square?
        
    tinv();                                            /// * top = inverse(B)
    Tensor &Bi = mmu.du2ten(POP());
    Tensor &C  = mmu.tensor(m, n);
    VLOG2("A[%d,%d]=%p / B[%d,%d]=%p => C=%p\n", m, ka, &A, kb, n, &B, &C);
    Tensor::mm(A, Bi, C);
    
    /// free matrices if desired
    mmu.free(Bi);                                      /// * drop Bi
    if (x==DROP) { drop(B); drop(A); }                 /// TODO: in-place
    
    PUSH(C);                                           /// * put result on TOS
}
///
/// matrix inversion GauseJordan (with Pivot)
///
__GPU__ void
TensorVM::tinv() {
    if (!IS_TEN(top)) { ERROR("tensor?"); return; }
    Tensor &A = mmu.du2ten(top);
    Tensor &I = mmu.tensor(A.H(), A.W()).identity();
    Tensor &C = mmu.copy(A);
    Tensor::inverse(C, I);
    mmu.free(C);
    PUSH(I);
}
///
/// LU conversion (no Pivot)
///
__GPU__ void
TensorVM::tlu() {
    if (!IS_TEN(top)) { ERROR("tensor?"); return; }
    Tensor &A  = mmu.du2ten(top);
    Tensor &LU = mmu.copy(A);             /// * hardcopy original matrix
    Tensor::lu(LU);                       /// * decompose A to LU
    PUSH(LU);
}
///
/// matrix determinant
///
__GPU__ void
TensorVM::tdet() {
    if (!IS_TEN(top)) { ERROR("tensor?"); return; }
    Tensor &A  = mmu.du2ten(top);
    Tensor &LU = mmu.copy(A);             /// * hardcopy original matrix
    Tensor &P  = mmu.tensor(A.H());       /// * dummy
    Tensor::plu(LU, P);                   /// * decompose A to LU
    mmu.free(P);
    PUSH(LU.det());                       /// * return determinant on TOS
}
__GPU__ void
TensorVM::ttrans() {
    if (!IS_TEN(top)) { ERROR("tensor?"); return; }
    Tensor &A = mmu.du2ten(top);
    U16 h = A.H(), w = A.W();
    Tensor &B = mmu.tensor(w, h);
    VLOG2("A[%d,%d]=%p => B[%d,%d]=%p", h, w, &A, B.H(), B.W(), &B);
    Tensor::transpose(A, B);
    PUSH(B);
}
__GPU__ void
TensorVM::solve() {
    if (!IS_TEN(ss[-1]) || !IS_TEN(top)) { ERROR("tensor?"); return; }
    Tensor &B = mmu.du2ten(ss[-1]);      /// B vector
    Tensor &A = mmu.du2ten(top);         /// A linear equations
    U16 m = A.H(), k = A.W(), n = B.W();
    if (m==k && B.rank==1 && k==B.H()) {
        tinv();                          /// * inverse (i.e. A^-1)
        Tensor &Ai = mmu.du2ten(POP());  /// * pop off A^-1
        Tensor &X  = mmu.tensor(k);      /// resultant vector
        Tensor::mm(Ai, B, X);            /// X = A^-1 x B
        PUSH(X);                         /// * put resultant on TOS
        mmu.free(Ai);                    /// * release inverse matrix
    }
    else ERROR("B A or dim?");
}
__GPU__ void
TensorVM::gemm() {                       ///< blas GEMM
    Tensor &C = mmu.du2ten(top);
    Tensor &B = mmu.du2ten(ss[-1]);
    Tensor &A = mmu.du2ten(ss[-2]);
    DU     b  = ss[-3];
    DU     a  = ss[-4];
    U16 m = A.H(), k = A.W(), n = B.W();
    if (k == B.H() && m == C.H() && n == C.W()) {
        Tensor &D = mmu.copy(C);         /// * hard copy C tensor
        Tensor::gemm(A, B, D, a, b);
        PUSH(D);
    }
    else ERROR("dim?");
}
///
/// Tensor specific dictionary constructor
///
__GPU__ void
TensorVM::init() {
    const Code prim[] = {       /// singleton, build once only
    ///@defgroup Tensor creation ops
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("vector",                        ///< allocate a vector
        IU sz = POPi;
        PUSH(mmu.tensor(sz))),
    CODE("matrix",                       ///< allocate a matrix
        IU w = POPi; IU h = POPi;
        PUSH(mmu.tensor(h, w))),
    CODE("tensor",                       ///< allocate a NHWC tensor
        IU c = POPi; IU w = POPi; IU h = POPi; IU n = POPi;
        PUSH(mmu.tensor(n, h, w, c))),
    CODE("vector{",                      ///< create a vector with literals
        IU sz = POPi;
        PUSH(mmu.tensor(sz));
        ten_off = 0; ten_lvl = 1),
    CODE("matrix{",                      ///< create a matrix with literals
        IU w = POPi; IU h = POPi;
        PUSH(mmu.tensor(h, w));
        ten_off = 0; ten_lvl = 1),
    CODE("copy",    PUSH(mmu.copy(top))),
    ///@}
    ///@defgroup Tensor shape ops
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("flatten",                      ///< reshape as a vector (1-D array)
        Tensor &t = mmu.du2ten(top);
        t.reshape(t.size)),
    CODE("reshape2",                     ///< reshape as matrix(h,w)
        IU w = POPi; IU h = POPi;
        mmu.du2ten(top).reshape(h, w)),
    CODE("reshape4",                     ///< reshape as Tensor(NHWC)
        IU c = POPi; IU w = POPi; IU h = POPi; IU n = POPi;
        mmu.du2ten(top).reshape(n, h, w, c)),
    ///@}
    ///@defgroup Tensor fill ops
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("={",                          ///< (n -- ) or ( -- )
         ten_off = IS_TEN(top) ? 0 : POPi;
         ten_lvl = IS_TEN(top) ? 1 : 0),
    CODE("zeros", if (IS_TEN(top)) mmu.du2ten(top).map(FILL, DU0)),
    CODE("ones",  if (IS_TEN(top)) mmu.du2ten(top).map(FILL, DU1)),
    CODE("full",  if (!IS_TEN(ss[-1])) return;
         DU d = POP(); mmu.du2ten(top).map(FILL, d)),
    CODE("eye",   if (IS_TEN(top)) mmu.du2ten(top).identity()),
    CODE("rand",  top = mmu.rand(top, UNIFORM)),  ///< uniform randomize a tensor or number
    CODE("randn", top = mmu.rand(top, NORMAL)),   ///< normal dist. randomize a tensor
    ///@}
    ///@defgrup Tensor slice and dice
    ///@{
    CODE("sum",
        if (IS_TEN(top)) {
            DU d =  mmu.du2ten(top).sum();
            PUSH(d);
        }),
    CODE("{",   if (IS_TEN(top) && ten_lvl > 0) ++ten_lvl),
    CODE("}",   if (IS_TEN(top) && ten_lvl > 0) --ten_lvl),
    CODE("slice",
         IU y1 = POPi; IU y0 = POPi; IU x1 = POPi; IU x0 = POPi;
         if (IS_TEN(top)) {
             Tensor &t0 = mmu.du2ten(top);
             Tensor &t1 = mmu.slice(t0, x0, x1, y0, y1);
             PUSH(t1);
         }),
    ///@}
    ///@defgroup Tensor matrix ops (destructive, as in Forth)
    ///@{
    CODE("exp",                    ///< (A -- A') 
         if (!IS_OBJ(top)) {       /// * scalar
             top = EXP(top);
             SCALAR(top);          /// * mask off object-bit if any
         }
         else mmu.du2ten(top).map(EXP)),
    CODE("tanh",                   ///< (A -- A') 
         if (!IS_OBJ(top)) {       /// * scalar
             top = tanh(top);
             SCALAR(top);          /// * mask off object-bit if any
         }
         else mmu.du2ten(top).map(TANH)),
    CODE("relu", 
         if (IS_TEN(top)) mmu.du2ten(top).map(RELU);
         else top = top > DU0 ? top : DU0),
    CODE("+=",        tmat(ADD, DROP)),
    CODE("-=",        tmat(SUB, DROP)),
    CODE("*=",        tmat(MUL, DROP)),
    CODE("/=",        tmat(DIV, DROP)),
    CODE("@=",        tmul(DROP)),
    ///@defgroup Tensor matrix ops
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("inverse",   tinv()),     ///< (A -- A Ai')   matrix inversion (GaussJordan)
    CODE("det",       tdet()),     ///< (A -- A d)     matrix determinant
    CODE("lu",        tlu()),      ///< (A -- A A')    LU decomposition
    CODE("luinv",                  ///< (A -- A A')    inverse an LU matrix
         if (!IS_TEN(top)) return;
         Tensor &t0 = mmu.du2ten(top);
         Tensor::inverse(t0)),
    CODE("upper", if (!IS_TEN(top)) return;
         Tensor &t0 = mmu.du2ten(top);
         Tensor &t1 = mmu.copy(t0);
         t1.triu();
         PUSH(t1)),
    CODE("lower", if (!IS_TEN(top)) return;
         Tensor &t0 = mmu.du2ten(top);
         Tensor &t1 = mmu.copy(t0);
         t1.tril();
         PUSH(t1)),
    CODE("transpose", ttrans()),   ///< (A -- A At)    matrix transpose
    CODE("matmul",    tmul(KEEP)), ///< (A B -- A B C) matrix multiplication
    CODE("matdiv",    tdiv(KEEP)), ///< (A B -- A B C) matrix division
    CODE("solve",     solve()),    ///< (B A -- B A X) solve linear equations AX = B
    CODE("gemm",      gemm()),     ///< (a b A B C -- a b A B C') GEMM (C updated)
    ///@}
    };
    const Code over[] = {          /// extended (overload) words
    ///@defgroup redefined tensor ops
    ///@{
    CODE(".",   tprint(POP())),
    CODE("+",   tmat(ADD, KEEP)),
    CODE("-",   tmat(SUB, KEEP)),
    CODE("*",   tmat(MUL, KEEP)),
    CODE("/",   tmat(DIV, KEEP)),
    CODE("@",
         if (IS_TEN(top)) tmul(KEEP);
         else {
             IU w = POPi; PUSH(mmu.rd((IU)w));
         }),
    CODE("abs",
         if (!IS_OBJ(top)) top = ABS(top);
         else mmu.du2ten(top).map(ABS)),
    CODE("negate",
         if (!IS_OBJ(top)) top *= -DU1;
         else mmu.du2ten(top).map(SCALE, -DU1)),
    ///@}
    CODE("boot", mmu.clear(FIND("gemm") + 1))
    };
    ForthVM::init();

    mmu.append(prim, sizeof(prim)/sizeof(Code)); /// * append tensor words
    mmu.merge(over,  sizeof(over)/sizeof(Code)); /// * overload existed words
    mmu.status();
};
///
/// override with tensor handler
///
__GPU__ int
TensorVM::number(char *str) {
    char *p;
    DU n = (STRCHR(idiom, '.'))
        ? STRTOF(idiom, &p)
        : STRTOL(idiom, &p, radix);
    if (*p != '\0') return 0;
    SCALAR(n);                           /// * mask out object bit
    if (compile) {                       /// * add literal when in compile mode
        VLOG2("%f\n", n);
        add_w(DOLIT);                    ///> dovar (+parameter field)
        add_du(n);                       ///> store literal
    }
    else if (ten_lvl > 0) {              /// * append literal into tensor storage
        VLOG2("T[%d]=%f\n", ten_off, n);
        add_to_tensor(n);
    }
    else {                               ///> or, add value onto data stack
        VLOG2("ss.push(%08x)\n", *(U32*)&n);
        PUSH(n);
    }
}
#endif  // T4_ENABLE_OBJ
//=======================================================================================
