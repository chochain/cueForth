/** -*- c++ -*-
 * @File
 * @brief - Neural Network Model Backward Propagation implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "model.h"

#if T4_ENABLE_OBJ
__GPU__ Model&
Model::backprop(Tensor &tgt) {
    printf("here\n");
    Tensor &nx = (*this)[numel - 1];
    if (!nx.is_same_shape(tgt)) {
        ERROR("Model#backprop target dim?\n");
        return *this;
    }
    ///
    /// cascade execution layer by layer backward
    ///
    nx -= tgt;
    for (U16 i = numel - 2; i > 0; i--) {
        Tensor &in = (*this)[i], &out = (*this)[i + 1];
        printf("%2d> %s [%d,%d,%d] p=%d <=",
               i, d_nname(in.grad_fn), in.H(), in.W(), in.C(), in.parm); 
        _bstep(in, out);
        printf("\n");
    }
    return *this;
}
/// ========================================================================
/// private methods 
///
#define TILE3    (T4_WARP_SZ - 3 + 1)      /** 14 */
#define TILE5    (T4_WARP_SZ - 5 + 1)      /** 12 */

__GPU__ void
Model::_bstep(Tensor &in, Tensor &out) {
    DU   *da = in.data, *dc = out.data;              ///< input, output data
    int  H = out.H(), W = out.W(), C = out.C();      ///< output HWC
    dim3 blk(T4_WARP_SZ, T4_WARP_SZ, C), grd(        ///< GPU warp size setup
        (W + blk.x - 1) / blk.x,
        (H + blk.y - 1) / blk.y
    );
    auto dump = [](DU *v, int H, int W, int C) {
        for (int k = 0; k < C; k++) {
            printf("\nC=%d ---", k);
            for (int i = 0; i < H; i++) {
                printf("\n");
                for (int j = 0; j < W; j++) {
                    DU x = v[k + (j + i * W) * C];
                    printf(x < DU0 ? "%.2f" : " %.2f", x);
                }
            }
        }
        printf("\n");
    };
    ///
    /// layer function dispatcher
    ///
    printf(" out'[%d,%d,%d]", H, W, C);
    t4_layer fn = in.grad_fn;                 ///< layer function
    switch(fn) {
    case L_CONV:   {
        Tensor &f = *in.grad[0];              ///< filter tensor
        Tensor &b = *in.grad[1];              ///< bias tensor
        printf(" f[%d][%d,%d,%d,%d], b[%d]",
               f.parm, f.N(), f.H(), f.W(), f.C(), b.numel);
    } break;
    case L_LINEAR: {                          ///< out = w @ in + b
        Tensor &dw = *in.grad[2];             ///< weight tensor
        Tensor &db = *in.grad[3];             ///< bias tensor
        int M = dw.H(), N = dw.W();           ///< fully connected dimensions
        printf(" out'[%d] @ dw[%d,%d].t", out.numel, M, N);
        if (db.numel == out.numel) {
            db += out;
        }
        else ERROR("db, out' dim?\n");
        dump(db.data, 1, db.numel, 1);
    } break;
    case L_FLATTEN: break;
    case L_RELU:    break;
    case L_TANH:    break;
    case L_SIGMOID: break;
    case L_SOFTMAX: break;
    case L_MAXPOOL:
    case L_AVGPOOL: 
    case L_MINPOOL: break;
    case L_DROPOUT: break;
    }
    hipDeviceSynchronize();
}
#endif  // T4_ENABLE_OBJ
//==========================================================================
