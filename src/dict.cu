/*! @file
  @brief
  cueForth - dictionary manager
*/
#include <iomanip>          // setw, setbase
#include "dict.h"

Dict::Dict() {
    hipMallocManaged(&_dict, sizeof(Code)*CUEF_DICT_SZ);
    hipMallocManaged(&_pmem, sizeof(U8)*CUEF_HEAP_SZ);
    GPU_CHK();
}
Dict::~Dict() {
    GPU_SYNC();
    hipFree(_pmem);
    hipFree(_dict);
}
///
/// dictionary search functions - can be adapted for ROM+RAM
///
__GPU__ int
Dict::find(const char *s, bool compile, bool ucase) {
	printf("find(%s) => ", s);
    for (int i = _didx - (compile ? 2 : 1); i >= 0; --i) {
    	const char *t = _dict[i].name;
    	if (ucase && STRCASECMP(t, s)==0) return i;
    	if (!ucase && STRCMP(t, s)==0) return i;
    }
    return -1;
}
///
/// debugging methods
///
__HOST__ void
Dict::to_s(std::ostream &fout, IU c) {
    fout << _dict[c].name << " " << c << (_dict[c].immd ? "* " : " ");
}
///
/// recursively disassemble colon word
///
__HOST__ void
Dict::see(std::ostream &fout, IU *cp, IU *ip, int dp) {
    IU   c  = ri(cp);
    Code *w = &_dict[c];
    fout << std::endl; for (int i=dp; i>0; i--) fout << "  ";       // indentation
    if (dp) fout << "[" << std::setw(2) << ri(ip) << ": ";          // ip offset
    else    fout << "[ ";
    fout << w->name;
    if (w->def) {                                                   // a colon word
        for (IU ip1=0, n=w->len; ip1<n; ip1+=sizeof(IU)) {          // walk through children
            IU *cp1 = (IU*)(pfa(c) + ip1);                          // next children node
            see(fout, cp1, &ip1, dp+1);                             // dive recursively
        }
    }
    switch (c) {
    case DOVAR: case DOLIT:
        fout << "= " << rd((DU*)(cp+1)); *ip += sizeof(DU); break;
    case DOSTR: case DOTSTR: {
    	char *s = (char*)(cp+1);
    	int  sz = strlen(s)+1;
        *ip += ALIGN2(sz);                                           // advance IP
        fout << "= \"" << s << "\"";
    } break;
    case BRAN: case ZBRAN: case DONEXT:
        fout << "j" << ri(cp+1); *ip += sizeof(IU); break;
    }
    fout << "] ";
}
///
/// display dictionary word list
///
__HOST__ void
Dict::words(std::ostream &fout) {
	fout << std::setbase(10);
    for (int i=0; i<_didx; i++) {
        if ((i%10)==0) { fout << std::endl; }
        to_s(fout, i);
    }
}
///
/// Forth pmem memory dump
/// TODO: dynamic parallel
///
#define C2H(c) { buf[x++] = i2h[(c)>>4]; buf[x++] = i2h[(c)&0xf]; }
#define IU2H(i){ C2H((i)>>8); C2H((i)&0xff); }
__HOST__ void
Dict::dump(std::ostream &fout, IU p0, int sz) {
	const char *i2h = "0123456789abcdef";
	char buf[80];
    for (IU i=ALIGN16(p0); i<=ALIGN16(p0+sz); i+=16) {
    	int x = 0;
    	buf[x++] = '\n'; IU2H(i); buf[x++] = ':'; buf[x++] = ' ';  // "%04x: "
        for (int j=0; j<16; j++) {
            U8 c = _pmem[i+j] & 0x7f;
            C2H(c);                                                // "%02x "
            buf[x++] = ' ';
            if (j%4==3) buf[x++] = ' ';
            buf[59+j]= (c==0x7f||c<0x20) ? '.' : c;                // %c
        }
        buf[75] = '\0';
        fout << buf;
    }
}
