#include "hip/hip_runtime.h"
/*! @file
  @brief
  cueForth Forth Vritual Machine implementation
*/
#include "dict.h"
#include "eforth.h"
///
/// Forth Virtual Machine operational macros
///
#define INT(f)    (static_cast<int>(f))       /** cast float to int                        */
#define I2DU(i)   (static_cast<DU>(i))        /** cast int back to float                   */
#define LWIP      (dict[-1].plen)             /** parameter field tail of latest word      */
#define JMPIP     (IP0 + *(IU*)IP)            /** branching target address                 */
#define IPOFF     ((IU)(IP - PMEM0))          /** IP offset relative parameter memory root */
#define FIND(s)   (dict.find(s, compile, ucase))

__GPU__
ForthVM::ForthVM(Istream *istr,	Ostream *ostr, Dict *dict0)
	: fin(*istr), fout(*ostr), dict(*dict0) {
	PMEM0 = IP0 = IP = dict.mem0();
	printf("dict=%p, mem0=%p\n", dict0, PMEM0);
}
///
/// Forth inner interpreter (colon word handler)
///
__GPU__ char*
ForthVM::next_word()  {     // get next idiom
    fin >> idiom; return idiom;
}
__GPU__ char*
ForthVM::scan(char c) {
    fin.get_idiom(idiom, c); return idiom;
}
__GPU__ void
ForthVM::nest(IU c) {
    rs.push(IP - PMEM0); rs.push(WP);       /// * setup call frame
    IP0 = IP = dict.pfa(WP=c);              // CC: this takes 30ms/1K, need work
//  try                                     // kernal does not support exception
    {                                       // CC: is dict[c] kept in cache?
        U8 *ipx = IP + dict[c].plen;        // CC: this saves 350ms/1M
        while (IP < ipx) {                  /// * recursively call all children
            IU c1 = *IP; IP += sizeof(IU);  // CC: cost of (ipx, c1) on stack?
            call(c1);                       ///> execute child word
        }                                   ///> can do IP++ if pmem unit is 16-bit
    }
//    catch(...) {}                         ///> protect if any exeception
    yield();                                ///> give other tasks some time
    IP0 = dict.pfa(WP=rs.pop());            /// * restore call frame
    IP  = PMEM0 + INT(rs.pop());
}
///
/// Dict compiler proxy functions to reduce verbosity
///
__GPU__ __INLINE__ void ForthVM::add_iu(IU i) { dict.add_iu(i); }
__GPU__ __INLINE__ void ForthVM::add_du(DU d) { dict.add_du(d); }
__GPU__ __INLINE__ void ForthVM::add_str(IU op, const char *s) {
	dict.add_iu(op); dict.add_str(s);
}
__GPU__ __INLINE__ void ForthVM::call(IU w) {
    if (dict[w].def) nest(w);
    else             (*(fop*)(((uintptr_t)dict[w].xt)&~0x3))(w);
}
///==============================================================================
///
/// debug functions
///
__GPU__ void
ForthVM::dot_r(int n, DU v) {
    fout << v;
}
///
/// Stack dump
///
__GPU__ void
ForthVM::ss_dump() {
    fout << " <"; for (int i=0; i<ss.idx; i++) { fout << ss[i] << " "; }
    fout << top << "> ok" << ENDL;
}
///================================================================================
///
/// macros to reduce verbosity
///
#define CODE(s, g)    { s, [this] __GPU__ (IU c){ g; }}
#define IMMD(s, g)    { s, [this] __GPU__ (IU c){ g; }, true }
#define BOOL(f)       ((f)?-1:0)
#define ALU(a, OP, b) (INT(a) OP INT(b))
///
/// global memory access macros
///
#define PEEK(a)        (U8)(*(U8*)((uintptr_t)(a)))
#define POKE(a, c)     (*(U8*)((uintptr_t)(a))=(U8)(c))
///
/// dictionary initializer
///
__GPU__ void
ForthVM::init() {
	const Code prim[] = {       /// singleton, build once only
    ///
    /// @defgroup Execution flow ops
    /// @brief - DO NOT change the sequence here (see forth_opcode enum)
    /// @{
    CODE("nop",     {}),
    CODE("dovar",   PUSH(IPOFF); IP += sizeof(DU)),
    CODE("dolit",   PUSH(dict.rd((DU*)IP)); IP += sizeof(DU)),
    CODE("dostr",
        char *s  = (char*)IP;                     // get string pointer
        int  sz  = STRLENB(s)+1;
        PUSH(IPOFF); IP += ALIGN2(sz)),
    CODE("dotstr",
        char *s  = (char*)IP;                     // get string pointer
        int  sz  = STRLENB(s)+1;
        fout << s;  IP += ALIGN2(sz)),            // send to output console
    CODE("branch" , IP = JMPIP),                           // unconditional branch
    CODE("0branch", IP = POP() ? IP + sizeof(IU) : JMPIP), // conditional branch
    CODE("donext",
         if ((rs[-1] -= 1) >= 0) IP = JMPIP;       // rs[-1]-=1 saved 200ms/1M cycles
         else { IP += sizeof(IU); rs.pop(); }),
    CODE("does",                                   // CREATE...DOES... meta-program
         IU *ip  = (IU*)dict.pfa(WP);
         IU *ipx = (IU*)((U8*)ip + dict[WP].plen);         // range check
         while (ip < ipx && dict.ri(ip) != DOES) ip++;     // find DOES
         while (++ip < ipx) add_iu(dict.ri(ip));           // copy&paste code
         IP = (U8*)ipx),                                   // done
    CODE(">r",   rs.push(POP())),
    CODE("r>",   PUSH(rs.pop())),
    CODE("r@",   PUSH(rs[-1])),
    /// @}
    /// @defgroup Stack ops
    /// @brief - opcode sequence can be changed below this line
    /// @{
    CODE("dup",  PUSH(top)),
    CODE("drop", top = ss.pop()),
    CODE("over", PUSH(ss[-1])),
    CODE("swap", DU n = ss.pop(); PUSH(n)),
    CODE("rot",  DU n = ss.pop(); DU m = ss.pop(); ss.push(n); PUSH(m)),
    CODE("pick", DU i = top; top = ss[-i]),
    /// @}
    /// @defgroup Stack ops - double
    /// @{
    CODE("2dup", PUSH(ss[-1]); PUSH(ss[-1])),
    CODE("2drop",ss.pop(); top = ss.pop()),
    CODE("2over",PUSH(ss[-3]); PUSH(ss[-3])),
    CODE("2swap",
        DU n = ss.pop(); DU m = ss.pop(); DU l = ss.pop();
        ss.push(n); PUSH(l); PUSH(m)),
    /// @}
    /// @defgroup FPU/ALU ops
    /// @{
    CODE("+",    top += ss.pop()),
    CODE("*",    top *= ss.pop()),
    CODE("-",    top =  ss.pop() - top),
    CODE("/",    top =  ss.pop() / top),
    CODE("mod",  top =  fmod(ss.pop(), top)),          /// fmod = x - int(q)*y
    CODE("*/",   top =  ss.pop() * ss.pop() / top),
    CODE("/mod",
        DU n = ss.pop(); DU t = top;
        ss.push(fmod(n, t)); top = round(n / t)),
    CODE("*/mod",
        DU n = ss.pop() * ss.pop();  DU t = top;
        ss.push(fmod(n, t)); top = round(n / t)),
    CODE("and",  top = I2DU(INT(ss.pop()) & INT(top))),
    CODE("or",   top = I2DU(INT(ss.pop()) | INT(top))),
    CODE("xor",  top = I2DU(INT(ss.pop()) ^ INT(top))),
    CODE("abs",  top = abs(top)),
    CODE("negate", top = -top),
    CODE("max",  DU n=ss.pop(); top = (top>n)?top:n),
    CODE("min",  DU n=ss.pop(); top = (top<n)?top:n),
    CODE("2*",   top *= 2),
    CODE("2/",   top /= 2),
    CODE("1+",   top += 1),
    CODE("1-",   top -= 1),
    /// @}
    /// @defgroup Logic ops
    /// @{
    CODE("0= ",  top = BOOL(top == 0)),
    CODE("0<",   top = BOOL(top <  0)),
    CODE("0>",   top = BOOL(top >  0)),
    CODE("=",    top = BOOL(ss.pop() == top)),
    CODE(">",    top = BOOL(ss.pop() >  top)),
    CODE("<",    top = BOOL(ss.pop() <  top)),
    CODE("<>",   top = BOOL(ss.pop() != top)),
    CODE(">=",   top = BOOL(ss.pop() >= top)),
    CODE("<=",   top = BOOL(ss.pop() <= top)),
    /// @}
    /// @defgroup IO ops
    /// @{
    CODE("base@",   PUSH(radix)),
    CODE("base!",   radix = POP()),
    CODE("hex",     radix = 16),
    CODE("decimal", radix = 10),
    CODE("cr",      fout << ENDL),
    CODE(".",       fout << POP() << " "),
    CODE(".r",      DU n = POP(); dot_r(n, POP())),
    CODE("u.r",     DU n = POP(); dot_r(n, abs(POP()))),
    CODE(".f",      DU n = POP(); fout << setprec(n) << POP()),
    CODE("key",     PUSH(next_word()[0])),
    CODE("emit",    char b = (char)POP(); fout << b),
    CODE("space",   fout << " "),
    CODE("spaces",  for (DU n = POP(), i = 0; i < n; i++) fout << " "),
    /// @}
    /// @defgroup Literal ops
    /// @{
    CODE("[",       compile = false),
    CODE("]",       compile = true),
    IMMD("(",       scan(')')),
    IMMD(".(",      fout << scan(')')),
    CODE("\\",      scan('\n')),
    CODE("$\"",
        const char *s = scan('"')+1;        // string skip first blank
        add_str(DOSTR, s)),                 // dostr, (+parameter field)
    IMMD(".\"",
        const char *s = scan('"')+1;        // string skip first blank
        add_str(DOTSTR, s)),                // dotstr, (+parameter field)
    /// @}
    /// @defgroup Branching ops
    /// @brief - if...then, if...else...then
    /// @{
    IMMD("if",      add_iu(ZBRAN); PUSH(LWIP); add_iu(0)),  // if   ( -- here )
    IMMD("else",                                                      // else ( here -- there )
        add_iu(BRAN);
        IU h=LWIP;  add_iu(0); dict.setjmp(INT(POP())); PUSH(h)),
    IMMD("then",    dict.setjmp(INT(POP()))),                         // backfill jump address
    /// @}
    /// @defgroup Loops
    /// @brief  - begin...again, begin...f until, begin...f while...repeat
    /// @{
    IMMD("begin",   PUSH(LWIP)),
    IMMD("again",   add_iu(BRAN);  add_iu(POP())),          // again    ( there -- )
    IMMD("until",   add_iu(ZBRAN); add_iu(POP())),          // until    ( there -- )
    IMMD("while",   add_iu(ZBRAN); PUSH(LWIP); add_iu(0)),  // while    ( there -- there here )
    IMMD("repeat",  add_iu(BRAN);                           // repeat    ( there1 there2 -- )
        IU t=POP(); add_iu(POP()); dict.setjmp(t)),         // set forward and loop back address
    /// @}
    /// @defgrouop For loops
    /// @brief  - for...next, for...aft...then...next
    /// @{
    IMMD("for" ,    add_iu(TOR); PUSH(LWIP)),               // for ( -- here )
    IMMD("next",    add_iu(DONEXT); add_iu(POP())),         // next ( here -- )
    IMMD("aft",                                             // aft ( here -- here there )
        POP(); add_iu(BRAN);
        IU h=LWIP; add_iu(0); PUSH(LWIP); PUSH(h)),
    /// @}
    /// @defgrouop Compiler ops
    /// @{
    CODE(":", dict.colon(next_word()); compile=true),
    IMMD(";", compile = false),
    CODE("variable",                                        // create a variable
        dict.colon(next_word());                            // create a new word on dictionary
        add_iu(DOVAR);                                      // dovar (+parameter field)
        add_du(0)),                                         // data storage (32-bit integer now)
    CODE("constant",                                        // create a constant
        dict.colon(next_word());                            // create a new word on dictionary
        add_iu(DOLIT);                                      // dovar (+parameter field)
        add_du(POP())),                                     // data storage (32-bit integer now)
    /// @}
    /// @defgroup metacompiler
    /// @brief - dict is directly used, instead of shield by macros
    /// @{
    CODE("exit",  IP = dict.pfa(WP) + dict[WP].plen),       // quit current word execution
    CODE("exec",  call(POP())),                             // execute word
    CODE("create",
        dict.colon(next_word());                            // create a new word on dictionary
        add_iu(DOVAR)),                                     // dovar (+ parameter field)
    CODE("to",              // 3 to x                       // alter the value of a constant
        IU w = FIND(next_word());                           // to save the extra @ of a variable
        dict.wd((DU*)(dict.pfa(w) + sizeof(IU)), POP())),
    CODE("is",              // ' y is x                     // alias a word
        IU w = FIND(next_word());                           // can serve as a function pointer
        dict.wi((IU*)dict.pfa(POP()), dict[w].pidx)),       // but might leave a dangled block
    CODE("[to]",            // : xx 3 [to] y ;              // alter constant in compile mode
        IU w = dict.ri((IU*)IP); IP += sizeof(IU);          // fetch constant pfa from 'here'
        dict.wd((DU*)(dict.pfa(w) + sizeof(IU)), POP())),
    ///
    /// be careful with memory access, especially BYTE because
    /// it could make access misaligned which slows the access speed by 2x
    ///
    CODE("@",     IU w = POP(); PUSH(dict.rd(w))),                                 // w -- n
    CODE("!",     IU w = POP(); dict.wd(w, POP())),                                // n w --
    CODE(",",     DU n = POP(); add_du(n)),
    CODE("allot", DU v = 0; for (IU n = POP(), i = 0; i < n; i++) add_du(v)),      // n --
    CODE("+!",    IU w = POP(); dict.wd(w, dict.rd(w)+POP())),                     // n w --
    CODE("?",     IU w = POP(); fout << dict.rd(w) << " "),                        // w --
    /// @}
    /// @defgroup Debug ops
    /// @{
    CODE("here",  PUSH(dict.here())),
    CODE("ucase", ucase = POP()),
//    CODE("words", dict.words()),
    CODE("'",     IU w = FIND(next_word()); PUSH(w)),
    CODE(".s",    ss_dump()),
//    CODE("see",   IU w = FIND(next_word()); IU ip=0; dict.see(&w, &ip)),
//    CODE("dump",  DU n = POP(); IU a = POP(); dict.dump(a, INT(n))),
    CODE("forget",
        int w = FIND(next_word());
        if (w<0) return;
        IU b = FIND("boot")+1;
        dict.clear(w > b ? w : b)),
#if ARDUINO
    /// @}
    /// @defgroup Arduino specific ops
    /// @{
    CODE("pin",   DU p = POP(); pinMode(p, POP())),
    CODE("in",    PUSH(digitalRead(POP()))),
    CODE("out",   DU p = POP(); digitalWrite(p, POP())),
    CODE("adc",   PUSH(analogRead(POP()))),
    CODE("duty",  DU p = POP(); analogWrite(p, POP(), 255)),
    CODE("attach",DU p  = POP(); ledcAttachPin(p, POP())),
    CODE("setup", DU ch = POP(); DU freq=POP(); ledcSetup(ch, freq, POP())),
    CODE("tone",  DU ch = POP(); ledcWriteTone(ch, POP())),
#endif // ARDUINO
    /// @}
    /// @defgroup System ops
    /// @{
    CODE("peek",  DU a = POP(); PUSH(PEEK(a))),
    CODE("poke",  DU a = POP(); POKE(a, POP())),
    CODE("clock", PUSH(millis())),
    CODE("delay", delay(POP())),             // TODO: change to VM_WAIT
    CODE("bye",   status = VM_STOP),
    CODE("boot",  dict.clear(FIND("boot") + 1))
    /// @}
    };
	for (int i=0; i<sizeof(prim)/sizeof(Code); i++) {
	    dict.add_code((Code*)&prim[i]);
	    printf("%3d> %p %s\n", i, dict[i].name, dict[i].name);   // dump dictionary from device
	}
    status = VM_RUN;
    
    printf("init() this=%p sizeof(Code)=%d\n", this, sizeof(Code));
};
///
/// ForthVM Outer interpreter
///
__GPU__ void
ForthVM::outer() {
    while (fin >> idiom) {                   /// loop throught tib
        printf("%d>> %s => ", blockIdx.x, idiom);
        int w = FIND(idiom);                 /// * search through dictionary
        if (w>=0) {                          /// * word found?
            printf("%p %s %d\n", dict[w].xt, dict[w].name, w);
            if (compile && !dict[w].immd) {  /// * in compile mode?
                add_iu(w);                   /// * add found word to new colon word
            }
            else call(w);                    /// * execute forth word
            continue;
        }
        // try as a number
        char *p;
        int n = INT(STRTOL(idiom, &p, radix));
        printf("%d\n", n);
        if (*p != '\0') {                    /// * not number
            fout << idiom << "? " << ENDL;   ///> display error prompt
            compile = false;                 ///> reset to interpreter mode
            break;                           ///> skip the entire input buffer
        }
        // is a number
        if (compile) {                       /// * add literal when in compile mode
            add_iu(DOLIT);                   ///> dovar (+parameter field)
            add_du(n);                       ///> data storage (32-bit integer now)
        }
        else PUSH(n);                        ///> or, add value onto data stack
    }
    if (!compile) ss_dump();
    __syncthreads();
}
//=======================================================================================
