#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @file
 * @brief MMU class - memory manager implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include <iomanip>             // setw, setbase
#include "mmu.h"
///
/// Forth Virtual Machine operational macros to reduce verbosity
///
__HOST__
MMU::MMU() {
    MM_ALLOC(&_dict, sizeof(Code) * T4_DICT_SZ);
    MM_ALLOC(&_vmss, sizeof(DU) * T4_SS_SZ * VM_COUNT);
    MM_ALLOC(&_vmrs, sizeof(DU) * T4_RS_SZ * VM_COUNT);
    MM_ALLOC(&_pmem, T4_PMEM_SZ);
    
#if T4_ENABLE_OBJ    
    MM_ALLOC(&_mark, sizeof(DU) * T4_TFREE_SZ);
    MM_ALLOC(&_obj,  T4_OSTORE_SZ);
    _ostore.init(_obj, T4_OSTORE_SZ);
#endif // T4_ENABLE_OBJ
    
    MM_TRACE1(
        "\\ MMU: CUDA Managed Memory\n"
        "\\\tdict=%p\n"
        "\\\tvmss=%p\n"
        "\\\tvmrs=%p\n"
        "\\\tmem =%p\n"
        "\\\tmark=%p\n"
        "\\\tobj =%p\n",
        _dict, _vmss, _vmrs, _pmem, _mark, _obj);
}
__HOST__
MMU::~MMU() {
    GPU_SYNC();
    MM_TRACE1("\\ MMU releasing CUDA Managed Memory...\n");
    if (_obj)  MM_FREE(_obj);
    if (_mark) MM_FREE(_mark);
    MM_FREE(_pmem);
    MM_FREE(_vmrs);
    MM_FREE(_vmss);
    MM_FREE(_dict);
}
///
/// dictionary management methods
/// TODO: use const Code[] directly, as ROM, to prevent deep copy
///
__GPU__ int
MMU::find(const char *s, bool compile) {
    MM_TRACE2("find(%s) =>", s);
    for (int i = _didx - (compile ? 2 : 1); i >= 0; --i) {
        const char *t = _dict[i].name;
        if (STRCMP(t, s)==0) return i;
    }
    return -1;
}
__GPU__ void
MMU::status() {
    UFP x0 = ~0;                            ///< base of xt   allocations
    UFP n0 = ~0;                            ///< base of name allocations

    Code *c = _dict;
    for (int i=0; i<_didx; i++, c++) {      /// * scan thru for max range
        if ((UFP)c->xt   < x0) x0 = (UFP)c->xt;
        if ((UFP)c->name < n0) n0 = (UFP)c->name;
    }
    c = _dict;
    MM_TRACE2("Built-in Dictionary [name0=%lx, xt0=%lx]\n", n0, x0);
    for (int i=0; i<_didx; i++, c++) {      ///< dump dictionary from device
        MM_TRACE2("%4d> name=%5x, xt=%5x %s\n", i,
            (U32)((UFP)c->name - n0),
            (U32)((UFP)c->xt   - x0),
            c->name);
    }

    INFO("\\  MMU.stat dict[%d/%d], pmem[%d]=%0.1f%%, tfree[%d/%d]\n",
        _didx, T4_DICT_SZ, _midx, 100.0*(_midx/T4_PMEM_SZ), _fidx, T4_TFREE_SZ);
    ///
    /// display object store statistics
    ///
#if T4_ENABLE_OBJ    
    _ostore.status(_trace);
#endif // T4_ENABLE_OBJ
}
///
/// colon - dictionary word compiler
///
__GPU__ void
MMU::colon(const char *name) {
    MM_TRACE2("colon(%s) => ", name);
    int  sz = STRLENB(name);                // aligned string length
    Code &c = _dict[_didx++];               // get next dictionary slot
    align();                                // nfa 32-bit aligned (adjust _midx)
    c.didx = _didx-1;                       // directory index (reverse link)
    c.nfa  = _midx;                         // name field offset
    c.name = (const char*)&_pmem[_midx];    // assign name field index
    c.colon= 1;                             // specify a colon word
    add((U8*)name,  ALIGN2(sz+1));          // setup raw name field
    c.pfa  = _midx;                         // parameter field offset
}
///
/// tensor life-cycle methods
///
#if T4_ENABLE_OBJ // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
#include "model.h"             // in ../nn, include ../mmu/mmu.h
#include "dataset.h"           // in ../nn

#define OBJ2X(t)  ((U32)((U8*)&(t) - _obj))
__GPU__ void
MMU::mark_free(DU v) {            ///< mark a tensor free for release
    if (IS_VIEW(v)) return;
    T4Base &t = du2obj(v);
    MM_TRACE1("mmu#mark T[%x] to free[%d]\n", OBJ2X(t), _fidx);
//    lock();
    if (_fidx < T4_TFREE_SZ) _mark[_fidx++] = obj2du(t);
    else ERROR("ERR: tfree store full, increase T4_TFREE_SZ!");
//    unlock();                   ///< TODO: CC: DEAD LOCK, now!
}
__GPU__ void                      ///< release marked free tensor
MMU::sweep() {
//    lock();
    for (int i = 0; _fidx && i < _fidx; i++) {
        DU v = _mark[i];
        MM_TRACE1("mmu#release T[%x] from free[%d]\n", DU2X(v) & ~T4_TT_OBJ, i);
        drop(v);
    }
    _fidx = 0;
//  unlock();                      ///< TODO: CC: DEAD LOCK, now!
}
__GPU__ Tensor&                    ///< allocate a tensor from tensor space
MMU::talloc(U32 sz) {
    Tensor &t = *(Tensor*)_ostore.malloc(sizeof(Tensor));
    MM_TRACE1(" T[%x]", OBJ2X(t));
    void   *d = _ostore.malloc((U64)sizeof(DU) * sz);
    _ostore.status(_trace);
    t.reset(d, sz);
    return t;
}
__GPU__ Tensor&                    ///< create a one-dimensional tensor
MMU::tensor(U32 sz) {
    MM_TRACE1("mmu#tensor(%d) numel=%d", sz, sz);
    return talloc(sz);
}
__GPU__ Tensor&                    ///< create a 2-dimensional tensor
MMU::tensor(U16 h, U16 w) {
    U32 sz = h * w;
    MM_TRACE1("mmu#tensor(%d,%d) numel=%d", h, w, sz);
    Tensor &t = talloc(sz);
    t.reshape(h, w);
    return t;
}
__GPU__ Tensor&                    ///< create a NHWC tensor
MMU::tensor(U16 n, U16 h, U16 w, U16 c) {
    U32 sz = n * h * w * c;
    MM_TRACE1("mmu#tensor(%d,%d,%d,%d) numel=%d", n, h, w, c, sz);
    Tensor &t = talloc(sz);
    t.reshape(n, h, w, c);
    return t;
}
__GPU__ void
MMU::resize(Tensor &t, U32 sz) {
    if (t.rank != 1) { ERROR("mmu#resize rank==1 only\n"); return; }
    MM_TRACE1("mmu#resize numel=%d (was %d)", sz, t.numel);
    DU *d0 = t.data;             /// * keep original memory block
    t.data = (DU*)_ostore.malloc(sz * sizeof(DU));
    ///
    /// hardcopy tensor storage
    ///
    memcpy(t.data, d0, (t.numel < sz ? t.numel : sz) * sizeof(DU));
    t.H() = t.numel = sz;        /// * adjust tensor storage size
    
    _ostore.free(d0);            /// * release 
    _ostore.status(_trace);
}
__GPU__ void                     ///< release tensor memory blocks
MMU::free(Tensor &t) {
    MM_TRACE1("mmu#free(T%d) numel=%d T[%x]", t.rank, t.numel, OBJ2X(t));
    _ostore.free(t.data);        /// * free physical data
    if (t.grad_fn != L_NONE) {
        MM_TRACE1(" {\n");
        for (int i=0; t.mtum[i] && i < 4; i++) {
            if (t.mtum[i] == t.grad[i]) continue;   /// * dummy pointers for SGD
            MM_TRACE1("\t\t"); free(*t.mtum[i]);
        }
        for (int i=0; t.grad[i] && i < 4; i++) {
            MM_TRACE1("\t\t"); free(*t.grad[i]);    /// recursive
        }
        MM_TRACE1("\t}");
    }
    _ostore.free(&t);              /// * free tensor object itself
    _ostore.status(_trace);
}
#if T4_ENABLE_NN
__GPU__ Model&                     ///< create a NN model with NHWC input
MMU::model(U32 sz) {
    MM_TRACE1("mmu#model layers=%d", sz);
    Model  *m = (Model*)_ostore.malloc(sizeof(Model));
    Tensor &t = talloc(sz);        /// * allocate tensor storage
    m->reset(this, t);
    return *m;
}
__GPU__ Dataset&                   ///< create a Dataset holder
MMU::dataset(U16 batch_sz) {       /// * Note: data block is not allocated yet
    MM_TRACE1("mmu#dataset batch_sz=%d", batch_sz);
    Dataset *ds = (Dataset*)_ostore.malloc(sizeof(Dataset));
    ds->init(0, T4_DATASET, 4);
    ds->N()      = batch_sz;       /// * other members filled in host mode
    ds->batch_id = 0;              /// * setup control flag
    _ostore.status(_trace);
    return *ds;
}
__GPU__ void                     ///< release tensor memory blocks
MMU::free(Model &m) {
    MM_TRACE1("mmu#free(N%d) [\n", m.numel);
    for (int i = m.numel-1; i >= 0; i--) {
        MM_TRACE1("\t"); free(m[i]);
    }
    MM_TRACE1("]");
    _ostore.free(&m);
    _ostore.status(_trace);
}
#endif // T4_ENABLE_NN
///
/// deep copy a tensor
/// TODO: CDP
///
__GPU__ Tensor&
MMU::copy(Tensor &t0) {
    if (!t0.is_tensor()) return t0;    ///> skip, TODO: copy model

    Tensor &t1  = *(Tensor*)_ostore.malloc(sizeof(Tensor));
    memcpy(&t1, &t0, sizeof(Tensor));   /// * copy attributes
    ///
    /// set attributes
    ///
    for (int i=0; i<4; i++) t1.grad[i] = t1.mtum[i] = NULL;  /// * blank gradients
    t1.grad_fn = L_NONE;                /// * not a network layer
    t1.nref    = 1;                     /// * reset ref counter
    ///
    /// hard copy data block
    ///
    U64 bsz = sizeof(DU) * t0.numel;
    t1.data = (DU*)_ostore.malloc(bsz);
    t1 = t0;                            /// * copy all tensor elements
    
    MM_TRACE1("mmu#copy(T%d) numel=%d to T[%x]", t0.rank, t0.numel, OBJ2X(t1));
    _ostore.status(_trace);
    
    return t1;
}
__GPU__ Tensor&
MMU::random(Tensor &t, t4_rand_opt ntype, DU bias, DU scale) {
    MM_TRACE2("mmu#random(T%d) numel=%d bias=%.2f, scale=%.2f\n",
              t.rank, t.numel, bias, scale);
    k_rand<<<1, T4_RAND_SZ>>>(t.data, t.numel, bias, scale, _seed, ntype);
    GPU_SYNC();
    
    return t;
}
///
/// short hands for eforth tensor ucodes (for DU <-> Tensor conversion)
/// TODO: more object types
///
__GPU__ DU
MMU::dup(DU d)  { return IS_OBJ(d) ? AS_VIEW(d) : d; }
__GPU__ DU
MMU::copy(DU d) { return IS_OBJ(d) ? obj2du(copy((Tensor&)du2obj(d))) : d; }
__GPU__ void
MMU::drop(DU d) {
    if (!IS_OBJ(d) || IS_VIEW(d)) return;     /// non-object, just drop
    
    T4Base &t = du2obj(d);                    /// check reference count
#if T4_ENABLE_NN
    if (t.is_model()) free((Model&)t);        /// release TLSF memory block
    else              free((Tensor&)t);
#else  // T4_ENABLE_NN
    free((Tensor&)t);
#endif // T4_ENABLE_NN
}
///
/// tensor slice & dice
/// TODO: CDP
///
__GPU__ Tensor&
MMU::slice(Tensor &t0, U16 x0, U16 x1, U16 y0, U16 y1) {
    if (t0.rank < 2) { ERROR("dim?"); return t0; }
    if (x1 == (U16)-1) x1 = t0.W();
    if (y1 == (U16)-1) y1 = t0.H();
    Tensor &t1 = t0.rank==2
        ? tensor(y1-y0, x1-x0)
        : tensor(t0.N(), y1-y0, x1-x0, t0.C());
    ///
    /// hard copy data blocks
    ///
    U16 N   = t1.N(), C = t1.C();
    U64 bsz = sizeof(DU) * C * t1.W();              // size of one row
    for (int n = 0; n < N; n++) {                   // repeat N HWC
        for (int j = y0, j0=0; j < y1; j++, j0++) {
            DU *d0 = &t0.data[C * (j * t0.W() + x0)];
            DU *d1 = &t1.data[C * j0 * t1.W()];
            memcpy(d1, d0, bsz);
        }
    }
    MM_TRACE1("mmu#slice(T%d)[%d:%d,%d:%d,] numel=%d\n",
              t0.rank, t0.numel, x0, x1, y0, y1);
    return t1;
}
#endif // T4_ENABLE_OBJ // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
