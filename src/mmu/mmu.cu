#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @file
 * @brief MMU class - memory manager implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include <iomanip>             // setw, setbase
#include "mmu.h"

///@name static class member
///@note: CUDA does not support device static data
///@{
__GPU__  UFP _XT0;
__GPU__  UFP _NM0;
///@}
///
/// Forth Virtual Machine operational macros to reduce verbosity
///
__HOST__
MMU::MMU() {
    MM_ALLOC(&_dict, sizeof(Code) * T4_DICT_SZ);
    MM_ALLOC(&_vmss, sizeof(DU) * T4_SS_SZ * T4_VM_COUNT);
    MM_ALLOC(&_vmrs, sizeof(DU) * T4_RS_SZ * T4_VM_COUNT);
    MM_ALLOC(&_pmem, T4_PMEM_SZ);
    
#if T4_ENABLE_OBJ    
    MM_ALLOC(&_mark, sizeof(DU) * T4_TFREE_SZ);
    MM_ALLOC(&_obj,  T4_OSTORE_SZ);
    _ostore.init(_obj, T4_OSTORE_SZ);
#endif // T4_ENABLE_OBJ

    _midx = T4_USER_AREA;      // set aside user area (for base and maybe compile)
    
    TRACE(
        "\\ MMU: CUDA Managed Memory\n"
        "\\\tdict=%p\n"
        "\\\tvmss=%p\n"
        "\\\tvmrs=%p\n"
        "\\\tmem =%p\n"
        "\\\tmark=%p\n"
        "\\\tobj =%p\n",
        _dict, _vmss, _vmrs, _pmem, _mark, _obj);
}
__HOST__
MMU::~MMU() {
    if (_obj)  MM_FREE(_obj);
    if (_mark) MM_FREE(_mark);
    MM_FREE(_pmem);
    MM_FREE(_vmrs);
    MM_FREE(_vmss);
    MM_FREE(_dict);
    TRACE("\\ MMU: CUDA Managed Memory freed\n");
}
///
/// static functions (for type conversion)
///
__GPU__  FPTR MMU::XT(IU ioff)      { return (FPTR)(_XT0 + ioff);  }
__GPU__  IU   MMU::XTOFF(FPTR xt)   { return (IU)((UFP)xt - _XT0); }
///
/// dictionary management methods
/// TODO: use const Code[] directly, as ROM, to prevent deep copy
///
__GPU__ void
MMU::dict_validate() {
    UFP  x0 = ~0;                           ///< base of xt   allocations
    UFP  n0 = ~0;
    Code *c = _dict;
    for (int i=0; i < _didx; i++, c++) {    /// * scan thru for max range
        if ((UFP)c->xt   < x0) x0 = (UFP)c->xt;
        if ((UFP)c->name < n0) n0 = (UFP)c->name;
    }
    _XT0 = x0;
    _NM0 = n0;
    _dict[0].xt = (FPTR)x0;                 /// * borrow for xt0
}

__GPU__ IU
MMU::find(const char *s) {
    IU v = 0;
    DEBUG("mmu.find(%s) => ", s);
    for (IU i = _didx - 1; _didx && !v && i > 0; --i) {
        if (STRCMP(_dict[i].name, s)==0) v = i;
    }
    return v;
}

__GPU__ void
MMU::status() {
    INFO("\\ MMU.stat dict[%d/%d], pmem[%d]=%0.1f%%, tfree[%d/%d]\n",
        _didx, T4_DICT_SZ, _midx, 100.0*(_midx/T4_PMEM_SZ), _fidx, T4_TFREE_SZ);
    ///
    /// display object store statistics
    ///
#if T4_ENABLE_OBJ    
    _ostore.status(_trace);
#endif // T4_ENABLE_OBJ
}

__GPU__ void
MMU::dict_dump() {
    Code *c = _dict;
    DEBUG("Built-in Dictionary [name0=0x%lx, xt0=0x%lx]\n", _NM0, _XT0);
    for (int i=0; i<_didx; i++, c++) {      ///< dump dictionary from device
        IU  ix = c->udf ? c->pfa : (U32)(((UFP)c->xt & MSK_XT) - _XT0);
        U32 sz = ALIGN(STRLEN(c->name) + 1);
        DEBUG("%4d|%03x> name=%6x, %s=%6x %s\n", i, i,
              c->udf ? (c->pfa - sz) : (U32)((UFP)c->name - _NM0),
              c->udf ? "pf" : "xt", ix, c->name);
    }
}
///
/// colon - dictionary word compiler
///
__GPU__ void
MMU::colon(const char *name) {
    DEBUG("colon(%s) => ", name);
    int  sz = STRLENB(name);                // aligned string length
    Code &c = _dict[_didx++];               // get next dictionary slot
    align();                                // nfa 32-bit aligned (adjust _midx)
    c.didx = _didx-1;                       // directory index (reverse link)
    c.nfa  = _midx;                         // name field offset
    c.name = (const char*)&_pmem[_midx];    // assign name field index
    c.udf  = 1;                             // specify a colon word
    add((U8*)name,  ALIGN(sz+1));           // setup raw name field
    c.pfa  = _midx;                         // parameter field offset
}
///
/// tensor life-cycle methods
///
#if T4_ENABLE_OBJ // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
#include "model.h"             // in ../nn, include ../mmu/mmu.h
#include "dataset.h"           // in ../nn

#define OBJ2X(t)  ((U32)((U8*)&(t) - _obj))
__GPU__ void
MMU::mark_free(DU v) {            ///< mark a tensor free for release
    if (IS_VIEW(v)) return;
    T4Base &t = TBase::du2obj(v);
    DEBUG("mmu#mark T[%x] to free[%d]\n", OBJ2X(t), _fidx);
//    lock();
    if (_fidx < T4_TFREE_SZ) _mark[_fidx++] = T4base::obj2du(t);
    else ERROR("ERR: tfree store full, increase T4_TFREE_SZ!");
//    unlock();                   ///< TODO: CC: DEAD LOCK, now!
}
__GPU__ void                      ///< release marked free tensor
MMU::sweep() {
//    lock();
    for (int i = 0; _fidx && i < _fidx; i++) {
        DU v = _mark[i];
        DEBUG("mmu#release T[%x] from free[%d]\n", DU2X(v) & ~T4_TT_OBJ, i);
        drop(v);
    }
    _fidx = 0;
//  unlock();                      ///< TODO: CC: DEAD LOCK, now!
}
__GPU__ Tensor&                    ///< allocate a tensor from tensor space
MMU::talloc(U32 sz) {
    Tensor &t = *(Tensor*)_ostore.malloc(sizeof(Tensor));
    DEBUG(" T[%x]", OBJ2X(t));
    void   *d = _ostore.malloc((U64)sizeof(DU) * sz);
    _ostore.status(_trace);
    t.reset(d, sz);
    return t;
}
__GPU__ Tensor&                    ///< create a one-dimensional tensor
MMU::tensor(U32 sz) {
    DEBUG("mmu#tensor(%d) numel=%d", sz, sz);
    return talloc(sz);
}
__GPU__ Tensor&                    ///< create a 2-dimensional tensor
MMU::tensor(U16 h, U16 w) {
    U32 sz = h * w;
    DEBUG("mmu#tensor(%d,%d) numel=%d", h, w, sz);
    Tensor &t = talloc(sz);
    t.reshape(h, w);
    return t;
}
__GPU__ Tensor&                    ///< create a NHWC tensor
MMU::tensor(U16 n, U16 h, U16 w, U16 c) {
    U32 sz = n * h * w * c;
    DEBUG("mmu#tensor(%d,%d,%d,%d) numel=%d", n, h, w, c, sz);
    Tensor &t = talloc(sz);
    t.reshape(n, h, w, c);
    return t;
}
__GPU__ void
MMU::resize(Tensor &t, U32 sz) {
    if (t.rank != 1) { ERROR("mmu#resize rank==1 only\n"); return; }
    DEBUG("mmu#resize numel=%d (was %d)", sz, t.numel);
    DU *d0 = t.data;             /// * keep original memory block
    t.data = (DU*)_ostore.malloc(sz * sizeof(DU));
    ///
    /// hardcopy tensor storage
    ///
    memcpy(t.data, d0, (t.numel < sz ? t.numel : sz) * sizeof(DU));
    t.H() = t.numel = sz;        /// * adjust tensor storage size
    
    _ostore.free(d0);            /// * release 
    _ostore.status(_trace);
}
__GPU__ void                     ///< release tensor memory blocks
MMU::free(Tensor &t) {
    DEBUG("mmu#free(T%d) numel=%d T[%x]", t.rank, t.numel, OBJ2X(t));
    _ostore.free(t.data);        /// * free physical data
    if (t.grad_fn != L_NONE) {
        DEBUG(" {\n");
        for (int i=0; t.mtum[i] && i < 4; i++) {
            if (t.mtum[i] == t.grad[i]) continue;   /// * dummy pointers for SGD
            DEBUG("\t\t"); free(*t.mtum[i]);
        }
        for (int i=0; t.grad[i] && i < 4; i++) {
            DEBUG("\t\t"); free(*t.grad[i]);    /// recursive
        }
        DEBUG("\t}");
    }
    _ostore.free(&t);              /// * free tensor object itself
    _ostore.status(_trace);
}
#if T4_ENABLE_NN
__GPU__ Model&                     ///< create a NN model with NHWC input
MMU::model(U32 sz) {
    DEBUG("mmu#model layers=%d", sz);
    Model  *m = (Model*)_ostore.malloc(sizeof(Model));
    Tensor &t = talloc(sz);        /// * allocate tensor storage
    m->reset(this, t);
    return *m;
}
__GPU__ Dataset&                   ///< create a Dataset holder
MMU::dataset(U16 batch_sz) {       /// * Note: data block is not allocated yet
    DEBUG("mmu#dataset batch_sz=%d", batch_sz);
    Dataset *ds = (Dataset*)_ostore.malloc(sizeof(Dataset));
    ds->init(0, T4_DATASET, 4);
    ds->N()      = batch_sz;       /// * other members filled in host mode
    ds->batch_id = 0;              /// * setup control flag
    _ostore.status(_trace);
    return *ds;
}
__GPU__ void                     ///< release tensor memory blocks
MMU::free(Model &m) {
    DEBUG("mmu#free(N%d) [\n", m.numel);
    for (int i = m.numel-1; i >= 0; i--) {
        DEBUG("\t"); free(m[i]);
    }
    DEBUG("]");
    _ostore.free(&m);
    _ostore.status(_trace);
}
#endif // T4_ENABLE_NN
///
/// deep copy a tensor
/// TODO: CDP
///
__GPU__ Tensor&
MMU::copy(Tensor &t0) {
    if (!t0.is_tensor()) return t0;    ///> skip, TODO: copy model

    Tensor &t1  = *(Tensor*)_ostore.malloc(sizeof(Tensor));
    memcpy(&t1, &t0, sizeof(Tensor));   /// * copy attributes
    ///
    /// set attributes
    ///
    for (int i=0; i<4; i++) t1.grad[i] = t1.mtum[i] = NULL;  /// * blank gradients
    t1.grad_fn = L_NONE;                /// * not a network layer
    t1.nref    = 1;                     /// * reset ref counter
    ///
    /// hard copy data block
    ///
    U64 bsz = sizeof(DU) * t0.numel;
    t1.data = (DU*)_ostore.malloc(bsz);
    t1 = t0;                            /// * copy all tensor elements
    
    DBUG("mmu#copy(T%d) numel=%d to T[%x]", t0.rank, t0.numel, OBJ2X(t1));
    _ostore.status(_trace);
    
    return t1;
}
__GPU__ Tensor&
MMU::random(Tensor &t, t4_rand_opt ntype, DU bias, DU scale) {
    DEBUG("mmu#random(T%d) numel=%d bias=%.2f, scale=%.2f\n",
              t.rank, t.numel, bias, scale);
    k_rand<<<1, T4_RAND_SZ>>>(t.data, t.numel, bias, scale, _seed, ntype);
    GPU_SYNC();
    
    return t;
}
///
/// tensor slice & dice
/// TODO: CDP
///
__GPU__ Tensor&
MMU::slice(Tensor &t0, U16 x0, U16 x1, U16 y0, U16 y1) {
    if (t0.rank < 2) { ERROR("dim?"); return t0; }
    if (x1 == (U16)-1) x1 = t0.W();
    if (y1 == (U16)-1) y1 = t0.H();
    Tensor &t1 = t0.rank==2
        ? tensor(y1-y0, x1-x0)
        : tensor(t0.N(), y1-y0, x1-x0, t0.C());
    ///
    /// hard copy data blocks
    ///
    U16 N   = t1.N(), C = t1.C();
    U64 bsz = sizeof(DU) * C * t1.W();              // size of one row
    for (int n = 0; n < N; n++) {                   // repeat N HWC
        for (int j = y0, j0=0; j < y1; j++, j0++) {
            DU *d0 = &t0.data[C * (j * t0.W() + x0)];
            DU *d1 = &t1.data[C * j0 * t1.W()];
            memcpy(d1, d0, bsz);
        }
    }
    DEBUG("mmu#slice(T%d)[%d:%d,%d:%d,] numel=%d\n",
              t0.rank, t0.numel, x0, x1, y0, y1);
    return t1;
}
#endif // T4_ENABLE_OBJ // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
