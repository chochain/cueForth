#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief tensorForth Async IO module implementation
 *
 * <pre>Copyright (C) 2021- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include <cstdio>        // printf
#include <iostream>      // cin, cout
#include <iomanip>       // setbase, setprecision
#include "model.h"
#include "aio.h"
///
/// AIO takes managed memory blocks as input and output buffers
/// which can be access by both device and host
///
using namespace std;

__HOST__ int
AIO::readline() {
    _istr->clear();
    char *tib = _istr->rdbuf();
    cin.getline(tib, T4_IBUF_SZ, '\n');
    return strlen(tib);
}

__HOST__ void
AIO::print_node(obuf_node *node) {
    hipDeviceSynchronize();        /// * make sure data is completely written
    char *v = (char*)node->data;
    switch (node->gt) {
    case GT_INT:   cout << (*(I32*)v); break;
    case GT_FLOAT: cout << (*(F32*)v); break;
    case GT_STR:   cout << v;          break;
    case GT_FMT:   {
        obuf_fmt *f = (obuf_fmt*)v;
        //printf("FMT: b=%d, w=%d, p=%d, f='%c'\n", f->base, f->width, f->prec, f->fill);
        cout << std::setbase(_radix = f->base)
             << std::setw(f->width)
             << std::setprecision(f->prec ? f->prec : -1)
             << std::setfill((char)f->fill);
    } break;
    case GT_OBJ: _print_obj(*(DU*)v); break;
    case GT_OPX: {
        _opx *o = (_opx*)v;
        // printf("OP=%d a=%d, n=0x%08x=%f\n", o->op, o->a, DU2X(o->n), o->n);
        switch (o->op) {
        case OP_WORDS: _mmu->words(cout);                               break;
        case OP_SEE:   _mmu->see(cout, (IU)o->a);                       break;
        case OP_DUMP:  _mmu->mem_dump(cout, (IU)o->a, (IU)o->n);        break;
        case OP_SS:    _mmu->ss_dump(cout, (IU)node->id, o->a, _radix); break;
        case OP_LOAD:  _mmu->load(cout, (IU)node->id, o->n);            break;
        }
    } break;
    default: cout << "print type not supported: " << (int)node->gt; break;
    }
}

#define NEXTNODE(n) ((obuf_node*)((char*)&node->data[0] + node->sz))
__HOST__ void
AIO::flush() {
    obuf_node *node = (obuf_node*)_ostr->rdbuf();
    while (node->gt != GT_EMPTY) {          // 0
        print_node(node);
        node = NEXTNODE(node);
    }
    _ostr->clear();
}
///
/// private methods
///
#if T4_ENABLE_OBJ
__HOST__ void
AIO::_print_obj(DU v) {
    T4Base &b = _mmu->du2obj(v);
    switch (b.ttype) {
    case T4_VIEW:
    case T4_TENSOR: _print_tensor(v);
    case T4_MODEL:  _print_model(v);
    }
}
__HOST__ void
AIO::_print_vec(DU *d, int mi, int ri, int ci) {
    cout << "{";
    for (int i=0; i<ri; i++) {
        DU *dx = &d[i * ci];
        for (int c=0; c < ci; c++) {
            cout << (c>0 ? "_" : " ") << *dx++;
        }
    }
    int x = mi - ri;
    if (x > ri) cout << " ...";
    for (int i=(x > ri ? x : ri); i<mi; i++) {
        DU *dx = &d[i * ci];
        for (int c=0; c < ci; c++) {
            cout << (c>0 ? "_" : " ") << *dx++;
        }
    }
    cout << " }";
}
__HOST__ void
AIO::_print_mat(DU *d, int mi, int mj, int ri, int rj, int ci) {
    bool full = (mi * mj) <= _thres;
    int  xi   = full ? mi : ri;
    DU   *d0  = d;
    for (int j=0, j1=1; j<rj; j++, j1++, d0+=(mi * ci)) {
        _print_vec(d0, mi, xi, ci);
        cout << (j1==mj ? "" : "\n\t");
    }
    int y = full ? rj : mj - rj;
    if (y > rj) cout << "...\n\t";
    else y = rj;
    DU *d1 = (d + y * mi * ci);
    for (int j=y, j1=j+1; j<mj; j++, j1++, d1+=(mi * ci)) {
        _print_vec(d1, mi, xi, ci);
        cout << (j1==mj ? "" : "\n\t");
    }
}
__HOST__ void
AIO::_print_tensor(DU v) {
    auto   range = [this](int n) { return (n < _edge) ? n : _edge; };

    Tensor &t = (Tensor&)_mmu->du2obj(v);
    DU     *d = t.data;                     /// * short hand
    WARN("aio#print_tensor::T[%x]=%p data=%p\n", DU2X(v), &t, d);

    ios::fmtflags fmt0 = cout.flags();
    cout.flags(ios::showpos | ios::right | ios::fixed);
    cout << setprecision(_prec);
    switch (t.rank) {
    case 1: {
        cout << "vector[" << t.numel << "] = ";
        int ri = (t.numel < _thres) ? t.numel : range(t.numel);
        _print_vec(d, t.numel, ri, 1);
    } break;
    case 2: {
        cout << "matrix[" << t.H() << "," << t.W() << "] = {\n\t";
        int mj = t.H(), mi = t.W(), rj = range(mj),  ri = range(mi);
        _print_mat(d, mi, mj, ri, rj, 1);
        cout << " }";
    } break;
    case 4: {
        cout << "tensor["
             << t.N() << "," << t.H() << "," << t.W() << "," << t.C()
             << "] = {\n\t";
        int mj = t.H(), mi = t.W(), rj = range(mj),  ri = range(mi);
        _print_mat(d, mi, mj, ri, rj, t.C());
        cout << " }";
    } break;
    case 5: {
        cout << "tensor[" << t.parm << "]["
             << t.N() << "," << t.H() << "," << t.W() << "," << t.C()
             << "] = {...}";
    } break;        
    default: cout << "tensor rank=" << t.rank << " not supported";
    }
    cout << "\n";
    cout.flags(fmt0);
}
__HOST__ void
AIO::_print_model(DU v) {
    auto tinfo = [this](Tensor &t, int i, int fn) { ///> layer info
        cout << "[" << std::setw(3) << i << "] "
             << Model::nname(fn) << ":";
        _mmu->to_s(cout, t);
        int sz = t.grad[0] ? t.grad[0]->numel : 0;
        sz += t.grad[1] ? t.grad[1]->numel : 0;
        cout << ", #param=" << sz;
    };
    auto finfo = [this](Tensor **g) {
        for (int i=0; g[i] && i < 2; i++) {
            cout << " "; _mmu->to_s(cout, *g[i]);
        }
    };
    Model &m = (Model&)_mmu->du2obj(v);
    int   sz = m.numel;
    if (!m.is_model()) return;
    
    cout << "NN model[" << sz-1 << "/" << m.slots() << "]" << endl;
    for (int i = 1; i < sz; i++) {  /// skip root[0]
        Tensor &t = m[i];
        tinfo(t, i, (i==(sz-1)) ? 0 : t.grad_fn);
        if (_trace && t.grad_fn != L_NONE) finfo(t.grad);
        cout << endl;
    }
}
#endif // T4_ENABLE_OBJ
