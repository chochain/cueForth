#include "hip/hip_runtime.h"
/** -*- c++ -*- 
 * @File
 * @brief - tensorForth GUI - Image Viewer base class
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "vu.h"

__HOST__ int
Vu::build_texture() {
    ///
    /// rebuild GL texture in uchar4 format
    /// 
    int c   = dset.C;
    int bsz = X * Y * sizeof(uchar4);        ///< texture block size

    h_tex = (uchar4*)malloc(bsz);            ///< alloc block
    if (!h_tex) return -1;
    
    U8     *s = dset.h_data;
    uchar4 *t = h_tex;
    for (int i = 0; i < Y; i++) {
        for (int j = 0; j < X; j++, t++, s+=c) {
            t->x = *s;
            t->y = c < 2 ? *s   : *(s+1);
            t->z = c < 3 ? *s   : *(s+2);
            t->w = c < 4 ? 0xff : *(s+3);
        }
    }
    _bind_to_cuda();
    return 1;                                ///< has malloc
}

__HOST__ void
Vu::_bind_to_cuda() {
    int pitch = sizeof(uchar4) * X;
    
    hipChannelFormatDesc fmt = hipCreateChannelDesc<uchar4>();
    hipMallocArray(&d_ary, &fmt, X, Y); GPU_CHK();
    hipMemcpy2DToArray(
        d_ary, 0, 0, h_tex, pitch, pitch, Y, hipMemcpyHostToDevice);
    GPU_CHK();

    hipResourceDesc res;
    memset(&res, 0, sizeof(hipResourceDesc));
    res.resType           = hipResourceTypeArray;
    res.res.array.array   = d_ary;

    hipTextureDesc desc;
    memset(&desc, 0, sizeof(hipTextureDesc));
    desc.normalizedCoords = false;
    desc.filterMode       = hipFilterModeLinear;
    desc.addressMode[0]   = hipAddressModeWrap;
    desc.addressMode[1]   = hipAddressModeWrap;
    desc.readMode         = hipReadModeNormalizedFloat;
  
    hipCreateTextureObject(&cu_tex, &res, &desc, NULL);
    GPU_CHK();
}



