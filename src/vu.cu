#include "hip/hip_runtime.h"
/** -*- c++ -*- 
 * @File
 * @brief - tensorForth GUI - Image Viewer base class
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "vu.h"

Vu::Vu(const char *name) : fname(name) {}
Vu::~Vu() {
    if (!h_src) return;
    
    free(h_src);             /// * free host memory = stbi_image_free(h_src)
    hipFreeArray(d_ary);    /// * free device memory
    GPU_CHK();
}

Vu &Vu::setup() {
    int pitch = sizeof(uchar4) * W;
    
    hipChannelFormatDesc fmt = hipCreateChannelDesc<uchar4>();
    hipMallocArray(&d_ary, &fmt, W, H); GPU_CHK();
    hipMemcpy2DToArray(
        d_ary, 0, 0, h_src, pitch, pitch, H, hipMemcpyHostToDevice);
    GPU_CHK();

    hipResourceDesc res;
    memset(&res, 0, sizeof(hipResourceDesc));
    res.resType           = hipResourceTypeArray;
    res.res.array.array   = d_ary;

    hipTextureDesc desc;
    memset(&desc, 0, sizeof(hipTextureDesc));
    desc.normalizedCoords = false;
    desc.filterMode       = hipFilterModeLinear;
    desc.addressMode[0]   = hipAddressModeWrap;
    desc.addressMode[1]   = hipAddressModeWrap;
    desc.readMode         = hipReadModeNormalizedFloat;
  
    hipCreateTextureObject(&img, &res, &desc, NULL);
    GPU_CHK();

    return *this;
}

