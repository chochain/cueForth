#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @file
 * @brief TensorForth class - tensorForth main driver between CUDA and host
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 *
 * Benchmark: 1K*1K cycles on 3.2GHz AMD, Nvidia GTX1660
 *    + 19.0 msec - REALLY SLOW! Probably due to heavy branch divergence.
 *    + 21.1 msec - without NXT cache in nest() => branch is slow
 *    + 19.1 msec - without push/pop WP         => static ram access is fast
 *    + 20.3 msec - token indirect threading    => not that much worse but portable
 */
#include <iostream>          // cin, cout
#include <signal.h>
#include "ten4.h"            // wrapper

using namespace std;
///
/// tensorForth kernel - VM dispatcher
/// Note: 1 block per VM, thread 0 active only (wasteful?)
///
__GPU__ VM *d_vm_pool[VM_COUNT];

__KERN__ void
k_vm_init(System *sys, VM_Handle *pool) {
    int id = threadIdx.x;
    if (id >= VM_COUNT) return;
    
    VM *vm = pool[id].vm = new VM_TYPE(id, sys);
    vm->init();
    vm->state = id==0 ? HOLD : STOP;

    if (id==0) sys->mu->status();
}

__KERN__ void
k_vm_done(VM_Handle *pool) {
    int id = threadIdx.x;
    if (id >= VM_COUNT) return;
    
    delete pool[id].vm;
}

__KERN__ void
k_vm_exec(VM *vm) {
    __shared__ DU ss[T4_SS_SZ];      ///< shared mem for ss, rs (much faster)
    __shared__ DU rs[T4_RS_SZ];      ///< CC: make sure SZ < 32
    
    bool i   = threadIdx.x;
    bool t0  = i == 0 && blockIdx.x == 0;
    DU   *s0 = vm->ss.v;
    DU   *r0 = vm->rs.v;

    ///> copy stacks from global to shared mem
    for (int n = 0; n < T4_SS_SZ; n += WARP_SZ)
        if (i < vm->ss.idx) ss[n + i] = s0[n + i];
    for (int n = 0; n < T4_RS_SZ; n += WARP_SZ)
        if (i < vm->rs.idx) rs[n + i] = r0[n + i];
    __syncthreads();
    
    if (t0) {
        vm->ss.v = ss;
        vm->rs.v = rs;
        ///
        /// * enter ForthVM outer loop
        /// * Note: single-threaded, dynamic parallelism when needed
        ///
//    vm->outer();                               /// * enter VM outer loop
        /*
        DU ss0 = vm->ss[0];
        for (int i=0; i<10; i++) {
            vm->ss[0] = (DU)i;
        }
        vm->ss[0] = ss0;
        */
    }
    __syncthreads();
    
    ///> copy updated stacks back to global mem
    for (int n = 0; n < T4_SS_SZ; n += WARP_SZ)
        if (i < vm->ss.idx) s0[n + i] = ss[n + i];
    for (int n = 0; n < T4_SS_SZ; n += WARP_SZ)
        if (i < vm->rs.idx) r0[n + i] = rs[n + i];

    if (t0) {
        vm->ss.v = s0;                           /// * restore stack pointers
        vm->rs.v = r0;
    }
}

TensorForth::TensorForth(int device, int verbose) {
    ///
    /// set active device
    ///
    hipError_t err = hipSetDevice(device);
    if (err != hipSuccess) {
        cerr << "\nERR: failed to activate GPU " << device << "\n";
        exit(1);
    }
    ///
    /// query GPU shader clock rate
    ///
    int khz = 0;
    GPU_ERR(hipDeviceGetAttribute(&khz, hipDeviceAttributeClockRate, device));

    cout << "\\ GPU "  << device
         << " at "     << khz/1000 << "MHz"
         << ", dict["  << T4_DICT_SZ << "]"
         << ", pmem="  << T4_PMEM_SZ/1024 << "K"
         << ", ostor=" << T4_OSTORE_SZ/1024/1024 << "M"
         << ", vmss["  << T4_SS_SZ << "*" << VM_COUNT << "]"
         << ", vmrs["  << T4_RS_SZ << "*" << VM_COUNT << "]"
         << endl;
    ///
    /// allocate tensorForth system memory blocks
    ///
    sys = new System(cin, cout, khz, T4_VERBOSE);
    ///
    /// allocate VM handle pool
    ///
    MM_ALLOC(&vm_pool, sizeof(VM_Handle) * VM_COUNT);
}

__HOST__ void
TensorForth::setup() {
    for (int i=0; i < VM_COUNT; i++) {
        VM_Handle *h = &vm_pool[i];
        GPU_ERR(hipStreamCreate(&h->st));          /// * allocate stream
        GPU_ERR(hipEventCreate(&h->t0));           /// * allocate timers
        GPU_ERR(hipEventCreate(&h->t1));
    }
    k_vm_init<<<1, WARP(VM_COUNT)>>>(sys, vm_pool); /// * initialize all VMs
    GPU_CHK();
}

__HOST__ int
TensorForth::tally() {
    if (sys->trace() <= 1) return 0;
    
    int cnt[4] = { 0, 0, 0, 0};                    /// STOP, HOLD, QUERY, NEST
    for (int i=0; i < VM_COUNT; i++) {
        cnt[vm_pool[i].vm->state]++;
    }
    cout << "VM.state[STOP,HOLD,QUERY,NEST]=[";
    for (int i = 0; i < 4; i++) cout << " " << cnt[i];
    cout << " ]" << std::endl;
    
#if T4_VERBOSE > 1
    int m0 = (int)sys->mu->here() - 0x80;
    sys->db->mem_dump(m0 < 0 ? 0 : m0, 0x80);
#endif // T4_VERBOSE > 1
        
    return 1;
}

__HOST__ int
TensorForth::run() {
    int n_vm = 1;
    while (n_vm && sys->readline()) {
        n_vm = 0;
        for (int i=0; i<VM_COUNT; i++) {
            VM_Handle *h  = &vm_pool[i];
            VM        *vm = h->vm;
            if (vm->state == STOP) continue;
            n_vm++;
            
            hipEventRecord(h->t0, h->st);
            k_vm_exec<<<1, 1, 0, h->st>>>(vm);  // one block per VM
            GPU_CHK();
            hipEventRecord(h->t1, h->st);
            hipStreamWaitEvent(h->st, h->t1);  // CPU will wait here
            
            float dt;
            hipEventElapsedTime(&dt, h->t0, h->t1);
            
            switch (vm->state) {
            case HOLD:  VLOG1("%d} VM[%d] HOLD\n", vm->id, vm->id);   break;
#if T4_ENABLE_OBJ                
            case QUERY: if (!vm->compile) db->ss_dump(i, vm->ss.idx); break;
#endif // T4_ENABLE_OBJ                
            }
        }
        sys->flush();             /// * flush output buffer
        tally();                  /// * tally debug info
    }
    return 0;
}

__HOST__ void
TensorForth::teardown(int sig) {
    k_vm_done<<<1, WARP(VM_COUNT)>>>(vm_pool);
    GPU_CHK();
    for (int i=0; i < VM_COUNT; i++) {
        VM_Handle *h = &vm_pool[i];
        GPU_ERR(hipEventDestroy(h->t1));
        GPU_ERR(hipEventDestroy(h->t0));
        GPU_ERR(hipStreamDestroy(h->st));
    }
    MM_FREE(vm_pool);
}
///
/// main program
///
void sigsegv_handler(int sig, siginfo_t *si, void *arg) {
    cout << "Exception caught at: " << si->si_addr << endl;
    exit(1);
}

void sigtrap() {
    struct sigaction sa;
    memset(&sa, 0, sizeof(struct sigaction));
    sigemptyset(&sa.sa_mask);
    sa.sa_sigaction = sigsegv_handler;
    sa.sa_flags     = SA_SIGINFO;
    sigaction(SIGSEGV, &sa, NULL);
}

#include "opt.h"
int main(int argc, char**argv) {
    sigtrap();
    
    const string APP = string(T4_APP_NAME) + " " + T4_VERSION;
    Options opt;
    opt.parse(argc, argv);
    
    GPU_ERR(hipDeviceSetLimit(hipLimitStackSize, T4_PER_THREAD_STACK));
    // GPU_ERR(hipDeviceSetLimit(hipLimitMallocHeapSize, 16*1024*1024));
    
    if (opt.help) {
        opt.print_usage(std::cout);
        opt.check_devices(std::cout);
        cout << "\nRecommended GPU: " << opt.device_id << std::endl;
        return 0;
    }
    else opt.check_devices(std::cout, false);

    cout << APP << endl;

    TensorForth *f = new TensorForth(opt.device_id, opt.verbose);
    f->setup();
//    f->run();

    cout << APP << " done." << endl;
    f->teardown();

    return 0;
}


    
