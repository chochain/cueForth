#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @file
 * @brief TensorForth class - tensorForth main driver between CUDA and host
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 *
 * Benchmark: 1K*1K cycles on 3.2GHz AMD, Nvidia GTX1660
 *    + 19.0 msec - REALLY SLOW! Probably due to heavy branch divergence.
 *    + 21.1 msec - without NXT cache in nest() => branch is slow
 *    + 19.1 msec - without push/pop WP         => static ram access is fast
 *    + 20.3 msec - token indirect threading    => not that much worse but portable
 */
#include <iostream>          // cin, cout
#include <signal.h>

using namespace std;
#include "ten4.h"            // wrapper
///
/// tensorForth kernel - VM dispatcher
/// Note: 1 block per VM, thread 0 active only (wasteful?)
///
__KERN__ void
k_vm_exec(VM *vm) {
    extern __shared__ DU *ss;                   ///< shared mem for ss (much faster)
    
    DU *ss0 = vm->ss.v;                         ///< VM's data stack
    MEMCPY(ss, ss0, sizeof(DU) * T4_SS_SZ);     /// * copy stack into shared memory block
    vm->ss.v = ss;                              /// * redirect data stack to shared memory
    ///
    /// * enter ForthVM outer loop
    /// * Note: single-threaded, dynamic parallelism when needed
    ///
    vm->outer();                                /// * enter VM outer loop
        
    MEMCPY(ss0, ss, sizeof(DU) * T4_SS_SZ);     /// * copy updated stack back to global mem
    vm->ss.v = ss0;                             /// * restore stack ptr
}

TensorForth::TensorForth(int device, int verbose) {
    ///
    /// set active device
    ///
    hipError_t err = hipSetDevice(device);
    if (err != hipSuccess) {
        cerr << "\nERR: failed to activate GPU " << device << "\n";
        exit(1);
    }
    ///
    /// query GPU shader clock rate
    ///
    int khz = 0;
    GPU_ERR(hipDeviceGetAttribute(&khz, hipDeviceAttributeClockRate, device));

    cout << "\\  GPU " << device
         << " initialized at " << khz/1000 << "MHz"
         << ", dict["          << T4_DICT_SZ << "]"
         << ", pmem="          << T4_PMEM_SZ/1024 << "K"
         << ", vmss["          << T4_SS_SZ << "*" << VM_COUNT << "]"
         << ", tensor="        << T4_OSTORE_SZ/1024/1024 << "M"
         << endl;
    ///
    /// allocate tensorForth system memory blocks
    ///
    sys = new System(cin, cout, khz, verbose);
}

__HOST__ void
TensorForth::setup() {
    for (int i=0; i < VM_COUNT; i++) {
        VM_Handle *h = &vm_pool[i];
        MM_ALLOC(&h->vm, sizeof(VM_TYPE));
//        (h->vm = new VM_TYPE(i, sys))->init();     ///< instantiate VMs
        GPU_ERR(hipStreamCreate(&h->st));
        GPU_ERR(hipEventCreate(&h->t0));
        GPU_ERR(hipEventCreate(&h->t1));
    }
    vm_pool[0].vm->state = HOLD;
}

__HOST__ int
TensorForth::tally() {
    if (sys->trace() <= 1) return 0;
    
    int cnt[4] = { 0, 0, 0, 0};                    /// STOP, HOLD, QUERY, NEST
    for (int i=0; VM_COUNT; i++) {
        cnt[vm_pool[i].vm->state]++;
    }
    cout << "VM.state[STOP,HOLD,QUERY,NEST]=[";
    for (int i = 0; i < 4; i++) cout << " " << cnt[i];
    cout << " ]" << std::endl;
    
#if T4_MMU_DEBUG
    int m0 = (int)sys->mm->here() - 0x80;
    db->mem_dump(cout, m0 < 0 ? 0 : m0, 0x80);
#endif // T4_MMU_DEBUG
        
    return 1;
}

__HOST__ int
TensorForth::run() {
    int n_vm = 1;
    while (n_vm && sys->readline()) {
        n_vm = 0;
        for (int i=0; i<VM_COUNT; i++) {
            VM_Handle *h  = &vm_pool[i];
            VM        *vm = h->vm;
            if (vm->state == STOP) continue;
            n_vm++;
            
            hipEventRecord(h->t0, h->st);
            k_vm_exec<<<1, 1, T4_SS_SZ, h->st>>>(vm);
            GPU_CHK();
            hipEventRecord(h->t1, h->st);
            hipStreamWaitEvent(h->st, h->t1);       // CPU will wait here
            
            float dt;
            hipEventElapsedTime(&dt, h->t0, h->t1);
            
            switch (vm->state) {
            case HOLD:  VLOG1("%d} VM[%d] HOLD\n", vm->id, vm->id);   break;
#if T4_ENABLE_OBJ                
            case QUERY: if (!vm->compile) db->ss_dump(i, vm->ss.idx); break;
#endif // T4_ENABLE_OBJ                
            }
        }
        sys->flush();             /// * flush output buffer
        tally();                  /// * tally debug info
    }
    return 0;
}

__HOST__ void
TensorForth::teardown(int sig) {
    for (int i=0; i < VM_COUNT; i++) {
        VM_Handle *h = &vm_pool[i];
        
        GPU_ERR(hipEventDestroy(h->t1));
        GPU_ERR(hipEventDestroy(h->t0));
        GPU_ERR(hipStreamDestroy(h->st));
    }
}
///
/// main program
///
void sigsegv_handler(int sig, siginfo_t *si, void *arg) {
    cout << "Exception caught at: " << si->si_addr << endl;
    exit(1);
}

void sigtrap() {
    struct sigaction sa;
    memset(&sa, 0, sizeof(struct sigaction));
    sigemptyset(&sa.sa_mask);
    sa.sa_sigaction = sigsegv_handler;
    sa.sa_flags     = SA_SIGINFO;
    sigaction(SIGSEGV, &sa, NULL);
}

#include "opt.h"
int main(int argc, char**argv) {
    sigtrap();
    
    const string APP = string(T4_APP_NAME) + " " + T4_MAJOR_VER + "." + T4_MINOR_VER;
    Options opt;
    opt.parse(argc, argv);
    
    GPU_ERR(hipDeviceSetLimit(hipLimitStackSize, T4_PER_THREAD_STACK));
    // GPU_ERR(hipDeviceSetLimit(hipLimitMallocHeapSize, 16*1024*1024));
    
    if (opt.help) {
        opt.print_usage(std::cout);
        opt.check_devices(std::cout);
        cout << "\nRecommended GPU: " << opt.device_id << std::endl;
        return 0;
    }
    else opt.check_devices(std::cout, false);

    cout << APP << endl;

    TensorForth *f = new TensorForth(opt.device_id, opt.verbose);
    f->run();

    cout << APP << " done." << endl;
    f->teardown();

    return 0;
}


    
