#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @file - ten4.cu
 * @brief - tensorForth value definitions non-optimized
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 *
 * Benchmark: 1K*1K cycles on 3.2GHz AMD, Nvidia GTX1660
 *    + 19.0 sec - REALLY SLOW! Probably due to heavy branch divergence.
 *    + 21.1 sec - without NXT cache in nest() => branch is slow
 *    + 19.1 sec - without push/pop WP         => static ram access is fast
 *    + 20.3 sec - token indirect threading    => not that much worse but portable
 */
#include <iostream>          // cin, cout
#include <signal.h>
using namespace std;

#include "ten4_config.h"
#include "aio.h"             // CUDA async IO
#include "eforth.h"          // eForth core
#include "ten4.h"            // wrapper

#define MAJOR_VERSION        "2"
#define MINOR_VERSION        "0"

__GPU__ ForthVM *vm_pool[VM_MIN_COUNT];
///
/// instantiate VMs (threadIdx.x is vm_id)
///
__KERN__ void
ten4_init(int khz, Istream *istr, Ostream *ostr, MMU *mmu) {
    int i = threadIdx.x;
    if (i >= VM_MIN_COUNT) return;

    ForthVM *vm = vm_pool[i] = new ForthVM(khz, istr, ostr, mmu);  // instantiate VM
    vm->ss.init(mmu->vss(i), T4_SS_SZ);  // point data stack to managed memory block

    if (i==0) vm->init();                // initialize common dictionary (once only)
}
///
/// check VM status (using parallel reduction - overkill?)
///
__KERN__ void
ten4_busy(int *busy) {
    extern __shared__ bool b[];          // share memory for fast calc

    int i = threadIdx.x;
    b[i] = (i < VM_MIN_COUNT) ? vm_pool[i]->status==VM_RUN : 0;
    __syncthreads();

    for (int n=blockDim.x>>1; n>16; n>>=1) {
        if (i < n) b[i] |= b[i + n];
        __syncthreads();
    }
    if (i < 16) {                        // reduce spinning threads
        b[i] |= b[i + 16];
        b[i] |= b[i + 8];
        b[i] |= b[i + 4];
        b[i] |= b[i + 2];
        b[i] |= b[i + 1];
    }
    if (i==0) *busy = b[0];
}
///
/// tensorForth kernel - VM dispatcher
///
#include <stdio.h>
__KERN__ void
ten4_exec() {
    const char *st[] = {"READY", "RUN", "WAITING", "STOPPED"};
    extern __shared__ DU shared_ss[];
    if (threadIdx.x!=0) return;

    int      b   = blockIdx.x;
    ForthVM *vm  = vm_pool[b];
    DU      *ss  = &shared_ss[b * T4_SS_SZ];    // adjust stack pointer based on VM id
    DU      *ss0 = vm->ss.v;                    // capture VM data stack
    MEMCPY(ss, ss0, sizeof(DU) * T4_SS_SZ);     // copy stack into shared memory block
    vm->ss.v = ss;                              // redirect data stack to shared memory

    if (vm->status == VM_RUN) vm->outer();
    else {
        DEBUG("VM[%d] %s\n", blockIdx.x, st[vm->status]);
    }

    __syncthreads();
    MEMCPY(ss0, ss, sizeof(DU) * T4_SS_SZ);     // copy updated stack to managed memory
    vm->ss.v = ss0;                             // restore stack back to VM
}
///
/// clean up marked free tensors
///
__KERN__ void
ten4_sweep(MMU *mmu) {
//    mmu->lock();
    if (blockIdx.x ==0 && threadIdx.x == 0) {
        mmu->sweep();
    }
    __syncthreads();
//    mmu->unlock(); !!! DEAD LOCK now
}

TensorForth::TensorForth(int device, bool trace) {
    ///
    /// set active device
    ///
    hipError_t err = hipSetDevice(device);
    if (err != hipSuccess) {
        cerr << "\nERR: failed to activate GPU " << device << "\n";
        exit(1);
    }
    ///
    /// query GPU shader clock rate
    ///
    int khz = 0;
    hipDeviceGetAttribute(&khz, hipDeviceAttributeClockRate, device);
    GPU_CHK();
    
#if T4_VERBOSE
    cout << "\\  GPU " << device
         << " initialized at " << khz/1000 << "MHz"
         << ", dict["          << T4_DICT_SZ << "]"
         << ", vss["           << T4_SS_SZ << "*" << VM_MIN_COUNT << "]"
         << ", pmem="          << T4_PMEM_SZ/1024 << "K"
         << ", tensor="        << T4_TENSOR_SZ/1024/1024 << "M"
         << endl;
#endif // T4_VERBOSE
    ///
    /// allocate cuda memory blocks
    ///
    mmu = new MMU();                            ///> instantiate memory manager
    aio = new AIO(mmu, trace);                  ///> instantiate async IO manager
    hipMalloc((void**)&busy, sizeof(int));     ///> allocate managed busy flag
    GPU_CHK();
    ///
    /// instantiate virtual machines
    ///
    int t = WARP(VM_MIN_COUNT);                 ///> thread count = 32 modulo
    ten4_init<<<1, t>>>(khz, aio->istream(), aio->ostream(), mmu); // create VMs
    GPU_CHK();
}

TensorForth::~TensorForth() {
    delete aio;
    hipFree(busy);
    hipDeviceReset();
}

__HOST__ int
TensorForth::is_running() {
    int h_busy;
    //LOCK();                 // TODO: lock on vm_pool
    int t = WARP(VM_MIN_COUNT);
    ten4_busy<<<1, t, t * sizeof(bool)>>>(busy);
    GPU_SYNC();
    //UNLOCK();               // TODO:

    hipMemcpy(&h_busy, busy, sizeof(int), D2H);

    return h_busy;
}

#define VSS_SZ (sizeof(DU) * T4_SS_SZ * VM_MIN_COUNT)
__HOST__ int
TensorForth::run() {
    while (is_running()) {
        if (aio->readline()) {        // feed from host console to managed input buffer
            ten4_exec<<<1, 1, VSS_SZ>>>();
            GPU_CHK();                // hipDeviceSynchronize() and check error
            aio->flush();             // flush output buffer
            ten4_sweep<<<1, 1>>>(mmu);
        }
        yield();
#if MMU_DEBUG
        int m0 = (int)mmu->here() - 0x80;
        mmu->mem_dump(cout, m0 < 0 ? 0 : m0, 0x80);
#endif // MMU_DEBUG
    }
    return 0;
}

__HOST__ void
TensorForth::teardown(int sig) {}
///
/// main program
///
void sigsegv_handler(int sig, siginfo_t *si, void *arg) {
    cout << "Exception caught at: " << si->si_addr << endl;
    exit(1);
}

void sigtrap() {
    struct sigaction sa;
    memset(&sa, 0, sizeof(struct sigaction));
    sigemptyset(&sa.sa_mask);
    sa.sa_sigaction = sigsegv_handler;
    sa.sa_flags     = SA_SIGINFO;
    sigaction(SIGSEGV, &sa, NULL);
}

#include "opt.h"
int main(int argc, const char**argv) {
    sigtrap();
    
    const string APP = string(T4_APP_NAME) + " " + MAJOR_VERSION + "." + MINOR_VERSION;
    Options opt;
    opt.parse(argc, argv);
    
    if (opt.help) {
        opt.check_devices(std::cout);
        return 0;
    }

    cout << APP << endl;
    
    TensorForth *f = new TensorForth(opt.device_id);
    f->run();

    cout << APP << " done." << endl;
    f->teardown();

    return 0;
}


    
