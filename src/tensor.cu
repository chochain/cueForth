/**
 * @file
 * @brief tensorForth tensor class implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "tensor.h"
///
/// kernel matrix randomizer
///
__KERN__ void k_matrix_randomize(DU *mat, int nrow, int ncol, int seed=0)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < ncol && j < nrow) {
        int off = i + j * ncol;      /* row major */

        // Generate arbitrary elements.
        int const k = 16807;
        int const m = 16;
        DU v = DU(((off + seed) * k % m) - m / 2);

        mat[off] = v;
    }
}
///
/// GEMM kernel (used CUDA dynamic parallelism)
///     C = alpha * A x B + beta * C
///     where A = MxK, B = KxN, C = MxN
///
__KERN__ void k_GEMM(
    int M, int N, int K,
    DU *A, DU *B, DU *C,   /* MxK, KxN, MxN */
    DU alpha, DU beta)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < N && j < M) {
        DU acc = 0;
        for (int k = 0; k < K; ++k) {
            acc += A[k + j * K] * B[i + k * N];      /* row major */
        }
        C[i + j * N] = alpha * acc + beta * C[i + j * N];
    }
}
//
// GEMM test driver kernel code
//
__GPU__ Tensor&
Tensor::gemm(
    Tensor &A, Tensor &B, Tensor &C, DU alpha, DU beta) {
    int m = A.H(), k = A.W(), n = B.W();
    if (k != B.H() || m != C.H() || n != C.W()) {
        PRINTF("ERR: %s\n", "GEMM MxNxK dimension mismatched");
        return;
    }
    PRINTF("\nGEMM M=%d, N=%d, K=%d", m, n, k);
    
    dim3 block(16, 16), grid(
        (n + block.x - 1) / block.x,
        (m + block.y - 1) / block.y
    );
    k_GEMM<<<grid, block>>>(
        m, n, k,
        (DU*)A.data, (DU*)B.data, (DU*)C.data,
        alpha, beta);
    hipDeviceSynchronize();     // TODO: deprecated 11.6, use cooperative_groups.sync()
    return C;
}

__HOST__
Tensor::Tensor() :
    dsize(sizeof(DU)),
    size(0),
    rank(0),
    stride{0, 0, 0, 0},
    shape{0, 0, 0, 0} {}

__HOST__
Tensor::Tensor(U32 sz) :
    dsize(sizeof(DU)),
    size(sz),
    rank(1),
    stride{0, 0, 0, 0},
    shape{0, 0, 0, 0} {
    hipMallocManaged((void**)&data, (size_t)size * dsize);
    GPU_CHK();
    printf("tensor[%d] allocated\n", size);
}

__HOST__
Tensor::Tensor(U16 h, U16 w) :
    dsize(sizeof(DU)),
    size(h * w),
    rank(2),
    stride{1, 1, 0, 0},
    shape{h, w, 0, 0} {
    hipMallocManaged((void**)&data, (size_t)size * dsize);
    GPU_CHK();
    printf("matrix(%d,%d) allocated\n", shape[0], shape[1]);
}

__HOST__
Tensor::Tensor(U16 n, U16 h, U16 w, U16 c) :
    dsize(sizeof(DU)),
    size(n * h * w * c),
    rank(4),
    stride{1, 1, 1, 1},
    shape{h, w, n, c} {
    hipMallocManaged((void**)&data, (size_t)size * dsize);
    GPU_CHK();
    printf("tensor(%d,%d,%d,%d) allocated\n", shape[2], shape[0], shape[1], shape[3]);
}

__HOST__
Tensor::~Tensor()
{
    if (!data) return;
    hipFree((void*)data);
    switch (rank) {
    case 2: printf("matrix(%d,%d) freed\n", shape[0], shape[1]); break;
    case 4: printf("tensor(%d,%d,%d,%d) freed\n", shape[2], shape[0], shape[1], shape[3]); break;
    default: printf("~Tensor error: rank=%d\n", rank);
    }
}

__BOTH__ Tensor&
Tensor::reset(void *mptr, U32 sz) {
    dsize  = sizeof(DU);
    size   = sz;
    rank   = 1;
    memset(stride, 0, sizeof(stride));
    memset(shape,  0, sizeof(shape));
    attr   = 0;
    data   = (U8*)mptr;
    printf("tensor reset(%p, %d)\n", mptr, sz);
    return *this;
}

__BOTH__ Tensor&
Tensor::reshape(U32 sz) {
    if (sz == size) {
        reset(data, size);
        printf("tensor reshaped(%d)\n", size);
    }
    else {
        printf("reshape sz != size (%d != %d)\n", sz, size);
    }
    return *this;
}

__BOTH__ Tensor&
Tensor::reshape(U16 h, U16 w) {
    U32 sz = h * w;
    if (sz == size) {
        rank   = 2;
        U16 t[4] = {1, 1, 0, 0}; memcpy(stride, t, sizeof(t));
        U16 s[4] = {h, w, 0, 0}; memcpy(shape,  s, sizeof(s));
        printf("tensor reshaped(%d,%d)\n", shape[0], shape[1]);
    }
    else {
        printf("reshape sz != size (%d != %d)\n", sz, size);
    }
    return *this;
}

__BOTH__ Tensor&
Tensor::reshape(U16 n, U16 h, U16 w, U16 c) {
    U32 sz = n * h * w * c;
    if (sz == size) {
        rank   = 4;
        U16 t[4] = {1, 1, 1, 1}; memcpy(stride, t, sizeof(t));
        U16 s[4] = {h, w, n, c}; memcpy(shape,  s, sizeof(s));
        printf("tensor reshaped(%d,%d,%d,%d)\n", shape[2], shape[0], shape[1], shape[3]);
    }
    else {
        printf("reshape sz != size (%d != %d)\n", sz, size);
    }
    return *this;
}

__BOTH__ Tensor&
Tensor::fill(DU v) {
    DU  *d = (DU*)data;
    for (int i=0; i<size; i++) *d++ = v;
    return *this;
}

__BOTH__ Tensor&
Tensor::random(U32 seed) {
    int h = H();
    int w = W();
    dim3 block(16, 16), grid(
        (w + block.x - 1) / block.x,     /* row major */
        (h + block.y - 1) / block.y
    );
    k_matrix_randomize<<<grid, block>>>((DU*)data, h, w, seed);
    return *this;
}

