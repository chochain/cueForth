/**
 * @file
 * @brief tensorForth tensor class implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "tensor.h"
__KERN__ void k_matrix_randomize(DU *mat, int nrow, int ncol, int seed=0)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < ncol && j < nrow) {
        int off = i + j * ncol;      /* row major */

        // Generate arbitrary elements.
        int const k = 16807;
        int const m = 16;
        DU v = DU(((off + seed) * k % m) - m / 2);

        mat[off] = v;
    }
}

__HOST__
Tensor::Tensor() :
    dsize(sizeof(DU)),
    size(0),
    rank(0),
    stride{0, 0, 0, 0},
    shape{0, 0, 0, 0} {}

__HOST__
Tensor::Tensor(U32 sz) :
    dsize(sizeof(DU)),
    size(sz),
    rank(1),
    stride{0, 0, 0, 0},
    shape{0, 0, 0, 0} {
    hipMallocManaged((void**)&data, (size_t)size * dsize);
    GPU_CHK();
    printf("tensor[%d] allocated\n", size);
}

__HOST__
Tensor::Tensor(U16 h, U16 w) :
    dsize(sizeof(DU)),
    size(h * w),
    rank(2),
    stride{1, 1, 0, 0},
    shape{h, w, 0, 0} {
    hipMallocManaged((void**)&data, (size_t)size * dsize);
    GPU_CHK();
    printf("matrix(%d,%d) allocated\n", shape[0], shape[1]);
}

__HOST__
Tensor::Tensor(U16 n, U16 h, U16 w, U16 c) :
    dsize(sizeof(DU)),
    size(n * h * w * c),
    rank(4),
    stride{1, 1, 1, 1},
    shape{h, w, n, c} {
    hipMallocManaged((void**)&data, (size_t)size * dsize);
    GPU_CHK();
    printf("tensor(%d,%d,%d,%d) allocated\n", shape[2], shape[0], shape[1], shape[3]);
}

__HOST__
Tensor::~Tensor()
{
    if (!data) return;
    hipFree((void*)data);
    switch (rank) {
    case 2: printf("matrix(%d,%d) freed\n", shape[0], shape[1]); break;
    case 4: printf("tensor(%d,%d,%d,%d) freed\n", shape[2], shape[0], shape[1], shape[3]); break;
    default: printf("~Tensor error: rank=%d\n", rank);
    }
}

__BOTH__ Tensor&
Tensor::reset(void *mptr, U32 sz) {
    dsize  = sizeof(DU);
    size   = sz;
    rank   = 1;
    memset(stride, 0, sizeof(stride));
    memset(shape,  0, sizeof(shape));
    data   = (U8*)mptr;
    printf("tensor reset(%p, %d)\n", mptr, sz);
    return *this;
}

__BOTH__ Tensor&
Tensor::reshape(U16 h, U16 w) {
    U32 sz = h * w;
    if (sz == size) {
        rank   = 2;
        U16 t[4] = {1, 1, 0, 0}; memcpy(stride, t, sizeof(t));
        U16 s[4] = {h, w, 0, 0}; memcpy(shape,  s, sizeof(s));
        printf("tensor reshaped(%d,%d)\n", shape[0], shape[1]);
    }
    else {
        printf("reshape sz != size (%d != %d)\n", sz, size);
    }
    return *this;
}

__BOTH__ Tensor&
Tensor::reshape(U16 n, U16 h, U16 w, U16 c) {
    U32 sz = n * h * w * c;
    if (sz == size) {
        rank   = 4;
        U16 t[4] = {1, 1, 1, 1}; memcpy(stride, t, sizeof(t));
        U16 s[4] = {h, w, n, c}; memcpy(shape,  s, sizeof(s));
        printf("tensor reshaped(%d,%d,%d,%d)\n", shape[2], shape[0], shape[1], shape[3]);
    }
    else {
        printf("reshape sz != size (%d != %d)\n", sz, size);
    }
    return *this;
}

__BOTH__ Tensor&
Tensor::fill(DU v) {
    DU  *d = (DU*)data;
    for (int i=0; i<size; i++) *d++ = v;
    return *this;
}

__BOTH__ Tensor&
Tensor::random(int seed) {
    int h = H();
    int w = W();
    dim3 block(16, 16);
    dim3 grid(
        (w + block.x - 1) / block.x,     /* row major */
        (h + block.y - 1) / block.y
        );
    k_matrix_randomize<<<grid, block>>>((DU*)data, h, w, seed);
    return *this;
}
