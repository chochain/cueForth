#include "hip/hip_runtime.h"
#include <iomanip>          // setbase, setw, setfill
#include "ceforth.h"

__GPU__ int code_fence = 0, code_ip = 0;
///
/// Code class constructors
///
__GPU__ Code::Code(string n, fop fn, bool im) {
	name = n; token = code_fence++; immd = im; xt = fn;
}
__GPU__ Code::Code(string n, bool f)   { name = n; if (f) token = code_fence++; }
__GPU__ Code::Code(Code *c, DTYPE v)   { name = c->name; xt = c->xt; qf.push(v); }
__GPU__ Code::Code(Code *c, string s)  { name = c->name; xt = c->xt; if (s!=string()) literal = s;  }

__GPU__ Code*  Code::addcode(Code* w)  { pf.push(w);   return this; }
__GPU__ string Code::to_s()    { return name + " " + to_string(token) + (immd ? "*" : ""); }
__GPU__ string Code::see(int dp) {
    stringstream cout("");
    auto see_pf = [&cout](int dp, string s, ForthList<Code*> &a) {   // lambda for indentation and recursive dump
        int i = dp; cout << ENDL; while (i--) cout << "  "; cout << s;
        for (int i=0; i<a.size(); i++) cout << a[i]->see(dp + 1);
    };
    auto see_qf = [&cout](ForthList<DTYPE> &a) {
    	cout << " = "; for (int i=0; i<a.size(); i++) cout << a[i] << " ";
    };
    see_pf(dp, "[ " + to_s(), pf);
    if (pf1.size() > 0) see_pf(dp, "1--", pf1);
    if (pf2.size() > 0) see_pf(dp, "2--", pf2);
    if (qf.size()  > 0) see_qf(qf);
    cout << "]";
    return cout.str();
}
__GPU__ void Code::nest() {
    if (xt) xt(this);
    else {
    	int tmp = code_ip; code_ip = 0;
        for (Code* w : pf.v) { yield(); w->nest(); code_ip++; } /// run inner interpreter
        code_ip = tmp;
    }
}
///
/// ForthVM class constructor
///
__GPU__ ForthVM::ForthVM(istream &in, ostream &out) : cin(in), cout(out) {}
///
/// dictionary and input stream search functions
///
__GPU__ __INLINE__ DTYPE ForthVM::POP()         { DTYPE n = top; top = ss.pop(); return n; }
__GPU__ __INLINE__ DTYPE ForthVM::PUSH(DTYPE v) { ss.push(top); return top = v; }

/// search dictionary reversely
__GPU__ Code *ForthVM::find(string s) {
    for (int i = dict.size() - 1; i >= 0; --i) {
        if (s == dict[i]->name) return dict[i];
    }
    return NULL;
}
__GPU__ string ForthVM::next_idiom(char delim) {
    string s; delim ? getline(cin, s, delim) : cin >> s; return s;
}
__GPU__ void ForthVM::dot_r(int n, DTYPE v) {
    cout << setw(n) << setfill(' ') << v;
}
__GPU__ void ForthVM::ss_dump() {
    cout << " <"; for (DTYPE i : ss.v) { cout << i << " "; }
    cout << top << "> ok" << ENDL;
}
__GPU__ void ForthVM::words() {
    int i = 0;
    for (Code* w : dict.v) {
        if ((i++ % 10) == 0) { cout << ENDL; yield(); }
        cout << w->to_s() << " ";
    }
}
__GPU__ void ForthVM::call(Code *w) {
    int tmp = WP;                                       /// * setup call frame
    WP = w->token;
    w->nest(); 											/// * run inner interpreter recursively
/*
    try { w->nest(); }
    catch (exception& e) {
        string msg = e.what();                          /// * capture exception message
        if (msg != string()) cout << msg << ENDL;
    }
*/
    WP = tmp;                                           /// * restore call frame
    yield();
}
///
/// macros to reduce verbosity (but harder to single-step debug)
///
#define CODE(s, g) new Code(string(s), [this](Code *c){ g; })
#define IMMD(s, g) new Code(string(s), [this](Code *c){ g; }, true)
#define INT(f)         (static_cast<int>(f))
#define ALU(a, OP, b)  (INT(a) OP INT(b))
#define BOOL(f) ((f) ? -1 : 0)
///
/// dictionary initializer
///
__GPU__ void ForthVM::init() {
    const Code* prim[] = {       /// singleton, build once onl
    ///
    /// @defgroup Stack op
    /// @{
    CODE("dup",  PUSH(top)),
    CODE("drop", top = ss.pop()),
    CODE("over", PUSH(ss[-1])),
    CODE("swap", DTYPE n = ss.pop(); PUSH(n)),
    CODE("rot",  DTYPE n = ss.pop(); DTYPE m = ss.pop(); ss.push(n); PUSH(m)),
    CODE("-rot", DTYPE n = ss.pop(); DTYPE m = ss.pop(); PUSH(n); PUSH(m)),
    CODE("pick", DTYPE i = top; top = ss[-i]),
    CODE(">r",   rs.push(POP())),
    CODE("r>",   PUSH(rs.pop())),
    CODE("r@",   PUSH(rs[-1])),
    /// @}
    /// @defgroup Stack ops - double
    /// @{
    CODE("2dup", PUSH(ss[-1]); PUSH(ss[-1])),
    CODE("2drop",ss.pop(); top = ss.pop()),
    CODE("2over",PUSH(ss[-3]); PUSH(ss[-3])),
    CODE("2swap",
        DTYPE n = ss.pop(); DTYPE m = ss.pop(); DTYPE l = ss.pop();
        ss.push(n); PUSH(l); PUSH(m)),
    /// @}
    /// @defgroup ALU ops
    /// @{
    CODE("+",    top += ss.pop()),
    CODE("-",    top =  ss.pop() - top),
    CODE("*",    top *= ss.pop()),
    CODE("/",    top =  ss.pop() / top),
    CODE("mod",  top = ALU(ss.pop(), %, top)),
    CODE("*/",   top = ss.pop() * ss.pop() / top),
    CODE("*/mod",
        int n = INT(ss.pop() * ss.pop());
        int t = INT(top);
        ss.push(n % t); top = (n / t)),
    CODE("and",  top = ALU(ss.pop(), &, top)),
    CODE("or",   top = ALU(ss.pop(), |, top)),
    CODE("xor",  top = ALU(ss.pop(), ^, top)),
    CODE("negate", top = -top),
    CODE("abs",  top = abs(top)),
    /// @}
    /// @defgroup Logic ops
    /// @{
    CODE("0= ",  top = BOOL(top == 0)),
    CODE("0<",   top = BOOL(top <  0)),
    CODE("0>",   top = BOOL(top >  0)),
    CODE("=",    top = BOOL(ss.pop() == top)),
    CODE(">",    top = BOOL(ss.pop() >  top)),
    CODE("<",    top = BOOL(ss.pop() <  top)),
    CODE("<>",   top = BOOL(ss.pop() != top)),
    CODE(">=",   top = BOOL(ss.pop() >= top)),
    CODE("<=",   top = BOOL(ss.pop() <= top)),
    /// @}
    /// @defgroup IO ops
    /// @{
    CODE("base@",   PUSH(base)),
    CODE("base!",   cout << setbase(base = POP())),
    CODE("hex",     cout << setbase(base = 16)),
    CODE("decimal", cout << setbase(base = 10)),
    CODE("cr",      cout << ENDL),
    CODE(".",       cout << POP() << " "),
    CODE(".r",      int n = INT(POP()); dot_r(n, POP())),
    CODE("u.r",     int n = INT(POP()); dot_r(n, abs(POP()))),
    CODE(".f",      int n = INT(POP()); cout << setprecision(n) << POP()),
    CODE("key",     PUSH(next_idiom()[0])),
    CODE("emit",    char b = (char)POP(); cout << b),
    CODE("space",   cout << " "),
    CODE("spaces",  for (int n = INT(POP()), i = 0; i < n; i++) cout << " "),
    /// @}
    /// @defgroup Literal ops
    /// @{
    CODE("dotstr",  cout << c->literal),
    CODE("dolit",   PUSH(c->qf[0])),
    CODE("dovar",   PUSH(c->token)),
    CODE("[",       compile = false),
    CODE("]",       compile = true),
    IMMD("(",       next_idiom(')')),
    IMMD(".(",      cout << next_idiom(')')),
    CODE("\\",      cout << next_idiom('\n')),
    CODE("$\"",
        string s = next_idiom('"').substr(1);
        dict[-1]->addcode(new Code(find("dovar"), s))),
    IMMD(".\"",
        string s = next_idiom('"').substr(1);
        dict[-1]->addcode(new Code(find("dotstr"), s))),
    /// @}
    /// @defgroup Branching ops
    /// @brief - if...then, if...else...then
    /// @{
    IMMD("bran",
        bool f = POP() != 0;                        // check flag
        for (Code* w : (f ? c->pf.v : c->pf1.v)) call(w)),
    IMMD("if",
        dict[-1]->addcode(new Code(find("bran")));
        dict.push(new Code("temp"))),               // use last cell of dictionay as scratch pad
    IMMD("else",
        Code *temp = dict[-1]; Code *last = dict[-2]->pf[-1];
        last->pf.merge(temp->pf);
        temp->pf.clear();
        last->stage = 1),
    IMMD("then",
        Code *temp = dict[-1]; Code *last = dict[-2]->pf[-1];
        if (last->stage == 0) {                     // if...then
            last->pf.merge(temp->pf);
            dict.pop();
        }
        else {                                      // if...else...then, or
             last->pf1.merge(temp->pf);             // for...aft...then...next
             if (last->stage == 1) dict.pop();
             else temp->pf.clear();
        }),
    /// @}
    /// @defgroup Loops
    /// @brief  - begin...again, begin...f until, begin...f while...repeat
    /// @{
    CODE("loop",
        while (true) {
            for (Code* w : c->pf.v) call(w);                       // begin...
            int f = INT(top);
            if (c->stage == 0 && (top = ss.pop(), f != 0)) break;  // ...until
            if (c->stage == 1) continue;                           // ...again
            if (c->stage == 2 && (top = ss.pop(), f == 0)) break;  // while...repeat
            for (Code* w : c->pf1.v) call(w);
        }),
    IMMD("begin",
        dict[-1]->addcode(new Code(find("loop")));
        dict.push(new Code("temp"))),
    IMMD("while",
        Code *last = dict[-2]->pf[-1]; Code *temp = dict[-1];
        last->pf.merge(temp->pf);
        temp->pf.clear(); last->stage = 2),
    IMMD("repeat",
        Code *last = dict[-2]->pf[-1]; Code *temp = dict[-1];
        last->pf1.merge(temp->pf); dict.pop()),
    IMMD("again",
        Code *last = dict[-2]->pf[-1]; Code *temp = dict[-1];
        last->pf.merge(temp->pf);
        last->stage = 1; dict.pop()),
    IMMD("until",
        Code *last = dict[-2]->pf[-1]; Code *temp = dict[-1];
        last->pf.merge(temp->pf); dict.pop()),
    /// @}
    /// @defgrouop For loops
    /// @brief  - for...next, for...aft...then...next
    /// @{
    CODE("cycle",
        do { for (Code* w : c->pf.v) call(w); }
        while (c->stage == 0 && rs.dec_i() >= 0);    // for...next only
        while (c->stage > 0) {                       // aft
            for (Code* w : c->pf2.v) call(w);        // then...next
            if (rs.dec_i() < 0) break;
            for (Code* w : c->pf1.v) call(w);        // aft...then
        }
        rs.pop()),
    IMMD("for",
        dict[-1]->addcode(new Code(find(">r")));
        dict[-1]->addcode(new Code(find("cycle")));
        dict.push(new Code("temp"))),
    IMMD("aft",
        Code *last = dict[-2]->pf[-1]; Code *temp = dict[-1];
        last->pf.merge(temp->pf);
        temp->pf.clear(); last->stage = 3),
    IMMD("next",
        Code *last = dict[-2]->pf[-1]; Code *temp = dict[-1];
        if (last->stage == 0) last->pf.merge(temp->pf);
        else last->pf2.merge(temp->pf); dict.pop()),
    /// @}
    /// @defgrouop Compiler ops
    /// @{
    CODE("exec", int n = INT(top); call(dict[n])),
    CODE(":",
        dict.push(new Code(next_idiom(), true));    // create new word
        compile = true),
    IMMD(";", compile = false),
    CODE("variable",
        dict.push(new Code(next_idiom(), true));
        Code *last = dict[-1]->addcode(new Code(find("dovar"), 0));
        last->pf[0]->token = last->token),
    CODE("constant",
        dict.push(new Code(next_idiom(), true));
        Code *last = dict[-1]->addcode(new Code(find("dolit"), POP()));
        last->pf[0]->token = last->token),
    CODE("@",      int w = INT(POP()); PUSH(dict[w]->pf[0]->qf[0])),         // w -- n
    CODE("!",      int w = INT(POP()); dict[w]->pf[0]->qf[0] = POP()),       // n w --
    CODE("+!",     int w = INT(POP()); dict[w]->pf[0]->qf[0] += POP()),      // n w --
    CODE("?",      int w = INT(POP()); cout << dict[w]->pf[0]->qf[0] << " "),// w --
    CODE("array@", int a = INT(POP()); PUSH(dict[INT(POP())]->pf[0]->qf[a])),     // w a -- n
    CODE("array!", int a = INT(POP()); int w = INT(POP());  dict[w]->pf[0]->qf[a] = POP()),   // n w a --
    CODE("allot",                                           // n --
        for (int n = INT(POP()), i = 0; i < n; i++) dict[-1]->pf[0]->qf.push(DVAL)),
    CODE(",",      dict[-1]->pf[0]->qf.push(POP())),
    /// @}
    /// @defgroup metacompiler
    /// @{
    CODE("create",
        dict.push(new Code(next_idiom(), true));            // create a new word
        Code *last = dict[-1]->addcode(new Code(find("dovar"), DVAL));
        last->pf[0]->token = last->token;
        last->pf[0]->qf.clear()),
    CODE("does",
        ForthList<Code*> &src = dict[WP]->pf;               // source word : xx create...does...;
        int i = code_ip; int n = src.size();
        while (++i < n) dict[-1]->pf.push(src[i])),         // copy words after "does" to new the word
    CODE("to",                                              // n -- , compile only
        Code *tgt = find(next_idiom());
        if (tgt) tgt->pf[0]->qf[0] = POP()),                // update constant
    CODE("is",                                              // w -- , execute only
        Code *tgt = find(next_idiom());
        if (tgt) {
            tgt->pf.clear();
            tgt->pf.merge(dict[POP()]->pf);
        }),
    CODE("[to]",
        ForthList<Code*> &src = dict[WP]->pf;               // source word : xx create...does...;
        src[++code_ip]->pf[0]->qf[0] = POP()),              // change the following constant
    /// @}
    /// @defgroup Debug ops
    /// @{
    CODE("bye",   exit(0)),
    CODE("here",  PUSH(dict[-1]->token)),
    CODE("words", words()),
    CODE(".s",    ss_dump()),
    CODE("'",     Code *w = find(next_idiom()); PUSH(w->token)),
    CODE("see",
        Code *w = find(next_idiom());
        if (w) cout << w->see(0) << ENDL),
    CODE("forget",
        Code *w = find(next_idiom());
         if (w == NULL) return;
         dict.clear(code_fence=max(w->token, find("boot")->token + 1))),
    CODE("boot", dict.clear(code_fence=find("boot")->token + 1))
    /// @}
    };
    dict.merge((Code*)prim, code_fence);       /// * populate dictionary
}
///
/// ForthVM Outer interpreter
///
__GPU__ void ForthVM::outer() {
    string idiom;
    while (cin >> idiom) {
        //Serial.print(idiom.c_str()); Serial.print("=>");
    	//printf("%s=>", idiom.c_str());
        Code *w = find(idiom);                          /// * search through dictionary
        if (w) {                                        /// * word found?
            //Serial.println(w->to_s().c_str());
            //printf("%s\n", w->to_s().c_str());
            if (compile && !w->immd)                    /// * in compile mode?
                dict[-1]->addcode(w);                   /// * add to colon word
            else call(w);                               /// * execute forth word
            continue;
        }
        // try as a number
        char *p;
        int n = (int)strtol(idiom.c_str(), &p, base);
        //Serial.println(n, base);
        //printf("%d\n", n);
        if (*p != '\0') {                           /// * not number
            cout << idiom << "? " << ENDL;          ///> display error prompt
            compile = false;                        ///> reset to interpreter mode
            getline(cin, idiom, '\n');              ///> skip the entire line
            continue;
        }
        // is a number
        if (compile)                           /// * a number in compile mode?
            dict[-1]->addcode(new Code(find("dolit"), n)); ///> add to current word
        else PUSH(n);                           	///> or, add value onto data stack
    }
    if (!compile) ss_dump();  /// * dump stack and display ok prompt
}

/// main program
__KERN__ int eforth_init(U8 *cin, U8 *cout) {
	if (threadId.x!=0 || blockId.x!=0) return 0;
    string cmd;

    ForthVM *vm = new ForthVM(forth_in, forth_out);		// create FVM instance
    vm->init();                                 		// initialize dictionary

    while (getline(cin, cmd)) {							// fetch user input
    	//printf("cmd=<%s>\n", line.c_str());
    	forth_in.clear();								// clear any input stream error bit
    	forth_in.str(cmd);								// send command to FVM
        vm->outer();									// execute outer interpreter
        cout << forth_out.str();						// send VM result to output
        forth_out.str(string());						// clear output buffer
    }
    cout << "done!" << ENDL;
    return 0;
}
