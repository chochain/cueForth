/*! @file
  @brief
  cueForth value definitions non-optimized
*/
#include <iostream>
#include "cuef.h"

// forward declaration for implementation
extern "C" __KERN__ void mmu_init(void *ptr, U32 sz);
extern "C" __KERN__ void eforth_init(U8 *cin, U8 *cout);

__KERN__ void eforth_init(U8 *cin, U8 *cout) {
	//if (threadId.x!=0 || blockId.x!=0) return;
    //string cmd;
	return;
}

CueForth::CueForth(istream &in, ostream &out) : cin(in), cout(out) {}

__HOST__ void*
CueForth::_malloc(int sz, int type)
{
	void *mem;

	// TODO: to add texture memory
	switch (type) {
	case 0: 	hipMalloc(&mem, sz); break;			// allocate device memory
	default: 	hipMallocManaged(&mem, sz);			// managed (i.e. paged) memory
	}
    if (hipSuccess != hipGetLastError()) return NULL;

    return mem;
}

__HOST__ void
CueForth::_free(void *mem) {
	hipFree(mem);
}

__HOST__ int
CueForth::setup(int step, int trace)
{
	hipDeviceReset();

	PRINTF("cueForth initializing...");

	heap = (U8*)_malloc(CUEF_HEAP_SIZE, 1);					// allocate main block (i.e. RAM)
	if (!heap)  return -10;
	ibuf = (U8*)_malloc(CUEF_IBUF_SIZE, 1);					// allocate main block (i.e. RAM)
	if (!ibuf)  return -11;
	obuf = (U8*)_malloc(CUEF_OBUF_SIZE, 1);					// allocate output buffer
	if (!obuf)  return -12;

	//mmu_init<<<1,1>>>(mem, CUEF_HEAP_SIZE);				// setup memory management
	eforth_init<<<1,1>>>(ibuf, obuf);						// setup basic classes	(TODO: => ROM)
	GPU_SYNC();

    U32 sz0, sz1;
	hipDeviceGetLimit((size_t *)&sz0, hipLimitStackSize);
	hipDeviceSetLimit(hipLimitStackSize, (size_t)sz0*4);
	hipDeviceGetLimit((size_t *)&sz1, hipLimitStackSize);

	PRINTF("cueForth initialized, ready to go...");

    return 0;
}

__HOST__ int
CueForth::run()
{
	PRINTF("cuef session starting...");
	// kick up main loop until all VM are done
    string idiom;
	while (cin >> idiom) {
        printf("%s=>", idiom.c_str());
    }

	PRINTF("cuef session completed.");

	return 0;
}

__HOST__ void
CueForth::teardown(int sig)
{
	if (obuf) _free(obuf);
	if (ibuf) _free(ibuf);
	if (heap) _free(heap);
	hipDeviceReset();
}

int main(int argc, char**argv) {
	CueForth *f = new CueForth(cin, cout);
	f->setup();
	cout << "cueForth starting..." << ENDL;
	f->run();
	f->teardown();
    cout << "done!" << ENDL;
}

