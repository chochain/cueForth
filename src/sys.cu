/** -*- c++ -*-
 * @file
 * @brief System class - tensorForth System interface implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "sys.h"
#include "ldr/loader.h"
///
/// random number generator setup
/// Note: kept here because curandStates stays in CUDA memory
///
__KERN__ void
k_rand_init(hiprandState *st, U64 seed) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, x, 0, &st[x]);
}

__KERN__ void
k_rand(DU *mat, int sz, DU bias, DU scale, hiprandState *st, rand_opt ntype) {
    int tx = threadIdx.x;             ///< thread idx
    int n  = (sz / blockDim.x) + 1;   ///< loop counter
    
    hiprandState s = st[tx];           /// * cache state into local register
    for (int i=0, x=tx; i<n; i++, x+=blockDim.x) {  /// * scroll through pages
        if (x < sz) {
            mat[x]= scale * (
                bias + (ntype==NORMAL ? hiprand_normal(&s) : hiprand_uniform(&s))
                );
        }
    }
    st[tx] = s;                      /// * copy state back to global memory
}
///
/// Forth Virtual Machine operational macros to reduce verbosity
///
__HOST__
System::System(h_istr &i, h_ostr &o, int khz, int verbo)
    : _khz(khz), _istr(new Istream()), _ostr(new Ostream()), _trace(verbo) {
    mu = new MMU();                  ///> instantiate memory manager
    io = new AIO(i, o, verbo);       ///> instantiate async IO manager
    db = new Debug(mu, io);          ///> tracing instrumentation
        
#if (T4_ENABLE_OBJ && T4_ENABLE_NN)
    Loader::init(verbo);
#endif
    ///
    ///> setup randomizer
    ///
    MM_ALLOC(&_seed, sizeof(hiprandState) * T4_RAND_SZ);
    k_rand_init<<<1, T4_RAND_SZ>>>(_seed, time(NULL));  /// serialized randomizer
    GPU_CHK();
    
    INFO("\\ System OK\n");
}

System::~System() {
    GPU_SYNC();
    
    MM_FREE(_seed);
    delete io;
    delete db;
    delete mu;
    INFO("\\ System freed\n");
}

__GPU__ void
System::rand(DU *d, U64 sz, rand_opt n, DU bias, DU scale) {
//    DEBUG("mmu#random(T%d) numel=%ld bias=%.2f, scale=%.2f\n",
//          t.rank, t.numel, bias, scale);
//    k_rand<<<1, T4_RAND_SZ>>>(t.data, t.numel, bias, scale, _seed, ntype);
    k_rand<<<1, T4_RAND_SZ>>>(d, sz, bias, scale, _seed, n);
}
///
///> feed device input stream with a line from host input
///
#include <string.h>
__HOST__ int
System::readline() {
    _istr->clear();                          /// * clear device inpugt stream
    char *tib = _istr->rdbuf();              ///< device input buffer
    io->fin.getline(tib, T4_IBUF_SZ, '\n');  /// * feed input buffer
    return !io->fin.eof();                   /// * end of file
}

#define NEXT_EVENT(n) ((io_event*)((char*)&ev->data[0] + ev->sz))

__HOST__ io_event*
System::process_event(io_event *ev) {
    GPU_SYNC();                     /// * make sure data is completely written

    char   *v    = (char*)ev->data; ///< fetch payload in buffered print node
    h_ostr &fout = io->fout;        ///< host output stream
    switch (ev->gt) {
    case GT_INT:   fout << (*(S32*)v);                 break;
    case GT_U32:   fout << static_cast<U32>(*(U32*)v); break;
    case GT_FLOAT: fout << (*(F32*)v);                 break;
    case GT_STR:   fout << v;                          break;
    case GT_FMT:   {
        obuf_fmt *f = (obuf_fmt*)v;
        DEBUG("FMT: b=%d, w=%d, p=%d, f='%c'\n", f->base, f->width, f->prec, f->fill);
        fout << std::setbase(f->base)
             << std::setw(f->width)
             << std::setprecision(f->prec ? f->prec : -1)
             << std::setfill((char)f->fill);
    } break;
    case GT_OBJ: io->print(mmu->du2obj(*(DU*)v));      break;
    case GT_OPX: {
        _opx *o = (_opx*)v;
        DEBUG("OP=%d, m=%d, i=%d, n=0x%08x=%f\n", o->op, o->m, o->i, DU2X(o->n), o->n);
        switch (o->op) {
        case OP_DICT:  db->dict_dump();                    break;
        case OP_WORDS: db->words();                        break;
        case OP_SEE:   db->see((IU)o->i, (int)o->m);       break;
        case OP_DUMP:  db->mem_dump((IU)o->i, UINT(o->n)); break;
        case OP_SS:    db->ss_dump((IU)o->i>>10, (int)o->i&0x3ff, o->n, (int)o->m); break;
#if T4_ENABLE_OBJ // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
        case OP_TSAVE:
            ev = NEXT_EVENT(ev);
            io->_tsave((Tensor&)mu->du2obj(o->n), (char*)ev->data, o->fam);
            break;
#if T4_ENABLE_NN  //==========================================================
        case OP_DATA:
            ev = NEXT_EVENT(ev);                            ///< get dataset repo name
            io->_dsfetch(o->n, (char*)ev->data, o->fam);    /// * fetch first batch
            break;
        case OP_FETCH: io->_dsfetch(o->n, NULL, o->fam); break;  /// * fetch/rewind dataset batch
        case OP_NSAVE:
            ev = NEXT_EVENT(ev);                            ///< get dataset repo name
            io->_nsave((Tensor&)mu->du2obj(o->n), (char*)ev->data, o->fam);
            break;
        case OP_NLOAD:
            ev = NEXT_EVENT(ev);
            io->_nload(o->n, (char*)ev->data, o->fam);
            break;
#endif // T4_ENABLE_NN =======================================================
#endif // T4_ENABLE_OBJ ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
        }
    } break;
    default: fout << "print type not supported: " << (int)ev->gt; break;
    }
    return NEXT_EVENT(ev);
}

__HOST__ void
System::flush() {
    io_event *e = (io_event*)_ostr->rdbuf();
    while (e->gt != GT_EMPTY) {          // 0
        e = process_event(e);
    }
    _ostr->clear();
}


