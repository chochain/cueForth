#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - Neural Network Model implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "model.h"

#if T4_ENABLE_OBJ
///
/// Row convolution filter
///
template<int TS, int KS, int CS> ///> tile size, kernel size, 1:grey, 3:RGB
__KERN__ void k_conv(
    DU *I, DU *F, DU *B, DU *O,  ///> input A[HxW], F[KxK] kernel B[C] bias, output C[HxW]
    int H, int W                 ///< HWC
    ) {
    __shared__ DU d[CS][T4_WARP_SZ * T4_WARP_SZ];  ///< shared memory [3][16x16]
    
    const int tx = threadIdx.x, j0 = tx + blockIdx.x * TS;
    const int ty = threadIdx.y, i0 = ty + blockIdx.y * TS;
    const int tz = threadIdx.z, c0 = tz + blockIdx.z * blockDim.z;
    const int i  = i0 - int(KS / 2);         ///< transfrom to input coordinate
    const int j  = j0 - int(KS / 2);
    const int z0 = c0 + (j0 + i0 * W) * CS;  ///< output array index

    for (int c = 0; c < CS; c++) {           /// channel depth
        d[c][tx + ty * T4_WARP_SZ] =         ///< shared memory with zero padding
            (i >= 0 && i < H && j >= 0 && j < W)
            ? I[c + (j + i * W) * CS] : DU0;
    }
    __syncthreads();
    ///
    /// sum of element-wise multiplication
    ///
    if (tx < TS && ty < TS) {                /// * within tile [12x12]
        DU sum = DU0;
        for (int c = 0; c < CS; c++) {               /// 3D filter
            for (int y = 0; y < KS; y++) {           /// * process one cell
                int d0 = tx + (y + ty) * T4_WARP_SZ; ///< offset to smem block
                int b0 = y * KS;
                for (int x = 0; x < KS; x++) {
                    sum += F[x + b0] * d[c][x + d0]; 
                }
            }
        }
        if (i0 < H && j0 < W && c0 < CS) {  /// * update C[i][j]
            O[z0] = sum + B[c0];            /// * update output matrix with bias
        }
    }
}

template<int KS>                           /// kernel size
__KERN__ void k_pooling(
    DU *I, DU *O,
    int H, int W, int C,                   /// HWC
    t4_layer op
    ) {
    const int j0 = threadIdx.x + blockIdx.x * blockDim.x;
    const int i0 = threadIdx.y + blockIdx.y * blockDim.y;
    const int c0 = threadIdx.z + blockIdx.z * blockDim.z;
    const int z0 = j0 + i0 * W;            ///< output array index
    const int z1 = j0 + i0 * W * KS;       ///< input array index 
    
    if (i0 < H && j0 < W && c0 < C) {
        DU *d  = &I[c0 + z1 * KS * C];
        DU2 v  = op==L_AVGPOOL ? DU0 : *d;
        for (int y = 0; y < KS; y++) {
            for (int x = 0; x < KS; x++) {
                DU dx = *d;
                switch (op) {
                case L_MAXPOOL: if (dx > v) v = dx; break;
                case L_AVGPOOL: v += dx;            break;
                case L_MINPOOL: if (dx < v) v = dx; break;
                }
                d += C;                   
            }
            d += (W - 1) * KS * C;
        }
        O[c0 + z0 * C] = op==L_AVGPOOL ? v / (KS * KS) : v;
    }
}

__KERN__ void k_relu(
    DU *I, DU *O,
    int H, int W, int C                    ///< HWC
    ) {
    const int j0 = threadIdx.x + blockIdx.x * blockDim.x;
    const int i0 = threadIdx.y + blockIdx.y * blockDim.y;
    const int c0 = threadIdx.z + blockIdx.z * blockDim.z;
    const int z0 = c0 + (i0 + j0 * W) * C;
    
    if (i0 < H && j0 < W && c0 < C) {
        O[z0] = (I[z0] >= DU0) ? I[z0] : DU0;
    }
}

__HOST__ const char*
Model::nname(int i) {               ///< network layer name
    static const char *name[] = {   /// double check with t4_layer
    "output ", "conv2d ", "linear ", "flatten", "relu   ",
    "tanh   ", "sigmoid", "softmax", "maxpool", "avgpool",
    "minpool", "dropout"
    };
    return name[i];
}
///
/// NN layer factory
///
__GPU__ Model&
Model::add(t4_layer fn, U16 n, DU bias, U16 *opt) {
    Tensor &in = (Tensor&)_mmu->du2obj(data[numel - 1]);
    if (!autograd || in.grad_fn != L_NONE) return *this;

    switch(fn) {
    case L_CONV:    _iconv(in, n, bias, opt);   break;
    case L_LINEAR:  _ilinear(in, n, bias);      break;
    case L_FLATTEN: _iflatten(in);              break;
    case L_RELU:
    case L_TANH:
    case L_SIGMOID: _icopy(in);                 break;
    case L_SOFTMAX: _isoftmax(in);              break;
    case L_MAXPOOL:
    case L_AVGPOOL:
    case L_MINPOOL: _ipooling(in, n);           break;
    case L_DROPOUT: _idropout(in, n);           break;
    }
    in.grad_fn = fn;
    return *this;
}

__GPU__ Model&
Model::forward(Tensor &input) {
    static const char *name[] = {   /// double check with t4_layer
    "output ", "conv2d ", "linear ", "flatten", "relu   ",
    "tanh   ", "sigmoid", "softmax", "maxpool", "avgpool",
    "minpool", "dropout"
    };
    Tensor &in = (*this)[1];
    if (!in.is_same_shape(input)) {
        ERROR("Model#forward input dim?\n");
        return *this;
    }
    Tensor::copy(input, in);       /// * feed input into model
    ///
    /// cascade execution layer by layer
    /// TODO: model execution becomes a superscalar pipeline
    ///
    for (int i = 2; i < numel; i++) {
        printf("%2d> %s [%d,%d,%d] p=%d =>",
               i-1, name[in.grad_fn], in.H(), in.W(), in.C(), in.parm); 
        Tensor &out = (*this)[i];
        _fstep(in, out);
        in = out;
        printf("\n");
    }
    return *this;
}
__GPU__ Model&
Model::backprop(Tensor &output) {
    return *this;
}
/// ========================================================================
/// private methods 
///
#define TILE3    (T4_WARP_SZ - 3 + 1)      /** 14 */
#define TILE5    (T4_WARP_SZ - 5 + 1)      /** 12 */

__GPU__ void
Model::_fstep(Tensor &in, Tensor &out) {
    DU   *da = in.data, *dc = out.data;              ///< input, output data
    int  H = out.H(), W = out.W(), C = out.C();      ///< HWC
    dim3 blk(T4_WARP_SZ, T4_WARP_SZ, C), grd(        ///< GPU warp size setup
        (W + blk.x - 1) / blk.x,
        (H + blk.y - 1) / blk.y
    );
    auto conv = [da, dc, H, W, C, blk](U16 kc, DU *f, DU *b) {
        dim3 g3((W+TILE3-1)/TILE3, (H+TILE3-1)/TILE3);
        dim3 g5((W+TILE5-1)/TILE5, (H+TILE5-1)/TILE5);
        switch(kc) {            /// * TODO: handles rectangular filters
        case 0x31: k_conv<TILE3,3,1><<<g3,blk>>>(da, f, b, dc, H, W); break;
        case 0x33: k_conv<TILE3,3,3><<<g3,blk>>>(da, f, b, dc, H, W); break;
        case 0x51: k_conv<TILE5,5,1><<<g5,blk>>>(da, f, b, dc, H, W); break;
        case 0x53: k_conv<TILE5,5,3><<<g5,blk>>>(da, f, b, dc, H, W); break;
        default: return -1;
        }
        return 0;
    };
    auto pooling = [da, dc, H, W, C, blk, grd](int ks, t4_layer fn) {
        switch(ks) {           /// pooling kernel size
        case 0x2: k_pooling<2><<<grd,blk>>>(da, dc, H, W, C, fn); break;
        case 0x3: k_pooling<3><<<grd,blk>>>(da, dc, H, W, C, fn); break;
        default: return -1;
        }
        return 0;
    };
    auto dump = [](DU *v, int H, int W, int C) {
        for (int k = 0; k < C; k++) {
            printf("\nC=%d ---", k);
            for (int i = 0; i < H; i++) {
                printf("\n");
                for (int j = 0; j < W; j++) {
                    printf("%.2f ", v[k + (j + i * W) * C]);
                }
            }
        }
        printf("\n");
    };
    ///
    /// layer function dispatcher
    ///
    printf(" out[%d,%d,%d]", H, W, C);
    t4_layer fn = in.grad_fn;                 ///< layer function
    switch(fn) {
    case L_CONV:   {
        Tensor &f = *in.grad[0];              ///< filter tensor
        Tensor &b = *in.grad[1];              ///< bias tensor
        U16 kc = f.H() << 4 | C;              ///< (kerneal_size, channel_depth)
        printf(" f[%d,%d,%d], b[%d]", f.H(), f.W(), f.C(), b.C());
        if (conv(kc, f.data, b.data)) {
            ERROR("model#conv kernel_size=0x%02x not supported\n", kc);
        }
        dump(dc, H, W, C);
    } break;
    case L_LINEAR: {                          ///< out = w @ in + b
        Tensor &w = *in.grad[0];  
        Tensor &b = *in.grad[1];
        printf(" w[%d,%d] @ in[%d,%d] + b[%d,%d]",
               w.H(), w.W(), in.H(), in.W(), b.H(), b.W());
        Tensor::copy(b, out);                 ///< add bias first
        Tensor::gemm(w, in, out, 1.0, 1.0);   ///< out += W * in
        dump(dc, (out.numel+6)/7, 7, 1);
    } break;
    case L_FLATTEN: Tensor::copy(in, out);                 break;
    case L_RELU:    k_relu<<<grd, blk>>>(da, dc, H, W, C); break;
    case L_TANH:    break;
    case L_SIGMOID: break;
    case L_SOFTMAX: {
        Tensor &t = *in.grad[0];             ///< tmp tensor
        Tensor::copy(in, t);                 /// * copy content for exp calc
        DU sum = t.map(O_EXP).sum() + DU_EPS;/// * sum all probabilities
        printf(" sum=%.2f ", sum);
        Tensor::mat(O_MUL, t, DU1/sum, out); /// * p / sum(p)
        dump(dc, 1, out.numel, 1);
    } break;
    case L_MAXPOOL:
    case L_AVGPOOL: 
    case L_MINPOOL: {
        U16 ks = in.parm;                    ///< kerneal_size
        if (pooling(ks, fn)) {
            ERROR("model#pooling kernel_size=%d not supported\n", ks);
        }
        dump(dc, H, W, C);
    } break;
    case L_DROPOUT: Tensor::copy(in, out); break;
    }
    hipDeviceSynchronize();
}
__GPU__ void
Model::_bstep(Tensor &in, Tensor &out) {}
///
/// Convolution and Linear ops
///
__GPU__ void
Model::_iconv(Tensor &in, U16 C, DU bias, U16 *opt) {
    U16 M = opt[0], N = opt[1];                  ///> filter sizing
    U16 p = opt[2] ? opt[2] : int((M-1)/2);      ///> padding
    U16 s = opt[3], d = opt[4];                  ///> stride, dilation
    U16 h = (in.H() - M + p*2) / s + 1;          ///> output height
    U16 w = (in.W() - N + p*2) / s + 1;          ///> output width
    if (M != N || (M != 3 && M != 5)) {
        ERROR("Model#conv2d f=[%d,%d]? 3x3 and 5x5 supported only.\n", M, N);
        return;
    }
    in.stride[0] = in.stride[1] = s;
    Tensor *f  = in.grad[0] = &tensor(1, M, N, C).map(O_FILL, DU1); ///> f
    Tensor *df = in.grad[2] = &tensor(1, M, N, C).map(O_FILL, DU0); ///> df
    Tensor *b  = in.grad[1] = &tensor(1, 1, 1, C).map(O_FILL, DU0); //bias); ///> b
    Tensor *db = in.grad[3] = &tensor(1, 1, 1, C).map(O_FILL, DU0);  ///> db
//    _mmu->random(*f, UNIFORM);                   /// * randomize f
//    Tensor::mat(O_SUB, *f, 0.5, *f);
    
    Tensor &out= tensor(1, h, w, C).map(O_FILL, DU0);  ///> output tensor
    npush(out);                                  /// * stage for next stage
}
__GPU__ void
Model::_ilinear(Tensor &in, U16 n, DU bias) {
    U16 m = in.H();
    Tensor *w  = in.grad[0] = &tensor(1, n, m, 1).identity();  ///> w
    Tensor *dw = in.grad[2] = &tensor(1, n, m, 1).map(O_FILL, DU0);  ///> dw
    Tensor *b  = in.grad[1] = &vector(n).map(O_FILL, DU0); //bias);          ///> b
    Tensor *db = in.grad[3] = &vector(n).map(O_FILL, DU0);           ///> db
//    Tensor::mat(O_MUL, *w, 0.001, *w);
//    _mmu->random(*w, UNIFORM);                   /// * randomize w
    
    Tensor &out = vector(n);                     ///> output tensor sizing
    npush(out);                                  /// * stage for next stage
}
__GPU__ void
Model::_iflatten(Tensor &in) {
    Tensor &out = vector(in.numel);
    in.parm     = in.numel;
    npush(out);
}
///
/// Activation ops
///
__GPU__ void
Model::_icopy(Tensor &in) {
    Tensor &out = _mmu->copy(in); ///> output tensor sizing
    npush(out);                   /// * stage for next stage
}
__GPU__ void
Model::_isoftmax(Tensor &in) {
    Tensor &out = _mmu->copy(in); ///> output tensor sizing
    in.grad[0] = &_mmu->copy(in); ///> tmp for exponental 
    npush(out);                   /// * stage for next stage
}
///
/// Pooling and Dropout ops
///
__GPU__ void
Model::_ipooling(Tensor &in, U16 f) {
    if (f != 2 && f != 3) {
        ERROR("Model#pooling f=[%d,%d]? 2x2 and 3x3 supported only\n", f, f);
        return;
    }
    in.parm = f;                  /// * keep pooling width
    U16 m = int((in.H() - f) / f) + 1;
    U16 n = int((in.W() - f) / f) + 1;
    U16 s[4] = { f, f, 1, 1 }; memcpy(in.stride, s, sizeof(s));  // stride
    
    Tensor &out = tensor(1, m, n, in.C());
    npush(out);                 /// * stage for next stage
}
__GPU__ void
Model::_idropout(Tensor &in, U16 f) {
    Tensor &out = _mmu->copy(in);
    in.parm = f;
    npush(out);
}
#endif  // T4_ENABLE_OBJ
//==========================================================================
