#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - Neural Network Model implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "model.h"

#if T4_ENABLE_OBJ
///
/// Row convolution filter
///
#define CONV_SZ_3    (WARP_SZ - 3 + 1)
#define CONV_SZ_5    (WARP_SZ - 5 + 1)

template<int Z, int T, int K>    ///> WARP_SIZE, TILE_SIZE, KERNEL_SIZE
__KERN__ void k_conv2d(          ///< TODO: C
    DU *A, DU *F, DU *B, DU *C,  ///> input A[MxN], F[KxK] kernel B[KxK] bias, output C[MxN]
    int M, int N
    ) {
    __shared__ DU d[Z * Z];                 ///< shared memory [WARPxWARP]
    
    const int tx = threadIdx.x, j0 = tx + blockIdx.x * T;
    const int ty = threadIdx.y, i0 = ty + blockIdx.y * T;
    const int i  = i0 - int(K / 2);         ///< transfrom to input coordinate
    const int j  = j0 - int(K / 2);
    
    d[tx + ty * Z] =                        ///< shared memory with zero padding
        (i >= 0 && i < M && j >= 0 && j < N)
        ? A[j + i * N] : DU0;
    __syncthreads();

    if (tx < T && ty < T) {                 /// * within tile
        DU sum = DU0;
        for (int y = 0; y < K; y++) {       /// * process one cell
            int d0 = tx + (y + ty) * Z;     ///< offset to smem block
            int b0 = y * K;
            for (int x = 0; x < K; x++) {
                sum += F[x + b0] * d[x + d0] + B[x]; 
            }
        }
        if (i0 < M && j0 < N) {             /// * update C[i][j]
            C[j0 + i0 * N] = sum;           /// * output matrix
        }
    }
}

__KERN__ void k_linear(                     ///< TODO: C
    DU *W, DU *A, DU *B, DU *C,
    int M, int N
    ) {
    const int i = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < M) {
        DU2 acc = DU0;
        for (int j = 0; j < N; j++) {       ///< TODO: CDP
            acc += W[j + i * N] * A[j];
        }
        C[i] = acc + B[i];
    }
}

__KERN__ void k_pooling(                    ///< TODO: C
    DU *A, DU *C,
    int M, int N, int K,
    t4_pool_op op
    ) {
    const int tx = threadIdx.x, j0 = tx + blockIdx.x * blockDim.x;
    const int ty = threadIdx.y, i0 = ty + blockIdx.y * blockDim.y;
    const int j  = j0 * K;
    const int i  = i0 * K;
    
    if (i0 < M && j0 < N) {                ///< TODO: CDP
        DU2 v = (op==POOL_AVG) ? DU0 : A[j + i * N];
        for (int y = 0; y < K; y++) {
            DU *d = &A[j + (y + i) * N];
            for (int x = 0; x < K; x++, d++) {
                switch (op) {
                case POOL_MAX: if (*d > v) v = *d; break;
                case POOL_AVG: v += *d;            break;
                case POOL_MIN: if (*d < v) v = *d; break;
                }
            }
        }
        C[j0 + i0 * N] = (op==POOL_AVG) ? v/K/K : v;
    }
}
        
__KERN__ void k_relu(                                        ///< TODO: C
    DU *A, DU *C, int M, int N) {
    const int j = threadIdx.x + blockIdx.x * blockDim.x;
    const int i = threadIdx.y + blockIdx.y * blockDim.y;
    const int k = i + j * N;
    
    if (i < M && j < N) {                                    ///< TODO: CDP
        C[k] = (A[k] >= DU0) ? A[k] : DU0;
    }
}

__HOST__ const char*
Model::nname(int i) {               ///< network layer name
    static const char *name[] = {   /// double check with t4_layer
    "output ", "conv2d ", "linear ", "flatten", "relu   ",
    "tanh   ", "sigmoid", "softmax", "maxpool", "avgpool",
    "minpool", "dropout"
    };
    return name[i];
}
///
/// NN layer factory
///
__GPU__ Model&
Model::add(t4_layer fn, U16 n, DU bias, U16 *opt) {
    Tensor &in = _mmu->du2ten(data[numel - 1]);
    if (!autograd || in.grad_fn != L_NONE) return *this;
    
    switch(fn) {
    case L_CONV2D:  _iconv2d(in, n, bias, opt); break;
    case L_LINEAR:  _ilinear(in, n, bias);      break;
    case L_FLATTEN: _iflatten(in);              break;
    case L_RELU:    _irelu(in);                 break;
    case L_TANH:    _itanh(in);                 break;
    case L_SIGMOID: _isigmoid(in);              break;
    case L_SOFTMAX: _isoftmax(in);              break;
    case L_MAXPOOL: _imaxpool(in, n);           break;
    case L_AVGPOOL: _iavgpool(in, n);           break;
    case L_MINPOOL: _iminpool(in, n);           break;
    case L_DROPOUT: _idropout(in, n);           break;
    }
    in.grad_fn = fn;
    return *this;
}

__GPU__ Model&
Model::forward(Tensor &input) {
    Tensor &in = (*this)[1];
    if (!in.is_same_shape(input)) {
        ERROR("model#forward dim?\n");
        return *this;
    }
    Tensor::copy(input, in);
    /*
    for (int i = 2; i < (model.numel - 1); i++) {
        Tensor &out = model[i];
        model.forward(in, out);
        in = out;
    }
    */
    return *this;
}
__GPU__ Model&
Model::backprop(Tensor &output) {
    return *this;
}
/// ========================================================================
/// private methods 
///
__GPU__ void
Model::_step(Tensor &in, Tensor &out) {
    DU   *da = in.data;
    DU   *dc = out.data;
    int  m   = out.H();
    int  n   = out.W();
    int  k   = in.parm;
    dim3 blk(WARP_SZ, WARP_SZ);
    dim3 grd((n + WARP_SZ - 1)/WARP_SZ, (m + WARP_SZ - 1)/WARP_SZ);

    auto conv3x3 = [da, dc, m, n, blk](DU *f, DU *b) {
        dim3 g((n+CONV_SZ_3-1)/CONV_SZ_3, (m+CONV_SZ_3-1)/CONV_SZ_3);
        k_conv2d<WARP_SZ, CONV_SZ_3, 3><<<g, blk>>>(da, f, b, dc, m, n);
    };
    auto conv5x5 = [da, dc, m, n, blk](DU *f, DU *b) {
        dim3 g((n+CONV_SZ_5-1)/CONV_SZ_5, (m+CONV_SZ_5-1)/CONV_SZ_5);
        k_conv2d<WARP_SZ, CONV_SZ_5, 5><<<g, blk>>>(da, f, b, dc, m, n);
    };
    
    switch(in.grad_fn) {
    case L_CONV2D: {
        Tensor &f = *in.grad[0];     ///< filter tensor
        Tensor &b = *in.grad[1];     ///< bias tensor
        int ks = f.H();
        switch(ks) {
        case 3: conv3x3(f.data, b.data); break;
        case 5: conv5x5(f.data, b.data); break;
        default: ERROR("model#conv2d kernel_size=%d not supported\n", ks);
        }
    } break;
    case L_LINEAR:  {                ///< dc = W * da + B
        Tensor &w = *in.grad[0];  
        Tensor &b = *in.grad[1];
        int    W2   = WARP_SZ * WARP_SZ;
        dim3   blk1(1, W2), grd1(1, (w.H() + W2 - 1) / W2);
        k_linear<<<grd1, blk1>>>(w.data, da, b.data, dc, w.H(), w.W());
    } break;
    case L_FLATTEN: out.reshape(out.numel); break;
    case L_RELU:    k_relu<<<grd, blk>>>(da, dc, m, n); break;
    case L_TANH:    break;
    case L_SIGMOID: break;
    case L_SOFTMAX: {
        Tensor &tmp = _mmu->copy(in);
        DU sum = tmp.map(O_EXP).sum();        /// * sum all probabilities
        Tensor::mat(O_DIV, tmp, sum, out);    /// * p / sum(p)
    } break;
    case L_MAXPOOL: k_pooling<<<grd, blk>>>(da, dc, m, n, k, POOL_MAX); break;
    case L_AVGPOOL: k_pooling<<<grd, blk>>>(da, dc, m, n, k, POOL_AVG); break;
    case L_MINPOOL: k_pooling<<<grd, blk>>>(da, dc, m, n, k, POOL_MIN); break;
    case L_DROPOUT: break;
    }
}
///
/// Convolution and Linear ops
///
__GPU__ void
Model::_iconv2d(Tensor &in, U16 c, DU bias, U16 *opt) {
    U16 m = opt[0], n = opt[1];                  ///> filter sizing
    U16 p = opt[2] ? opt[2] : int((m-1)/2);      ///> padding
    U16 s = opt[3], d = opt[4];                  ///> stride, dilation
    U16 h = in.H() - 2 * (p - int(m/2));         ///> output height
    U16 w = in.W() - 2 * (p - int(n/2));         ///> output width

    Tensor *f  = in.grad[0] = &tensor(1, m, n, c);                   ///> f
    Tensor *df = in.grad[2] = &tensor(1, m, n, c).map(O_FILL, DU0);  ///> df
    Tensor *b  = in.grad[1] = &tensor(1, h, w, 1).map(O_FILL, bias); ///> b
    Tensor *db = in.grad[3] = &tensor(1, h, w, 1).map(O_FILL, DU0);  ///> db
    _mmu->random(*f, NORMAL);                    /// * randomize f
    
    Tensor &out= tensor(1, h, w, c).map(O_FILL, DU0);  ///> output tensor
    npush(out);                                  /// * stage for next stage
}
__GPU__ void
Model::_ilinear(Tensor &in, U16 n, DU bias) {
    U16 m = in.H();
    Tensor *w  = in.grad[0] = &tensor(1, n, m, 1);                   ///> w
    Tensor *dw = in.grad[2] = &tensor(1, n, m, 1).map(O_FILL, DU0);  ///> dw
    Tensor *b  = in.grad[1] = &vector(n).map(O_FILL, bias);          ///> b
    Tensor *db = in.grad[3] = &vector(n).map(O_FILL, DU0);           ///> db
    _mmu->random(*w, NORMAL);                    /// * randomize w
    
    Tensor &out = vector(n);                     ///> output tensor sizing
    npush(out);                                  /// * stage for next stage
}
__GPU__ void
Model::_iflatten(Tensor &in) {
    Tensor &out = vector(in.numel);
    in.parm     = in.numel;
    npush(out);
}
///
/// Activation ops
///
__GPU__ void
Model::_irelu(Tensor &in) {
    Tensor &out = _mmu->copy(in); ///> output tensor sizing
    npush(out);                   /// * stage for next stage
}
__GPU__ void
Model::_itanh(Tensor &in) {}

__GPU__ void
Model::_isigmoid(Tensor &in) {}

__GPU__ void
Model::_isoftmax(Tensor &in) {
    Tensor &out = _mmu->copy(in); ///> output tensor sizing
    npush(out);                   /// * stage for next stage
}
///
/// Pooling and Dropout ops
///
__GPU__ void
Model::_imaxpool(Tensor &in, U16 f) {
    in.parm = f;                  /// * keep pooling width
    U16 m = int((in.H() - f) / f) + 1;
    U16 n = int((in.W() - f) / f) + 1;
    U16 s[4] = { 1, f, f, 1 }; memcpy(in.stride, s, sizeof(s));  // stride
    
    Tensor &out = tensor(1, m, n, in.C());
    npush(out);                 /// * stage for next stage
}
__GPU__ void
Model::_iavgpool(Tensor &in, U16 n) {}

__GPU__ void
Model::_iminpool(Tensor &in, U16 n) {}

__GPU__ void
Model::_idropout(Tensor &in, U16 f) {
    Tensor &out = _mmu->copy(in);
    in.parm = f;
    npush(out);
}
#endif  // T4_ENABLE_OBJ
//==========================================================================
