#include "hip/hip_runtime.h"
/**
 * @file
 * @brief tensorForth - Memory Manager
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include <iomanip>          // setw, setbase
#include "mmu.h"
///
/// Forth Virtual Machine operational macros to reduce verbosity
///
__HOST__
MMU::MMU() {
    hipMallocManaged(&_dict, sizeof(Code) * T4_DICT_SZ);
    hipMallocManaged(&_pmem, T4_PMEM_SZ);
    hipMallocManaged(&_vss,  sizeof(DU) * T4_SS_SZ * VM_MIN_COUNT);
    hipMallocManaged(&_ten,  T4_TENSOR_SZ);
    GPU_CHK();

    tstore.init(_ten, T4_TENSOR_SZ);
    
    MMU_TRACE("MMU dict=%p, mem=%p, vss=%p, ten=%p\n", _dict, _pmem, _vss, _ten);
}
__HOST__
MMU::~MMU() {
    GPU_SYNC();
    hipFree(_ten);
    hipFree(_vss);
    hipFree(_pmem);
    hipFree(_dict);
}
///
/// dictionary search functions - can be adapted for ROM+RAM
///
__GPU__ int
MMU::find(const char *s, bool compile, bool ucase) {
    MMU_TRACE("find(%s) => ", s);
    for (int i = _didx - (compile ? 2 : 1); i >= 0; --i) {
        const char *t = _dict[i].name;
        if (ucase && STRCASECMP(t, s)==0) return i;
        if (!ucase && STRCMP(t, s)==0) return i;
    }
    return -1;
}
///
/// colon - dictionary word compiler
///
__GPU__ void
MMU::colon(const char *name) {
    MMU_TRACE("colon(%s) => ", name);
    int  sz = STRLENB(name);                // aligned string length
    Code &c = _dict[_didx++];               // get next dictionary slot
    align();                                // nfa 32-bit aligned (adjust _midx)
    c.name = (const char*)&_pmem[_midx];    // assign name field index
    c.def  = 1;                             // specify a colon word
    add((U8*)name,  ALIGN2(sz+1));          // setup raw name field
    c.pfa  = _midx;                         // capture code field index
}
///
/// Debugging methods
///
/// display dictionary word (wastefully one byte at a time)
///
__HOST__ void
MMU::to_s(std::ostream &fout, IU w) {
    /*
     * TODO: not sure why copying 32 byt does not work?
     * char name[36];
     * hipMemcpy(name, _dict[w].name, 32, D2H);
     */
    U8 c, i=0;
    hipMemcpy(&c, _dict[w].name, 1, D2H);
    while (c) {
        fout << c;
        hipMemcpy(&c, _dict[w].name+(++i), 1, D2H);
    }
#if T4_VERBOSE
    fout << " " << w << (_dict[w].immd ? "* " : " ");
#else   // T4_VERBOSE
    fout << " ";
#endif  // T4_VERBOSE
}
///
/// tensor life-cycle methods
///
__GPU__ Tensor&
MMU::tensor(U16 h, U16 w) {
    Tensor *t = (Tensor*)tstore.malloc(sizeof(Tensor));
    U32    sz = h * w;
    PRINTF("mmu#tensor(%d,%d) => size=%d\n", h, w, sz);
    
    void   *mptr = tstore.malloc((U64)sizeof(DU) * sz);
    t->reset(mptr, sz);
    t->reshape(h, w);
    __syncthreads();
    
    return *t;
};

__GPU__ Tensor&
MMU::tensor(U16 n, U16 h, U16 w, U16 c) {
    Tensor *t = (Tensor*)tstore.malloc(sizeof(Tensor));
    U32    sz = n * h * w * c;
    PRINTF("mmu#tensor(%d,%d,%d,%d) => size=%d\n", n, h, w, c, sz);
    
    void   *mptr = (void*)tstore.malloc((U64)sizeof(DU) * sz);
    t->reset(mptr, sz);
    t->reshape(n, h, w, c);
    __syncthreads();
    
    return *t;
}
///
/// display dictionary word list
///
__HOST__ void
MMU::words(std::ostream &fout) {
    fout << std::setbase(10);
    for (int i=0; i<_didx; i++) {
        if ((i%10)==0) { fout << std::endl; }
        to_s(fout, i);
    }
}
///
/// recursively disassemble colon word
///
__HOST__ void
MMU::see(std::ostream &fout, U8 *ip, int dp) {
    while (*(IU*)ip) {                                              /// * loop until EXIT
        fout << std::endl; for (int n=dp; n>0; n--) fout << "  ";   /// * indentation by level
           fout << "[" << std::setw(4) << (IU)(ip - _pmem) << ": ";
        IU c = *(IU*)ip;                                            /// * fetch word index
        to_s(fout, c);                                              /// * display word name
        if (_dict[c].def && dp < 2) {                               /// * check if is a colon word
            see(fout, &_pmem[_dict[c].pfa], dp+1);                  /// * go one level deeper
        }
        ip += sizeof(IU);                                           /// * advance instruction pointer
        switch (c) {
        case DOVAR: case DOLIT:
            fout << "= " << (*(DU*)ip); ip += sizeof(DU); break;      /// fetch literal
        case DOSTR: case DOTSTR: {
            char *s = (char*)ip;
            int  sz = strlen(s)+1;
            ip += ALIGN2(sz);                                       /// fetch string
            fout << "= \"" << s << "\"";
        } break;
        case BRAN: case ZBRAN: case DONEXT:
            fout << "j" << *(IU*)ip; ip += sizeof(IU); break;       /// fetch jump target
        }
        fout << "] ";
    }
}
__HOST__ void
MMU::see(std::ostream &fout, U16 w) {
    fout << "[ "; to_s(fout, w);
    if (_dict[w].def) see(fout, &_pmem[_dict[w].pfa]);
    fout << "]" << std::endl;
}
///
/// dump data stack content
///
__HOST__ void
MMU::ss_dump(std::ostream &fout, U16 vid, U16 n, int radix) {
    bool x = radix != 10;
    DU *ss = &_vss[vid * T4_SS_SZ];
    fout << " <";
    if (x) fout << std::setbase(radix);
    for (U16 i=0; i<n; i++) {
        if (x) fout << static_cast<int>(ss[i]);
        else   fout << ss[i];
        fout << " ";
    }
    if (x) fout << static_cast<int>(ss[T4_SS_SZ-1]);
    else   fout << ss[T4_SS_SZ-1];
    fout << "> ok" << std::endl;
}
///
/// Forth pmem memory dump
/// TODO: dynamic parallel
///
#define C2H(c) { buf[x++] = i2h[(c)>>4]; buf[x++] = i2h[(c)&0xf]; }
#define IU2H(i){ C2H((i)>>8); C2H((i)&0xff); }
__HOST__ void
MMU::mem_dump(std::ostream &fout, U16 p0, U16 sz) {
    const char *i2h = "0123456789abcdef";
    char buf[80];
    for (U16 i=ALIGN16(p0); i<=ALIGN16(p0+sz); i+=16) {
        int x = 0;
        buf[x++] = '\n'; IU2H(i); buf[x++] = ':'; buf[x++] = ' ';  // "%04x: "
        for (U16 j=0; j<16; j++) {
            //U8 c = *(((U8*)&_dict[0])+i+j) & 0x7f;               // to dump _dict
            U8 c = _pmem[i+j];
            C2H(c);                                                // "%02x "
            c &= 0x7f;                                             // mask off high bit
            buf[x++] = ' ';
            if (j%4==3) buf[x++] = ' ';
            buf[59+j]= (c==0x7f||c<0x20) ? '.' : c;                // %c
        }
        buf[75] = '\0';
        fout << buf;
    }
    fout << std::endl;
}
