/*! @file
  @brief
  cueForth - Memory Manager
*/
#include <iomanip>          // setw, setbase
#include "mmu.h"
///
/// Forth Virtual Machine operational macros to reduce verbosity
///
__HOST__
MMU::MMU() {
    hipMallocManaged(&_dict, sizeof(Code) * CUEF_DICT_SZ);
    hipMallocManaged(&_pmem, sizeof(U8) * CUEF_HEAP_SZ);
    hipMallocManaged(&_vss,  sizeof(DU) * CUEF_SS_SZ * MIN_VM_COUNT);
    GPU_CHK();
    printf("H: dict=%p, mem=%p, vss=%p\n", _dict, _pmem, _vss);
}
__HOST__
MMU::~MMU() {
    GPU_SYNC();
    hipFree(_vss);
    hipFree(_pmem);
    hipFree(_dict);
}
///
/// dictionary search functions - can be adapted for ROM+RAM
///
__GPU__ int
MMU::find(const char *s, bool compile, bool ucase) {
    printf("find(%s) => ", s);
    for (int i = _didx - (compile ? 2 : 1); i >= 0; --i) {
        const char *t = _dict[i].name;
        if (ucase && STRCASECMP(t, s)==0) return i;
        if (!ucase && STRCMP(t, s)==0) return i;
    }
    return -1;
}
///
/// colon - dictionary word compiler
///
__GPU__ void
MMU::colon(const char *name) {
    int  sz = STRLENB(name);                // aligned string length
    Code &c = _dict[_didx++];               // get next dictionary slot
    align();                                // nfa 32-bit aligned (adjust _midx)
    c.name = (const char*)&_pmem[_midx];    // assign name field index
    c.def  = 1;                             // specify a colon word
    add((U8*)name,  ALIGN2(sz+1));          // setup raw name field
    c.pfa  = _midx;                         // capture code field index
}
///
/// Debugging methods
///
/// display dictionary word (wastefully one byte at a time)
///
__HOST__ void
MMU::to_s(std::ostream &fout, IU w) {
    /*
     * TODO: not sure why copying 32 byt does not work?
     * char name[36];
     * hipMemcpy(name, _dict[w].name, 32, D2H);
     */
    U8 c, i=0;
    hipMemcpy(&c, _dict[w].name, 1, D2H);
    while (c) {
        fout << c;
        hipMemcpy(&c, _dict[w].name+(++i), 1, D2H);
    }
    fout << " " << w << (_dict[w].immd ? "* " : " ");
}
///
/// display dictionary word list
///
__HOST__ void
MMU::words(std::ostream &fout) {
    fout << std::setbase(10);
    for (int i=0; i<_didx; i++) {
        if ((i%10)==0) { fout << std::endl; }
        to_s(fout, i);
    }
}
///
/// recursively disassemble colon word
///
__HOST__ int
MMU::pfa2word(IU ix) {
    IU   def = ix & 1;
    IU   pfa = ix & ~0x1;             /// TODO: handle colon immediate words when > 64K
    UFP  xt  = _xt0 + ix;             /// function pointer
    for (int i = _didx - 1; i >= 0; --i) {
        if (def) {
            if (_dict[i].pfa == pfa) return i;      /// compare pfa in PMEM
        }
        else if ((UFP)_dict[i].xt == xt) return i;  /// compare xt (no immediate?)
    }
    return 0;                         /// not found, return EXIT
}

__HOST__ void
MMU::see(std::ostream &fout, U8 *p, U16 dp) {
	while (*(IU*)p) {                                               /// * loop until EXIT
        fout << std::endl; for (int n=dp; n>0; n--) fout << "  ";   /// * indentation by level
        fout << "[" << std::setw(4) << (IU)(p - _pmem) << ": ";
        IU c = pfa2word(*(IU*)p);                                   /// * convert pfa to word index
	    to_s(fout, c);                                              /// * display word name
        if (_dict[c].def && dp < 2) {                               /// * check if is a colon word
        	see(fout, &_pmem[_dict[c].pfa], dp+1);                  /// * go one level deeper
        }
        p += sizeof(IU);                                            /// * advance instruction pointer
        switch (c) {
        case DOVAR: case DOLIT:
            fout << "= " << *(DU*)p; p += sizeof(DU); break;        // fetch literal
        case DOSTR: case DOTSTR: {
            char *s = (char*)p;
            int  sz = strlen(s)+1;
            p += ALIGN2(sz);                                        // fetch string
            fout << "= \"" << s << "\"";
        } break;
        case BRAN: case ZBRAN: case DONEXT:
            fout << "j" << *(IU*)p; p += sizeof(IU); break;         // fetch jump target
        }
        fout << "] ";
	}
}
__HOST__ void
MMU::see(std::ostream &fout, IU w) {
    fout << "[ "; to_s(fout, w);
    if (_dict[w].def) see(fout, &_pmem[_dict[w].pfa], 1);
    fout << "] " << std::endl;
}
///
/// dump data stack content
///
__HOST__ void
MMU::ss_dump(std::ostream &fout, IU vid, U16 n) {
    DU *ss = &_vss[vid * CUEF_SS_SZ];
    fout << " <";
    for (U16 i=0; i<n; i++) { fout << ss[i] << " "; }
    fout << ss[CUEF_SS_SZ-1] << "> ok" << std::endl;
}
///
/// Forth pmem memory dump
/// TODO: dynamic parallel
///
#define C2H(c) { buf[x++] = i2h[(c)>>4]; buf[x++] = i2h[(c)&0xf]; }
#define IU2H(i){ C2H((i)>>8); C2H((i)&0xff); }
__HOST__ void
MMU::mem_dump(std::ostream &fout, IU p0, U16 sz) {
    const char *i2h = "0123456789abcdef";
    char buf[80];
    for (IU i=ALIGN16(p0); i<=ALIGN16(p0+sz); i+=16) {
        int x = 0;
        buf[x++] = '\n'; IU2H(i); buf[x++] = ':'; buf[x++] = ' ';  // "%04x: "
        for (IU j=0; j<16; j++) {
            //U8 c = *(((U8*)&_dict[0])+i+j) & 0x7f;               // to dump _dict
            U8 c = _pmem[i+j] & 0x7f;
            C2H(c);                                                // "%02x "
            buf[x++] = ' ';
            if (j%4==3) buf[x++] = ' ';
            buf[59+j]= (c==0x7f||c<0x20) ? '.' : c;                // %c
        }
        buf[75] = '\0';
        fout << buf;
    }
    fout << std::endl;
}
