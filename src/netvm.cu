#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - Neural Network Vritual Machine implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "netvm.h"

#if T4_ENABLE_OBJ
///===================================================================
/// static loss functions
///
__GPU__ void
NetVM::loss_nll(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::loss_mse(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::loss_ce(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::predict(Tensor &A, Tensor &B, Tensor &C) {
}
///
/// Convolution ops
///
__GPU__ void
NetVM::conv2d(U16 *opt) {
    if (IS_OBJ(top) || IS_OBJ(ss[-1])) return; ///> bias, c params requred
    
    U16 c      = POPi;       ///> number of output channels
    DU  bias   = POP();      ///> convolution bias
    ///
    /// create autograd tensors if not yet
    ///
    if (f_auto) model.init_conv2d(bias, c, opt);
    ///
    /// apply convolution filter
    ///
}
__GPU__ void
NetVM::conv2d() {
    U16 opt[] = { 3, 3, 1, 1, 1 };   ///> default 3x3 filter, padding=1, stride=1, dilation=1
    if (IS_TEN(top)) {
        Tensor &v = mmu.du2ten(top);
        if (v.rank == 1) {
            POP();
            DU  *d = (DU*)v.data;
            for (int i=0; i<5; i++) opt[i] = (U16)d[i];
        }
        else ERROR("vec?");
    }
    conv2d(opt);                     ///> perform 2D convolution
}
///
/// Pooling ops
///
__GPU__ void
NetVM::meanpool(U16 n) {
}
__GPU__ void
NetVM::avgpool(U16 n) {
}
__GPU__ void
NetVM::maxpool(U16 n) {
    
}
__GPU__ void
NetVM::minpool(U16 n) {
}
///
/// Activation ops
///
__GPU__ void
NetVM::relu() {
}
__GPU__ void
NetVM::tanh() {
}
__GPU__ void
NetVM::sigmoid() {
}
__GPU__ void
NetVM::softmax() {
}
///
/// Pooling ops
///
__GPU__ void
NetVM::linear(U16 n) {
}
///
/// Pooling ops
///
__GPU__ void
NetVM::dropout(U16 p) {
}
///
/// Back Propegation ops
///
__GPU__ void
NetVM::autograd(bool on) {
    f_auto = on;
}
__GPU__ void
NetVM::for_batch() {
    Tensor &A = mmu.tensor(1, 28, 28, 1);
    model.push(A);
}
__GPU__ void
NetVM::backprop() {
}
__GPU__ void
NetVM::sgd() {
}
__GPU__ void
NetVM::adam() {
}
///===================================================================
/// class methods
///
/// Neural Network specific dictionary constructor
///
__GPU__ void
NetVM::init() {
    const Code prim[] = {       /// singleton, build once only
    ///@defgroup Convolution ops
    ///@{
    CODE("conv2d", conv2d()),             ///> (Ta b c [A] -- Ta')
    ///@}
    ///@defgroup Activation ops
    ///@{
    CODE("relu",      {}),
    CODE("tanh",      {}),
    CODE("sigmoid",   {}),
    CODE("softmax",   {}),
    ///@}
    ///@defgroup Pooling ops
    ///@{
    CODE("meanpool",  {}),
    CODE("avgpool",   {}),
    CODE("maxpool",   {}),
    CODE("minpool",   {}),
    ///@}
    ///@defgroup Loss functions
    ///@{
    CODE("linear",    {}),
    CODE("loss_nll",  {}),
    CODE("loss_mse",  {}),
    CODE("loss_ce",   {}),
    CODE("predict",   {}),
    ///@}
    ///@defgroup Tensor fill ops
    ///@{
    CODE("batch_for", {}),
    CODE("batch_next",{}),
    CODE("sgd",       {}),
    CODE("adam",      {}),
    ///@}
    ///@defgroup Debugging ops
    ///@{
    CODE("network",   fout << opx(OP_NET, model.idx, model.data[0])),
    CODE(">n",        model.push(top); POP()),
    CODE("n>",        DU t = model.pop(); PUSH(t)),
    CODE("autograd",  {}),
    ///@}
    };
    const Code over[] = {          /// extended (overload) words
    CODE("boot", mmu.clear(FIND("autograd") + 1))
    };
    TensorVM::init();

    mmu.append(prim, sizeof(prim)/sizeof(Code)); /// * append tensor words
    mmu.merge(over,  sizeof(over)/sizeof(Code)); /// * overload existed words
    mmu.status();
};
#endif  // T4_ENABLE_OBJ
//=======================================================================================
