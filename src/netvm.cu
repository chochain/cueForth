#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - Neural Network Vritual Machine implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "netvm.h"

#if T4_ENABLE_OBJ
__GPU__ void
NetVM::nnop(t4_layer op) {     /// vtable dispatcher
    ///
    /// handle tensor ops (proxy)
    ///
    if (TOS1T) {
        switch (op) {
        case L_RELU:    xop1(O_RELU, DU0); break; ///> (Ta -- Ta Ta')
        case L_TANH:    xop1(O_TANH);      break; ///> (Ta -- Ta Ta')
        case L_SIGMOID: xop1(O_SIGM);      break; ///> (Ta -- Ta Ta')
        case L_FLATTEN:                           ///> (Ta -- Ta Ta')
            Tensor &t = TTOS;
            t.reshape(t.numel);            break; 
        }
        return;
    }
    ///
    /// model layer ops
    ///
    switch (op) {
    case L_CONV:  _conv(); break;
    case L_LINEAR:
        if (MN2D) {                                ///> param checking
            U16   n    = POPi;                     ///> number of output channels
            DU    bias = POP();                    ///> convolution bias
            NN0.add(L_LINEAR, n, bias);            ///> (N b c -- N')
        }
        else ERROR("linear: bias n required!");
        break;
    case L_FLATTEN:
    case L_RELU:
    case L_TANH:
    case L_SIGMOID:
    case L_SOFTMAX: if (MTOS) NN0.add(op); break;
    case L_MAXPOOL: 
    case L_AVGPOOL:
    case L_MINPOOL: if (MNOS) { U16 n = POPi; NN0.add(op, n); } break;
    case L_DROPOUT: if (MNOS) {
            U16 p = int(100.0 * POP() + 0.5); NN0.add(op, p);
        } break;
    default: ERROR("NetVM::nnop(%d) not supported\n", op);
    }
}
///===================================================================
/// static loss functions
///
__GPU__ void
NetVM::loss_nll(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::loss_mse(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::loss_ce(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::predict(Tensor &A, Tensor &B, Tensor &C) {
}
///
/// Convolution and Linear ops
/// @default: 3x3 filter, padding=1, stride=1, dilation=1
///
__GPU__ void
NetVM::_conv() {
    U16 opt[] = { 3, 3, 1, 1, 1 };   ///> default config vector
    if (TOS1T) {                     ///> if optional vector given
        Tensor &v = TTOS;
        if (v.rank == 1) {
            POP();
            for (int i=0; i<5; i++) opt[i] = (U16)v.data[i];
        }
        else { ERROR("vec?"); return; }
    }
    if (!MN2D) {
        ERROR("conv2d bias c required!"); return;
    }
    U16 c    = POPi;                 ///> number of output channels
    DU  bias = POP();                ///> convolution bias
    NN0.add(L_CONV, c, bias, opt);
}
///
/// Batch ops
///
__GPU__ void
NetVM::nn_for() {}

__GPU__ void
NetVM::nn_next() {}
///
/// NN model propegation
///
__GPU__ void
NetVM::sgd() {}

__GPU__ void
NetVM::adam() {}
///===================================================================
/// class methods
///
/// Neural Network specific dictionary constructor
///
__GPU__ void
NetVM::init() {
    const Code prim[] = {                   ///> singleton, build once only
    ///@defgroup Convolution and Linear ops
    ///@{
    CODE("nn.model",                          ///> (n h w c -- N)
         if (ss.idx < 4 ||                    /// * param check
             IS_OBJ(top) || IS_OBJ(ss[-1]) ||
             IS_OBJ(ss[-2]) || IS_OBJ(ss[-3])) {
             ERROR("n h w c?\n"); return;
         }
         U16 c=POPi; U16 w=POPi; U16 h=POPi; U16 n=POPi;
         Model  &m = mmu.model();             /// * create NN model
         Tensor &t = mmu.tensor(n,h,w,c);     /// * create input tensor
         m.npush(t);                          /// * serves as the 1st layer
         PUSH(m)),
    CODE("conv2d",    nnop(L_CONV)),          ///> (N b c [A] -- N')
    CODE("linear",    nnop(L_LINEAR)),        ///> (N b n -- N')
    ///@}
    ///@defgroup Activation ops
    ///@{
    CODE("relu",      nnop(L_RELU)),          ///> (N -- N')
    CODE("tanh",      nnop(L_TANH)),          ///> (N -- N')
    CODE("sigmoid",   nnop(L_SIGMOID)),       ///> (N -- N')
    CODE("softmax",   nnop(L_SOFTMAX)),       ///> (N -- N')
    ///@}
    ///@defgroup Pooling and Dropout ops
    ///@{
    CODE("maxpool",   nnop(L_MAXPOOL)),       ///> (N n -- N')
    CODE("avgpool",   nnop(L_AVGPOOL)),       ///> (N n -- N')
    CODE("minpool",   nnop(L_MINPOOL)),       ///> (N n -- N')
    CODE("dropout",   nnop(L_DROPOUT)),       ///> (N p -- N')
    ///@}
    ///@defgroup Loss functions
    ///@{
    CODE("loss.nll",  {}),
    CODE("loss.mse",  {}),
    CODE("loss.ce",   {}),
    ///@}
    ///@defgroup Gradiant ops
    ///@{
    CODE("nn.sgd",    {}),
    CODE("nn.adam",   {}),
    ///@}
    ///@defgroup Batch ops
    ///@{
    CODE("nn.for",    {}),
    CODE("nn.next",   {}),
    CODE("autograd",  if (MNOS) { bool on = POPi; NN0.autograd = on; }),
    CODE("forward", 
         if (TOS1T && IS_M(ss[-1])) {
             Tensor &t = TTOS; Model &m = NN1;
             m.forward(t);
             PUSH(mmu.view(m.output()));
         }),
    CODE("backprop",
         if (TOS1T && IS_M(ss[-1])) {
             Tensor &t = TTOS; NN1.backprop(t);
         }),
    CODE("predict",   {}),
    ///@}
    ///@defgroup Debugging ops
    ///@{
    CODE(">n",        if (MNOS) { DU t = POP(); NN0.npush(t); }),
    CODE("n@",        if (MNOS) { DU i = POPi; PUSH(mmu.view(NN0[i])); }),
    CODE("network",   if (MTOS) fout << top),
    ///@}
    };
    const Code over[] = {           /// extended (overload) words
    CODE("flatten",   nnop(L_FLATTEN)),
    CODE("boot",      mmu.clear(FIND("network") + 1))
    };
    TensorVM::init();

    mmu.append(prim, sizeof(prim)/sizeof(Code)); /// * append tensor words
    mmu.merge(over,  sizeof(over)/sizeof(Code)); /// * overload existed words
    mmu.status();
};
#endif  // T4_ENABLE_OBJ
//===========================================================================
