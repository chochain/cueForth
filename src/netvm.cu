#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - Neural Network Vritual Machine implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "netvm.h"

#if T4_ENABLE_OBJ
///===================================================================
/// static loss functions
///
__GPU__ void
NetVM::loss_nll(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::loss_mse(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::loss_ce(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::predict(Tensor &A, Tensor &B, Tensor &C) {
}
///
/// Convolution ops
///
__GPU__ void
NetVM::conv2d() {
    U16 opt[] = { 3, 3, 1, 1, 1 };   ///> default 3x3 filter, padding=1, stride=1, dilation=1
    if (IS_TEN(top)) {
        Tensor &v = mmu.du2ten(top);
        if (v.rank == 1) {
            POP();
            for (int i=0; i<5; i++) opt[i] = (U16)v.data[i];
        }
        else { ERROR("vec?"); return; }
    }
    if (IS_OBJ(top) || IS_OBJ(ss[-1])) {
        ERROR("conv2d bias c required!"); return;
    }
    
    U16 c      = POPi;                      ///> number of output channels
    DU  bias   = POP();                     ///> convolution bias

    if (wet()) model.iconv2d(bias, c, opt); /// create autograd tensors if needed
    ///
    /// perform 2D convolution
    ///
}
///
/// Pooling ops
///
__GPU__ void
NetVM::maxpool() {
    if (IS_OBJ(top)) { ERROR("#maxpool n required!"); return; }
    U16 n = POPi;
    if (wet()) model.imaxpool(n);
    ///
    /// perform maxpool
    ///
}
///
/// Activation ops
///
__GPU__ void
NetVM::relu() {
    if (wet()) model.irelu();
    ///
    /// perform ReLU
    ///
}
///
/// Linear ops
///
__GPU__ void
NetVM::linear() {
    if (IS_OBJ(top)) { ERROR("#linear n required!"); return; }
    U16 n = POPi;
    if (wet()) model.ilinear(n);
    ///
    /// perform linear transformation
    ///
}
///
/// Back Propegation ops
///
__GPU__ void
NetVM::autograd(bool on) {
    f_auto = on;
}
__GPU__ void
NetVM::for_batch() {
    Tensor &A = mmu.tensor(1, 28, 28, 1);
    model.push(A);
}
__GPU__ void
NetVM::backprop() {
}
__GPU__ void
NetVM::sgd() {
}
__GPU__ void
NetVM::adam() {
}
///===================================================================
/// class methods
///
/// Neural Network specific dictionary constructor
///
__GPU__ void
NetVM::init() {
    const Code prim[] = {       /// singleton, build once only
    ///@defgroup Convolution ops
    ///@{
    CODE("conv2d",    conv2d()),     ///> (Ta b c [A] -- Ta')
    ///@}
    ///@defgroup Activation ops
    ///@{
    CODE("relu",      relu()),       ///> (Ta -- Ta')
    CODE("tanh",      {}),
    CODE("sigmoid",   {}),
    CODE("softmax",   {}),
    ///@}
    ///@defgroup Pooling ops
    ///@{
    CODE("maxpool",   maxpool()),    ///> (Ta n -- Ta')
    CODE("meanpool",  {}),
    CODE("avgpool",   {}),
    CODE("minpool",   {}),
    ///@}
    ///@defgroup Linear ops
    ///@{
    CODE("linear",    linear()),     ///> (Ta n -- Ta')
    ///@}
    ///@defgroup Loss functions
    ///@{
    CODE("loss_nll",  {}),
    CODE("loss_mse",  {}),
    CODE("loss_ce",   {}),
    CODE("predict",   {}),
    ///@}
    ///@defgroup Tensor fill ops
    ///@{
    CODE("batch_for", {}),
    CODE("batch_next",{}),
    CODE("sgd",       {}),
    CODE("adam",      {}),
    ///@}
    ///@defgroup Debugging ops
    ///@{
    CODE("network",   fout << opx(OP_NET, model.idx, model.data[0])),
    CODE(">n",        model.push(top); POP()),
    CODE("n>",        DU t = model.pop(); PUSH(t)),
    CODE("autograd",  autograd(POPi)),
    ///@}
    };
    const Code over[] = {          /// extended (overload) words
    CODE("boot", mmu.clear(FIND("autograd") + 1))
    };
    TensorVM::init();

    mmu.append(prim, sizeof(prim)/sizeof(Code)); /// * append tensor words
    mmu.merge(over,  sizeof(over)/sizeof(Code)); /// * overload existed words
    mmu.status();
};
#endif  // T4_ENABLE_OBJ
//=======================================================================================
