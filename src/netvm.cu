#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - Neural Network Vritual Machine implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "netvm.h"

#if T4_ENABLE_OBJ
///===================================================================
/// static loss functions
///
__GPU__ void
NetVM::loss_nll(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::loss_mse(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::loss_ce(Tensor &A, Tensor &B, Tensor &C) {
}
__GPU__ void
NetVM::predict(Tensor &A, Tensor &B, Tensor &C) {
}
///
/// Convolution and Linear ops
///
__GPU__ void
NetVM::_conv2d() {
    U16 opt[] = { 3, 3, 1, 1, 1 };   ///> default 3x3 filter, padding=1, stride=1, dilation=1
    if (IS_OBJ(top)) {
        Tensor &v = TTOS;
        if (v.rank == 1) {
            POP();
            for (int i=0; i<5; i++) opt[i] = (U16)v.data[i];
        }
        else { ERROR("vec?"); return; }
    }
    if (IS_OBJ(top) || IS_OBJ(ss[-1])) {
        ERROR("conv2d bias c required!"); return;
    }
    U16   c    = POPi;                        ///> number of output channels
    DU    bias = POP();                       ///> convolution bias
    NN.add(L_CONV2D, c, bias, opt);
}
///
/// Batch ops
///
__GPU__ void
NetVM::nn_for() {
    Tensor &A = mmu.tensor(1, 28, 28, 1);
}
__GPU__ void
NetVM::nn_next() {
    Tensor &A = mmu.tensor(1, 28, 28, 1);
}
///
/// NN model propegation
///
__GPU__ void
NetVM::sgd() {
}
__GPU__ void
NetVM::adam() {
}
///===================================================================
/// class methods
///
/// Neural Network specific dictionary constructor
///
__GPU__ void
NetVM::init() {
    const Code prim[] = {       /// singleton, build once only
    ///@defgroup Convolution and Linear ops
    ///@{
    CODE("nn.model",  DU m = mmu.mdl2du(mmu.model(POPi)); PUSH(m)),
    CODE("nn.conv2d", _conv2d()),                          ///> (N b c [A] -- N')
    CODE("nn.linear",                                      ///> (N n -- N')
         if (!IS_OBJ(top) && !IS_OBJ(ss[-1])) {
             U16   n    = POPi;          ///> number of output channels
             DU    bias = POP();         ///> convolution bias
             NN.add(L_LINEAR, n, bias);                    ///> (N b c -- N')
         }
         else ERROR("linear: bias n required!")),
    ///@}
    ///@defgroup Activation ops
    ///@{
    CODE("nn.relu",   NN.add(L_RELU)),                     ///> (N -- N')
    CODE("nn.tanh",   NN.add(L_TANH)),                     ///> (N -- N')
    CODE("nn.sigmoid",NN.add(L_SIGMOID)),                  ///> (N -- N')
    CODE("nn.softmax",NN.add(L_SOFTMAX)),                  ///> (N -- N')
    ///@}
    ///@defgroup Pooling and Dropout ops
    ///@{
    CODE("pool.max",  U16 n = POPi; NN.add(L_MAXPOOL, n)), ///> (N n -- N')
    CODE("pool.avg",  U16 n = POPi; NN.add(L_AVGPOOL, n)), ///> (N n -- N')
    CODE("pool.min",  U16 n = POPi; NN.add(L_MINPOOL, n)), ///> (N n -- N')
    CODE("nn.dropout",                                     ///> (N p -- N')
         DU p = POP();
         NN.add(L_DROPOUT, int(100.0 * p + 0.5))),
    ///@}
    ///@defgroup Loss functions
    ///@{
    CODE("loss.nll",  {}),
    CODE("loss.mse",  {}),
    CODE("loss.ce",   {}),
    ///@}
    ///@defgroup Gradiant ops
    ///@{
    CODE("nn.sgd",    {}),
    CODE("nn.adam",   {}),
    ///@}
    ///@defgroup Batch ops
    ///@{
    CODE("nn.for",    {}),
    CODE("nn.next",   {}),
    CODE("autograd",  bool on = POPi; NN.autograd = on),
    CODE("forward",
         if (!IS_OBJ(top) || !IS_OBJ(ss[-1])) return;
         Tensor &t = mmu.du2ten(POP()); NN.forward(t)),
    CODE("backprop",
         if (!IS_OBJ(top) || !IS_OBJ(ss[-1])) return;
         Tensor &t = mmu.du2ten(POP()); NN.backprop(t)),
    CODE("predict",   {}),
    ///@}
    ///@defgroup Debugging ops
    ///@{
    CODE(">n",        DU t = POP(); NN.npush(t)),
    CODE("n@",        DU i = POPi; PUSH(NN[i])),
    CODE("network",   fout << opx(OP_NET, 0, top)),
    ///@}
    };
    const Code over[] = {           /// extended (overload) words
    CODE("flatten",
         Tensor &t = TTOS;
         if (t.is_tensor()) t.reshape(t.numel);   /// (Ta -- Ta')
         else NN.add(L_FLATTEN)),                 /// (N -- N')
    CODE("boot", mmu.clear(FIND("network") + 1))
    };
    TensorVM::init();

    mmu.append(prim, sizeof(prim)/sizeof(Code)); /// * append tensor words
    mmu.merge(over,  sizeof(over)/sizeof(Code)); /// * overload existed words
    mmu.status();
};
#endif  // T4_ENABLE_OBJ
//=======================================================================================
