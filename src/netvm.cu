#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - Neural Network Vritual Machine implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "netvm.h"

#if T4_ENABLE_OBJ
__GPU__ void
NetVM::nnop(t4_layer op) {     /// vtable dispatcher
    ///
    /// handle tensor ops (proxy)
    ///
    if (TOS1T) {
        switch (op) {
        case L_RELU:    xop1(O_RELU, DU0); break; ///> (Ta -- Ta Ta')
        case L_TANH:    xop1(O_TANH);      break; ///> (Ta -- Ta Ta')
        case L_SIGMOID: xop1(O_SIGM);      break; ///> (Ta -- Ta Ta')
        case L_FLATTEN:                           ///> (Ta -- Ta Ta')
            Tensor &t = TTOS;
            t.reshape(t.numel);            break; 
        }
        return;
    }
    ///
    /// model layer ops
    ///
    switch (op) {
    case L_CONV:  _conv(); break;
    case L_LINEAR:
        if (M2V) {                                 ///> param checking
            U16   n    = POPi;                     ///> number of output channels
            DU    bias = POP();                    ///> convolution bias
            MTOS.add(L_LINEAR, n, bias);           ///> (N b c -- N')
        }
        else ERROR("linear: bias n required!");
        break;
    case L_FLATTEN:
    case L_RELU:
    case L_TANH:
    case L_SIGMOID:
    case L_SOFTMAX: if (IS_M(top)) MTOS.add(op); break;
    case L_MAXPOOL: 
    case L_AVGPOOL:
    case L_MINPOOL: if (M1V) { U16 n = POPi; MTOS.add(op, n); } break;
    case L_DROPOUT: if (M1V) {
            U16 p = int(100.0 * POP() + 0.5); MTOS.add(op, p);
        } break;
    default: ERROR("NetVM::nnop(%d) not supported\n", op);
    }
}

__GPU__ void
NetVM::predict(Tensor &I, Tensor &P) {
}
///===================================================================
/// private functions
///
/// Batch ops
///
__GPU__ void
NetVM::nn_for() {}

__GPU__ void
NetVM::nn_next() {}
///
/// Convolution ops
/// @default: 3x3 filter, padding=1, stride=1, dilation=1
///
__GPU__ void
NetVM::_conv() {
    U16 opt[] = { 3, 3, 1, 1, 1 };   ///> default config vector
    if (TOS1T) {                     ///> if optional vector given
        Tensor &v = TTOS;
        if (v.rank == 1) {
            POP();
            for (int i=0; i<5; i++) opt[i] = (U16)v.data[i];
        }
        else { ERROR("vec?"); return; }
    }
    if (!M2V) {
        ERROR("conv2d bias c required!"); return;
    }
    U16 c    = POPi;                 ///> number of output channels
    DU  bias = POP();                ///> convolution bias
    MTOS.add(L_CONV, c, bias, opt);
}
///
/// loss functions
///
__GPU__ void
NetVM::_loss(t4_loss op) {
    if (!TOS1T || !IS_M(ss[-1])) { ERROR("target tensor?\n"); return; }
    Tensor &t = TTOS;
    DU      n = MNOS.loss(op, t);
    printf("NetVM#loss => %.3f", n);
    PUSH(n);
}
///
/// gradiant ops
///
__GPU__ void
NetVM::_sgd() {}

__GPU__ void
NetVM::_adam() {}

///===================================================================
/// class methods
///
/// Neural Network specific dictionary constructor
///
__GPU__ void
NetVM::init() {
    const Code prim[] = {                   ///> singleton, build once only
    ///@defgroup Convolution and Linear ops
    ///@{
    CODE("nn.model",                          ///> (n h w c -- N)
         if (ss.idx < 4 ||                    /// * param check
             IS_OBJ(top) || IS_OBJ(ss[-1]) ||
             IS_OBJ(ss[-2]) || IS_OBJ(ss[-3])) {
             ERROR("n h w c?\n"); return;
         }
         U16 c=POPi; U16 w=POPi; U16 h=POPi; U16 n=POPi;
         Model  &m = mmu.model();             /// * create NN model
         Tensor &t = mmu.tensor(n,h,w,c);     /// * create input tensor
         m.npush(t);                          /// * serves as the 1st layer
         PUSH(m)),
    CODE("conv2d",    nnop(L_CONV)),          ///> (N b c [A] -- N')
    CODE("linear",    nnop(L_LINEAR)),        ///> (N b n -- N')
    ///@}
    ///@defgroup Activation ops
    ///@{
    CODE("relu",      nnop(L_RELU)),          ///> (N -- N')
    CODE("tanh",      nnop(L_TANH)),          ///> (N -- N')
    CODE("sigmoid",   nnop(L_SIGMOID)),       ///> (N -- N')
    CODE("softmax",   nnop(L_SOFTMAX)),       ///> (N -- N')
    ///@}
    ///@defgroup Pooling and Dropout ops
    ///@{
    CODE("maxpool",   nnop(L_MAXPOOL)),       ///> (N n -- N')
    CODE("avgpool",   nnop(L_AVGPOOL)),       ///> (N n -- N')
    CODE("minpool",   nnop(L_MINPOOL)),       ///> (N n -- N')
    CODE("dropout",   nnop(L_DROPOUT)),       ///> (N p -- N')
    ///@}
    ///@defgroup Loss functions
    ///@{
    CODE("loss.nll",  _loss(LOSS_NLL)),       ///> (N T -- N T n)
    CODE("loss.mse",  _loss(LOSS_MSE)),       ///> (N T -- N T n)
    CODE("loss.ce",   _loss(LOSS_CE)),        ///> (N T -- N T n)
    ///@}
    ///@defgroup Gradiant ops
    ///@{
    CODE("nn.sgd",    {}),
    CODE("nn.adam",   {}),
    ///@}
    ///@defgroup Batch ops
    ///@{
    CODE("nn.for",    {}),
    CODE("nn.next",   {}),
    CODE("autograd",  if (M1V) { bool on = POPi; MTOS.autograd = on; }),
    CODE("forward", 
        if (TOS1T && IS_M(ss[-1])) {
            Tensor &t = TTOS; POP();
            MTOS.forward(t);
        }
        else ERROR("N set?\n")),
    CODE("backprop",
         if (TOS1T && IS_M(ss[-1])) {
             Tensor &t  = TTOS; POP();
             MTOS.backprop(t);
         }
         else ERROR("N tgt?\n")),
    CODE("predict",   {}),
    ///@}
    ///@defgroup Debugging ops
    ///@{
    CODE(">n",        if (M1V) { DU t = POP(); MTOS.npush(t); }),
    CODE("n@",        if (M1V) { I16 i = POPi; PUSH(mmu.view(MTOS[i])); }),
    CODE("network",   if (IS_M(top)) fout << top),
    CODE("load",
        char *fname = next_idiom();    /// retrieve file name
        fout << opx(OP_LOAD, 0, top) << fname;
        state = VM_WAIT),
    ///@}
    };
    const Code over[] = {              ///< extended (overload) words
    CODE("flatten",   nnop(L_FLATTEN)),
    CODE("boot",      mmu.clear(FIND("network") + 1))
    };
    TensorVM::init();

    mmu.append(prim, sizeof(prim)/sizeof(Code)); /// * append tensor words
    mmu.merge(over,  sizeof(over)/sizeof(Code)); /// * overload existed words
    
    VLOG1("NetVM::init ok\n");
};
#endif  // T4_ENABLE_OBJ
//===========================================================================
