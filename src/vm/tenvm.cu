#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @file
 * @brief TensorVM class - extend ForthVM, handle tensor ops implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "tenvm.h"

#if T4_ENABLE_OBJ
///
/// override with tensor handler
///
__GPU__ int
TensorVM::process(char *idiom) {
    state = QUERY;
    IU w = parse(idiom);                      /// * parse it as a word
    if (w) return 1;                          /// * success, done
    
    char *p;
    DU n = number(idiom, &p);                 /// * parse it as a literal number
    if (*p!='\0') return 0;                   /// * failed, bail

    SCALAR(n);                                /// * mask out object bit
    if (compile) {                            /// * add literal when in compile mode
        VLOG2("%d> %g\n", id, n);
        add_lit(n);                           ///> dovar (+parameter field)
    }
    else if (ten_lvl > 0) {                   /// * append literal into tensor storage
        VLOG2("%d> T[%d]=%g\n", id, ten_off, n);
        TTOS.data[ten_off++] = n;             /// * append to tensor.data (no stack used)
    }
    else {                                    ///> or, add value onto data stack
        VLOG2("%d> ss.push(%g)=%08x\n", id, n, DU2X(n));
        PUSH(n);
    }
    return 1;
}
///
/// 1-operand self math ops (destructive)
///
__GPU__ void
TensorVM::xop1(math_op op, DU v) {
    ///
    /// scalar handler
    ///
    if (!IS_OBJ(tos)) {                     /// * scalar value
        switch (op) {
        case ABS:  tos = ABS(tos);          break;
        case NEG:  tos = NEG(tos);          break;
        case EXP:  tos = EXP(tos);          break;
        case LN:   tos = LN(tos);           break;
        case LOG:  tos = LOG(tos);          break;
        case TANH: tos = TANH(tos);         break;
        case RELU: tos = MAX(tos, DU0);     break;
        case SIGM: tos = SIGMOID(tos);      break;
        case SQRT: tos = SQRT(tos);         break;
        case RCP:  tos = RCP(tos);          break;
        case SAT:  tos = SAT(tos);          break;
        case POW:  tos = POW(tos, v);       break;
        }
        SCALAR(tos);
        return;
    }
    ///
    /// single tensor handler (destructive)
    ///
    Tensor &A = TTOS;
    if (!A.is_tensor()) { ERROR("tensor?"); return; }
    
    switch (op) {        /// * defined in ~/src/util.h
    case ABS:
    case NEG:
    case EXP:
    case LN:
    case LOG:
    case TANH:
    case RELU:
    case SIGM:
    case SQRT:
    case RCP:
    case SAT:
    case FILL:
    case GFILL:
    case SCALE:
    case POW:   A.map(op, v);   break;
    case IDEN:  A.identity();   break;
    default: ERROR("tenvm#xop1(%d) not supprted\n", op);
    }
}
///
/// 2-operand tensor ops
///
__GPU__ void
TensorVM::xop2(math_op op, t4_drop_opt x) {
    static const char *opn[] = { MATH_OP };
    ///
    /// 2-operand operator (broadcasting)
    ///
    bool s0 = !IS_OBJ(tos), s1 = !IS_OBJ(ss[-1]); /// * scalar flags
    if (s0 && s1) return _ss_op(op);              /// * scalar scalar op
    if (s0) {                                     /// * tensor scaler op
        Tensor &O = _ts_op(op, x);
        VLOG2("tenvm# A[%d,%d] %s %g => O[%d,%d]\n",
              TNOS.H(), TNOS.W(), opn[op], tos, O.H(), O.W());
        if (x==T_KEEP) PUSH(O);
        else           POP();
        return;
    }
    if (s1) {                                     /// * scalar tensor op
        Tensor &O = _st_op(op, x);
        VLOG2("tenvm# %g %s A[%d,%d] => O[%d,%d]\n",
              ss[-1], opn[op], TTOS.H(), TTOS.W(), O.H(), O.W());
        if (x==T_KEEP) PUSH(O);
        else           ss.pop();
        return;
    }

    Tensor &O = _tt_op(op);                       /// * tensor tensor op
    if (O != TTOS) {
        VLOG2("tenvm# A[%d,%d] %s B[%d,%d] => O[%d,%d]\n",
             TNOS.H(), TNOS.W(), opn[op], TTOS.H(), TTOS.W(), O.H(), O.W());
        if (x==T_DROP) {
            DROP(POP());
            DROP(POP());
        }
        PUSH(O);
    }
}
///
/// 1-operand ops with new tensor created (on TOS)
///
__GPU__ void
TensorVM::xop1t(t4_ten_op op) {
    Tensor &A  = TTOS;
    if (!A.is_tensor() || A.rank != 2) { ERROR("tensor2?"); return; }
    ///
    /// single tensor handler
    ///
    Tensor &T = (op == T_INV) ? A : COPY(A);  /// * hardcopy original matrix if needed
    bool   tos = true;
    switch (op) {
    case T_INV: {
        Tensor &I = _tinv(A);                 /// * inverse A matrix
        PUSH(I);                              /// * put on TOS
        tos = false;                          /// * _tinv create its own temp
    } break;
    case T_DET: {                             /// * TODO: use PLU
        int    ns;                            ///> number of row flipping
        Tensor &P = mmu.tensor(A.H());        /// * dummy vector
        Tensor::plu(T, P, &ns);               /// * decompose A to PLU
        DU     v  = T.det();                  /// * multiply diagnal
        PUSH(ns&1 ? -v : v);                  /// * return determinant on TOS
        FREE(P);
        FREE(T);                          /// * not needed
        tos = false;
    } break;
    case T_LU:  Tensor::lu(T);    break;      /// * decompose A to LU
    case T_LUINV:
        Tensor::lu(T);                        /// * create the LU matrix
        Tensor::lu_inverse(T);    break;      /// * inverse it 
    case T_TRIU: T.triu();        break;
    case T_TRIL: T.tril();        break;
    case T_XPOS:
        T.reshape(A.W(), A.H());
        Tensor::transpose(A, T);  break;
    default:
        ERROR("tenvm#xop1t(%d) not supported\n", op);
        FREE(T);
        tos = false;
    }
    if (tos) PUSH(T);
}
///
/// 2-operand tensor ops
///
__GPU__ void
TensorVM::xop2t(t4_ten_op op, t4_drop_opt x) {
    if (!TOS2T) {
        ERROR("tenvm#xop2t TNOS TTOS required!\n");
        return;
    }
    Tensor &A = TNOS, &B = TTOS;
    switch (op){
    case T_DOT: {               ///< C = A @ B
        Tensor &C = _tdot(A, B);
        if (C != B && C != A) {
            if (x==T_DROP) {
                DROP(POP());
                DROP(POP());
            }
            PUSH(C);
        }
    } break;
    case T_DIV: {               ///< C = A @ inverse(B)
        Tensor &C = _tdiv(A, B);
        if (C != B) PUSH(C);
    } break;
    case T_SOLV: {              ///< solve B = AX
        Tensor &X = _solv(A, B);
        PUSH(X);
    } break;
    default:
        ERROR("tenvm#xop2t(%d) not supported\n", op);
    }
}
///
/// scalar-scalar ops
///
__GPU__ __INLINE__ void
TensorVM::_ss_op(math_op op) {               ///< scalar-scalar ops
    switch (op) {
    case ADD:  tos = ADD(ss.pop(), tos); break;
    case SUB:  tos = SUB(ss.pop(), tos); break;
    case MUL:  tos = MUL(ss.pop(), tos); break;
    case DIV:  tos = DIV(ss.pop(), tos); break;
    case MOD:  tos = MOD(ss.pop(), tos); break;
    case MAX:  tos = MAX(ss.pop(), tos); break;
    case MIN:  tos = MIN(ss.pop(), tos); break;
    case MUL2: tos = MUL2(ss.pop(), tos); break;
    case MOD2: tos = MOD2(ss.pop(), tos); break;
    }
    SCALAR(tos);                              /// * even +- can set LSB (rounding)
}

__GPU__ __INLINE__ Tensor&
TensorVM::_st_op(math_op op, t4_drop_opt x) { ///< scalar tensor op
    Tensor &A = TTOS;                         /// * Tensor on TOS
    DU     v  = ss[-1];                       /// * scalar as NOS
    Tensor &O = x==T_KEEP ? COPY(A) : A;      /// * make a hard copy (and parameters)
    if (op==DIV || op==SUB) {                 /// * op(scaler, tensor)
        Tensor &B = mmu.tensor(A.numel);      /// * working tensor
        B.map(FILL, v);                       /// * broadcast
        Tensor::ten_op(op, B, A, O);          /// * Hadamard ops
        FREE(B);                              /// * free working tensor
    }
    else Tensor::ten_op(op, A, v, O);         /// * broadcast_op(tensor, scalar)
    
    return O;
}

__GPU__ __INLINE__ Tensor&
TensorVM::_ts_op(math_op op, t4_drop_opt x) { ///< tensor scalar op
    Tensor &A = TNOS;                         ///< tensor on NOS
    Tensor &O = x==T_KEEP ? COPY(A) : A;      ///< make a hard copy of A
    Tensor::ten_op(op, A, tos, O);            /// * broadcast_op(tensor, scalar)
    return O;
}
///
/// op(tensor, tensor)
///
/**
  In NumPy, the behavior depends on the dimensionality of the Tensors as follows:
  - DONE: If both arguments are 2-dimensional, the matrix-matrix product is returned.
  - DONE: If both Tensors are 1-dimensional, the dot product (scalar) is returned.
  - TODO: If the first argument is 2-dimensional and the second argument is 1-dimensional, the matrix-vector product is returned.
  - TODO: If the first argument is 1-dimensional and the second argument is 2-dimensional, a 1 is prepended to its dimension for the purpose of the matrix multiply. After the matrix multiply, the prepended dimension is removed.
  - TODO: If both arguments are at least 1-dimensional and at least one argument is N-dimensional (where N > 2), then a batched matrix multiply is returned.  
    - If the first argument is 1-dimensional, a 1 is prepended to its dimension for the purpose of the batched matrix multiply and removed after.  
    - If the second argument is 1-dimensional, a 1 is appended to its dimension for the purpose of the batched matrix multiple and removed after. 
    - The non-matrix (i.e. batch) dimensions are broadcasted (and thus must be broadcastable).  
    - For example, if tensor1 is a (j x 1 x n x m) Tensor and tensor2 is a (k x m x p) Tensor, the returned tensor will be an (j x k x n x p) Tensor.
*/
__GPU__ __INLINE__ Tensor&
TensorVM::_tt_op(math_op op) {                ///< tensor-tensor ops
    Tensor &A = TNOS, &B = TTOS;
    ///
    /// tensor, tensor op
    ///
    if (!A.is_same_shape(B)) return (ERROR("dim?\n"), B);

    Tensor &O = COPY(A);                      ///< make a hard copy
    Tensor::ten_op(op, A, B, O);              /// * Hadamard ops
    if (A.rank==1) O.reshape(O.numel);
    
    return O;
}

__GPU__ Tensor&
TensorVM::_tinv(Tensor &A) {                 ///< matrix inverse
    Tensor &I = mmu.tensor(A.H(), A.W()).identity();
    Tensor &X = COPY(A);                     ///< hardcopy temp, keep A untouched
    Tensor::inverse(X, I);
    FREE(X);                                 /// * release temp 
    return I;
}

__GPU__ __INLINE__ Tensor&
TensorVM::_tdiv(Tensor &A, Tensor &B) {      ///< tensor division
    U16 m = A.H(), ka = A.W(), kb = B.H(), n = B.W();
    if (kb != n || ka != kb) return B;       /// * B square?

    Tensor &I = _tinv(B);
    Tensor &O = mmu.tensor(m, n);
    Tensor::mm(A, I, O);                     /// A * B^-1
    FREE(I);
    
    return O;
}

__GPU__ __INLINE__ Tensor&
TensorVM::_tdot(Tensor &A, Tensor &B) {      ///< A x B tensor dot product
    if (B.rank==1 &&                         ///> dot(vector, vector)
        A.rank==1 && A.numel==B.numel) {
        DU v = A.dot(B);
        PUSH(v);
        VLOG1("tenvm# A[%d] · B[%d] => %g\n", A.H(), B.H(), v);
        return B;                            /// * non-tensor
    }
    if (B.rank==1 && A.W()==B.numel) {       ///> inner(tensor, vector)
        Tensor &O = mmu.tensor(A.H());
        Tensor::mm(A, B, O);
        return O;
    }
    if (A.W()==B.H()) {                      /// * tensor @ tensor
        Tensor &O = mmu.tensor(A.H(), B.W());
        Tensor::mm(A, B, O);
        return O;
    }
    ERROR("A.W!=B.H dim?");
    
    return A;                                /// * i.e. skip in xop2
}

__GPU__ __INLINE__ Tensor&
TensorVM::_solv(Tensor &B, Tensor &A) {      /// Note: A, B flipped 
    U16 m = A.H(), k = A.W(), n = B.H();     /// B[3,1] = A[3,3] * X
    VLOG1("tenvm# solv B[%d] = [%d,%d]*X\n", n, m, k);
    
    if (B.rank!=1 || m!=k || k!=n) return B;
    
    Tensor &I = _tinv(A);
    Tensor &O = mmu.tensor(k);               /// resultant vector
    Tensor::mm(I, B, O);                     /// O = A^-1 x B
    FREE(I);
    
    return O;
}

__GPU__ __INLINE__ void
TensorVM::_gemm() {                          ///< blas GEMM
    if (!TOS3T) { ERROR("tensors?"); return; }
    
    Tensor &O = TTOS, &B = TNOS, &A = (Tensor&)mmu.du2obj(ss[-2]);
    DU     b  = ss[-3];
    DU     a  = ss[-4];
    U16    m  = A.H(), k = A.W(), n = B.W();
    if (k == B.H() && m == O.H() && n == O.W()) {
        Tensor &X = COPY(O);                 /// * hard copy O tensor
        Tensor::gemm(A, B, X, a, b);
        PUSH(X);
    }
    else ERROR("dim?");
}

__GPU__ void
TensorVM::_tprint(DU v) {
    sys.dot(DOT, v);                          /// * send v to output stream
    if (IS_OBJ(v)) {
        mmu.mark_free(v);                     /// * mark to release by host
        state = HOLD;                         /// * forced flush (wasteful but no dangling objects)
    }
}    
__GPU__ void
TensorVM::_pickle(bool save) {
    U8   mode= FAM_WO;                        ///< file mode (W/O,R/W)|BIN
    
    if (ss.idx > 1 && IS_OBJ(ss[-2])) { /* OK */ }
    else if (ss.idx > 2 && IS_OBJ(ss[-3])) mode |= POPi;
    else { ERROR("tensor adr len [mode]?\n"); return; }
    
    IU   len  = POPi;                         ///< string length (not used for now)
    IU   adr  = POPi;                         ///< address to pmem
    char *fn  = (char*)MEM(adr);              ///< pointer to string on PAD
    
    sys.op(OP_TSAVE, mode, tos);              /// * issue save command
    sys.op_fn(fn);                            /// * append filename
    state = HOLD;                             /// * return to CPU
}
///
/// Tensor Vocabulary
///
__GPU__ void
TensorVM::init() {
    if (id !=0) return;                       /// * only needed once
    ForthVM::init();
    ///
    ///@defgroup Tensor creation ops
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("vector",                            ///< allocate a vector
         IU sz = POPi;
         PUSH(mmu.tensor(sz)));
    CODE("matrix",                            ///< allocate a matrix
         IU w = POPi; IU h = POPi;
         PUSH(mmu.tensor(h, w)));
    CODE("tensor",                            ///< allocate a NHWC tensor
         IU c = POPi; IU w = POPi; IU h = POPi; IU n = POPi;
         PUSH(mmu.tensor(n, h, w, c)));
    CODE("vector{",                           ///< create a vector with literals
         IU sz = POPi;
         PUSH(mmu.tensor(sz));
         ten_off = 0; ten_lvl = 1);
    CODE("matrix{",                           ///< create a matrix with literals
         IU w = POPi; IU h = POPi;
         PUSH(mmu.tensor(h, w));
         ten_off = 0; ten_lvl = 1);
    CODE("view",   PUSH(DUP(tos)));           ///< create a view of a tensor
    CODE("copy",   PUSH(COPY(tos)));          ///< create a hardcopy of a tensor
    ///@}
    ///@defgroup Tensor shape ops
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("flatten",                           ///< reshape as a vector (1-D array)
         Tensor &t = TTOS;
         t.reshape(t.numel));
    CODE("reshape2",                          ///< reshape as matrix(h,w)
         IU w = POPi; IU h = POPi;
         TTOS.reshape(h, w));
    CODE("reshape4",                          ///< reshape as Tensor(NHWC)
         IU c = POPi; IU w = POPi; IU h = POPi; IU n = POPi;
         TTOS.reshape(n, h, w, c));
    CODE("same_shape?",
         if (IS_OBJ(tos) && IS_OBJ(ss[-1])) {
             Tensor &A=TTOS; Tensor &B=TNOS; PUSH(BOOL(A.is_same_shape(B)));
         }
         else ERROR("TOS,NOS tensors?"));
    ///@}
    ///@defgroup Tensor fill ops
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("={",                                 ///< (n -- ) or ( -- )
         ten_off = IS_OBJ(tos) ? 0 : POPi;
         ten_lvl = IS_OBJ(tos) ? 1 : 0);
    CODE("zeros", xop1(FILL, DU0));            ///< fill tensor with 0s
    CODE("ones",  xop1(FILL, DU1));            ///< fill tensor with 1s
    CODE("full",  xop1(FILL, POP()));          ///< fill tensor with a value
    CODE("gradfill", xop1(GFILL, DU1));        ///< gradient fill a tensor
    CODE("eye",   xop1(IDEN));                 ///< fill 1s in diag
    CODE("rand",  tos = sys.rand(tos, UNIFORM));   ///< uniform randomize a tensor or number
    CODE("randn", tos = sys.rand(tos, NORMAL));    ///< normal dist. randomize a tensor
    ///@}
    ///@defgrup Tensor slice and dice
    ///@{
    CODE("normalize",
         DU std = POP(); DU avg = POP();
         if (TOS1T) { TTOS.normalize(std, avg); });
    CODE("sum", if (TOS1T) PUSH(TTOS.sum()));
    CODE("avg", if (TOS1T) PUSH(TTOS.avg()));
    CODE("std", if (TOS1T) PUSH(TTOS.std()));
    CODE("{",   if (TOS1T && ten_lvl > 0) ++ten_lvl);
    CODE("}",   if (TOS1T && ten_lvl > 0) --ten_lvl);
    CODE("slice",
         IU y1 = POPi; IU y0 = POPi; IU x1 = POPi; IU x0 = POPi;
         if (TOS1T) {
             Tensor &t0 = TTOS;
             Tensor &t1 = mmu.slice(t0, x0, x1, y0, y1);
             PUSH(t1);
         });
    CODE("t@", 
         if (!IS_OBJ(ss[-1]) && IS_OBJ(tos)) {
             IU i = POPi; DU v = TTOS[i];
             SCALAR(v);
             PUSH(v);
         });
    CODE("t!",  DU v = POP(); IU i = POPi; if (IS_OBJ(tos)) TTOS[i]=v);
    ///@}
    ///@defgroup 1-tensor ops in-place (i.e. destructive, as in Forth)
    ///@{
    CODE("exp",       xop1(EXP));             ///< (A -- A')
    CODE("ln",        xop1(LN));                      
    CODE("log",       xop1(LOG));                     
    CODE("tanh",      xop1(TANH));
    CODE("relu",      xop1(RELU));
    CODE("sigmoid",   xop1(SIGM));
    CODE("sqrt",      xop1(SQRT));
    CODE("1/x",       xop1(RCP));             ///< reciprocal
    CODE("sat",       xop1(SAT));
    CODE("pow",       xop1(POW, POP()));      ///< scale tensor with TOS
    ///@}
    ///@defgroup 1-tensor ops that create new tensor
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("inverse",   xop1t(T_INV));          ///< (A -- A Ai')   matrix inversion (GaussJordan)
    CODE("det",       xop1t(T_DET));          ///< (A -- A d)     matrix determinant
    CODE("lu",        xop1t(T_LU));           ///< (A -- A A')    LU decomposition
    CODE("luinv",     xop1t(T_LUINV));        ///< (A -- A A')    inverse the LU matrix
    CODE("upper",     xop1t(T_TRIU));         ///< (A -- A A')    upper triangle
    CODE("lower",     xop1t(T_TRIL));         ///< (A -- A A')    lower triangle
    CODE("transpose", xop1t(T_XPOS));         ///< (A -- A At)    matrix transpose
    ///@}
    ///@defgroup 2-tensor matrix ops
    ///@{
    CODE("+=",        xop2(ADD, T_DROP));     ///< (A B -- C)
    CODE("-=",        xop2(SUB, T_DROP));
    CODE("*=",        xop2(MUL, T_DROP));
    CODE("/=",        xop2(DIV, T_DROP));
    CODE("@=",        xop2t(T_DOT, T_DROP));  ///< (A B -- C)
    CODE("matmul",    xop2t(T_DOT));          ///< (A B -- A B C) matrix multiply
    CODE("matdiv",    xop2t(T_DIV));          ///< (A B -- A B C) matrix divide
    CODE("solve",     xop2t(T_SOLV));         ///< (B A -- B A X) solve B = AX
    CODE("gemm",      _gemm());               ///< (a b A B C -- a b A B C') GEMM (C updated)
    ///@}
    ///@defgroup Tensor persistance
    ///@brief - stick to PyTorch naming when possible
    ///@{
    CODE("bin",       PUSH(FAM_RAW));         ///< raw/binary file
    CODE("w/o",       PUSH(FAM_WO));          ///< write-only file
    CODE("r/w",       PUSH(FAM_RW));          ///< read-write file
    CODE("save",      _pickle(true));         ///< ( T fn len -- T ) save tensor to a file
    CODE("load",      _pickle(false));        ///< ( T fn -- T' ) fill a tensor from file
    ///
    /// vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv
    ///
    ///@defgroup redefined tensor ops
    ///@{
    CODE("boot",      mmu.clear(FIND((char*)"load") + 1));
    CODE(".",         _tprint(POP()));           ///< print TOS
    CODE("+",         xop2(ADD, T_KEEP));
    CODE("-",         xop2(SUB, T_KEEP));
    CODE("*",         xop2(MUL, T_KEEP));
    CODE("/",         xop2(DIV, T_KEEP));
    CODE("abs",       xop1(ABS));
    CODE("negate",    xop1(NEG));
    CODE("@",
         if (TOS2T) xop2t(T_DOT);             ///< matrix @ product
         else {
             DU v = mmu.rd(POPi);
             PUSH(DUP(v));
         });
    CODE("max",
         if (IS_OBJ(tos)) PUSH(TTOS.max());
         else xop2(MAX));
    CODE("min",
         if (IS_OBJ(tos)) PUSH(TTOS.min());
         else xop2(MIN));
    ///@}
    TRACE("TensorVM[%d]::init ok, sizeof(Tensor)=%ld\n", id, sizeof(Tensor));
}
#endif  // T4_ENABLE_OBJ
//==========================================================================
