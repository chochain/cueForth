#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - Neural Network Model Backward Propagation implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "model.h"

#if T4_ENABLE_OBJ
///
/// convolution filter derivatives
/// TODO: stride, dilation, [C1]NCHW filter
///
template<int TS, int KS>         ///> tile size, kernel size
__KERN__ void k_dconv2d(
    DU *I, DU *F, DU *DF, DU *O, ///> input I[HxW], F,DF[KSxKS], output O[HxW]
    int H, int W, int C0         ///< H1==H0, W1==W0, output Channels
    ) {
    __shared__ DU it[T4_WARP_SQ];                    ///< input cache [16x16]
    __shared__ DU ot[T4_WARP_SQ];                    ///< output cache [16x16]
    __shared__ DU df[TS * TS * KS * KS];             ///< df cache [12x12x3x3]
    
    const int tx = threadIdx.x, j1 = tx + blockIdx.x * TS;
    const int ty = threadIdx.y, i1 = ty + blockIdx.y * TS;
    const int C1 = gridDim.z,   c1 = blockIdx.z;     ///< input channels
    const int z1 = c1 + (j1 + i1 * W) * C1;          ///< input array index
    const int zt = tx + ty * T4_WARP_SZ;             ///< tile index
    ///
    /// process z1, i.e. [TS, TS, C1] cells per kernel call
    ///
    const int i0 = i1 - int(KS / 2);                 ///< dY coordinates
    const int j0 = j1 - int(KS / 2);

    auto g = cg::this_thread_block();                ///< group all threads

    it[zt] = (i1 < H && j1 < W) ? I[z1] : DU0;       ///< cached input tile
    g.sync();
    
    for (int c0 = 0; c0 < C0; c0++) {                ///< each dY channel
        ot[zt] =                                     /// * cache dY tile
            (i0 >= 0 && i0 < H && j0 >= 0 && j0 < W) /// * with zero padding
            ? O[c0 + (j0 + i0 * W) * C0] : DU0;      /// * by channel
        g.sync();                                    /// * smem write barrier
        ///
        /// dX = sum(F * dY)
        /// dF = sum(dY * X)
        ///
        DU sum = DU0;
        const int zf = (c1 + c0 * C1) * KS * KS;     ///< filter index
        if (tx < TS && ty < TS) {                    /// * within tile [12x12]
            DU *fx = &F[zf + C1 * (KS * KS - 1)];    ///< F[KS-1,KS-1] rot180
            DU *dfx= &df[(tx + ty * TS) * KS * KS];  ///< df cache ptr
            for (int y = 0; y < KS; y++) {           /// * process one cell
                for (int x = 0; x < KS; x++, fx -= C1) {
                    int k = zt + x + y * T4_WARP_SZ;
                    sum      += (*fx) * ot[k];       /// * dX += F * dY
                    *(dfx++) =  ot[k] * it[k];       /// * df = dY * X
                }
            }
            if (i1 < H && j1 < W) {                  /// * update input matrix
                if (c0==0) I[z1] = sum;              /// * no bias
                else       I[z1] += sum;             /// * accumulate all c0
            }
        }
        g.sync();                                    /// * d read barrier
        ///
        /// collect dF (= dY * X), KS * KS threads
        ///
        if (tx < KS && ty < KS) {                    /// * TODO: CDP scan
            DU *DFx = &DF[c1 + (tx + (ty + c0 * KS) * KS) * C1];
            DU *dfx = &df[tx + ty * KS];
            for (int i = 0; i < TS * TS; i++, dfx += KS * KS) {
                *DFx += *dfx;                        /// dF += df (= dY * X)
            }
        }
        g.sync();                           /// * d read barrier
    }
}
template<int KS>                            /// kernel size
__KERN__ void k_dpool(
    DU *I, DU *O,                           ///< input, output buffers
    int H0, int W0,                         ///< output HW (C1==C0)
    t4_layer op
    ) {
    const int j0 = threadIdx.x + blockIdx.x * blockDim.x;
    const int i0 = threadIdx.y + blockIdx.y * blockDim.y;
    const int c  = blockIdx.z, C = gridDim.z;
    const int z0 = j0 + i0 * W0;            ///< output matrix index
    const int z1 = (j0 + i0 * W0 * KS) * KS;///< input tensor index
    const int zc = c + z0 * C;              ///< output tensor index
    auto g = cg::this_thread_block();
    
    if (i0 < H0 && j0 < W0 && c < C) {
        DU *ix = &I[c + z1 * C], *t = ix;
        DU2 v  = (op != L_AVGPOOL) ? *ix : O[zc] / (KS * KS);
        for (int y = 0; y < KS; y++) {      /// * handle one kernel
            for (int x = 0; x < KS; x++) {
                DU dx = *ix;
                switch (op) {
                case L_MAXPOOL:
                    *ix = DU0;              /// * zero out all elements
                    if (dx > v) { v = dx; t = ix; }  break;
                case L_AVGPOOL: *ix = v;             break;
                case L_MINPOOL:
                    *ix = DU0;
                    if (dx < v) { v = dx; t = ix; }  break;
                }
                ix += C;
            }
            ix += (W0 - 1) * KS * C;
        }
        if (op != L_AVGPOOL) *t = O[zc];   /// * update arg cell
    }
    g.sync();
}

__KERN__ void k_dfilter(
    DU *I, DU *F, DU *O,                   ///< input, filter, output
    int H, int W                           ///< H1==H0, W1==W0 (C1==C0)
    ) {
    const int j1 = threadIdx.x + blockIdx.x * blockDim.x;
    const int i1 = threadIdx.y + blockIdx.y * blockDim.y;
    const int c  = blockIdx.z, C = gridDim.z;
    const int z1 = c + (i1 + j1 * W) * C;
    auto g = cg::this_thread_block();
    
    if (i1 < H && j1 < W && c < C) {
        I[z1] = (F[z1] > DU0) ? O[z1] : DU0;
    }
    g.sync();
}

__GPU__ Model&
Model::backprop(Tensor &tgt) {
    Tensor &nx = (*this)[numel - 1];
    if (!nx.is_same_shape(tgt)) {
        ERROR("Model#backprop target dim?\n");
        return *this;
    }
    ///
    /// cascade execution layer by layer backward
    ///
    Tensor::copy(tgt, nx);
    for (U16 i = numel - 2; i > 0; i--) {
        Tensor &in = (*this)[i], &out = (*this)[i + 1];
        printf("%2d> %s [%d,%d,%d]\tp=%2d <= out'Σ=%6.2f [%d,%d,%d] ",
            i, d_nname(in.grad_fn),
            in.H(), in.W(), in.C(), in.parm,
            out.sum(), out.H(), out.W(), out.C());
        _bstep(in, out);
        printf("\n");
    }
    return *this;
}
/// ========================================================================
/// private methods 
///
#define TILE3    (T4_WARP_SZ - 3 + 1)      /** 14 */
#define TILE5    (T4_WARP_SZ - 5 + 1)      /** 12 */

__GPU__ void
Model::_bstep(Tensor &in, Tensor &out) {
    DU   *d1 = in.data, *d0 = out.data;              ///< input, output data
    int  H1 = in.H(), W1 = in.W(), C1 = in.C();      ///< input HWC
    dim3 blk(T4_WARP_SZ, T4_WARP_SZ, 1);
    dim3 grd(TGRID(W1, H1, C1, blk));

    auto dump = [](DU *v, int H, int W, int C) {
        for (int k = 0; k < C; k++) {
            printf("\nC=%d ---\n", k);
            DU sum = DU0;
            for (int i = 0; i < H; i++) {
                DU isum = DU0;
                for (int j = 0; j < W; j++) {
                    DU x = v[k + (j + i * W) * C];
                    isum += x;
                    printf("%5.2f", x);
                }
                printf(" Σ=%6.3f\n", isum);
                sum += isum;
            }
            printf(" ΣΣ=%6.3f\n", sum);
        }
    };
    auto dump_dbdf = [C1](DU *df, DU *db, int C0, int fsz) {
        DU sum = DU0;
        printf("\n\tdb=");
        for (int c0 = 0; c0 < C0; c0++) {
            printf("%6.3f ", db[c0]);
            sum += db[c0];
        }
        printf("Σ=%6.3f", sum);
        for (int c1 = 0; c1 < C1; c1++) {
            printf("\n\tdf[%d]=", c1);
            sum = DU0;
            for (int i=0; i<fsz; i++, df++) {
                sum += *df;
                printf("%6.3f", *df);
            }
            printf(" Σ=%6.3f", sum);
        }
    };
    auto conv = [d1, d0, H1, W1, C1, blk](int C0, int ks, DU *f, DU *df, DU *db) {
        dim3 g3((W1 + TILE3 - 1) / TILE3, (H1 + TILE3 - 1) / TILE3, C1);
        dim3 g5((W1 + TILE5 - 1) / TILE5, (H1 + TILE5 - 1) / TILE5, C1);
        switch (ks) {
        case 3: k_dconv2d<TILE3,3><<<g3,blk>>>(d1, f, df, d0, H1, W1, C0); break;
        case 5: k_dconv2d<TILE5,5><<<g5,blk>>>(d1, f, df, d0, H1, W1, C0); break;
        default: return -1;
        }
        /// accumulate dB = sum(dY), TODO: CDP
        for (int c0 = 0; c0 < C0; c0++, db++) {
            DU *ox = d0 + c0;
            for (int k = 0; k < H1 * W1; k++, ox+=C0) *db += *ox;
        }
        return 0;
    };
    ///
    /// layer function dispatcher
    ///
    t4_layer fn = in.grad_fn;                 ///< layer function
    switch(fn) {
    case L_CONV:   {
        Tensor &f = *in.grad[0], &df = *in.grad[2]; ///< filter tensor
        Tensor &b = *in.grad[1], &db = *in.grad[3]; ///< bias tensor
        const int C1 = f.parm, Nf = f.N(), Hf = f.H(), Wf = f.W(), Cf = f.C();
        printf(" f[%d][%d,%d,%d,%d], b[%d]", C1, Nf, Hf, Wf, Cf, b.numel);
        if (conv(out.C(), Hf, f.data, df.data, db.data)) {
            ERROR("model_back#conv kernel_size %d not supported\n", Hf);
        }
        dump_dbdf(df.data, db.data, out.C(), Nf * Hf * Wf * Cf);
        printf("\nin[%d,%d,%d]=", H1, W1, C1); dump(d1, H1, W1, C1);
    } break;
    case L_LINEAR: {                          ///< out = w @ in + b
        Tensor &w  = *in.grad[0];             ///< weight tensor
        Tensor &dw = *in.grad[2];             ///< d_weight tensor
        Tensor &db = *in.grad[3];             ///< d_bias tensor
        int H0 = out.H(), M = w.H(), N = w.W();///< fully connected dimensions
        /// dw += out[10,1] @ in^t[1,49]
        /// in = w^t[49,10] @ out[10,1]
        printf("\n\tdw[%d,%d] += out'[%d,1] @ in^t[1,%d]", M, N, H0, H1);
        printf("\n\tin[%d, 1]  = w^t[%d,%d] @ out'[%d,1]", H1, N, M, H0);
        db += out;
        Tensor::mm(out, in, dw, (t4_mm_opt)(MM_INC | MM_B_TXP));
        Tensor::mm(w, out, in, MM_A_TXP);
    } break;
    case L_FLATTEN: Tensor::copy(out, in); break;  /// * pass dY to X
    case L_RELU:    k_dfilter<<<grd,blk>>>(d1, d1, d0, H1, W1); break;
    case L_TANH:    break;
    case L_SIGMOID: break;
    case L_SOFTMAX: in -= out; /* delta */ break;
    case L_MAXPOOL:
    case L_AVGPOOL: 
    case L_MINPOOL: {
        U16 ks = in.parm;                              ///< pooling kernel size
        U16 W0 = out.W(), H0 = out.H(), C0 = out.C();  ///< output dimensions
        dim3 g(TGRID(W0, H0, C0, blk));
        switch(ks) {                           
        case 0x2: k_dpool<2><<<g,blk>>>(d1, d0, H0, W0, fn); break;
        case 0x3: k_dpool<3><<<g,blk>>>(d1, d0, H0, W0, fn); break;
        }
        if (H1 < 10) dump(d1, W1, H1, C1);
    } break;
    case L_DROPOUT:
        Tensor &msk = *in.grad[0];             ///< dropout mask
        k_dfilter<<<grd,blk>>>(d1, msk.data, d0, H1, W1);
        break;
    }
}
#endif  // T4_ENABLE_OBJ
//==========================================================================
