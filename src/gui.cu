#include "hip/hip_runtime.h"
/** -*- c++ -*- 
 * @File
 * @brief - tensorForth GUI - static, OpenGL in freeglut
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include <map>
#include "gui.h"

#define T4_VU_REFRESH_DELAY     100              /** ms     */
#define T4_VU_X_CENTER          600              /** pixels */
#define T4_VU_Y_CENTER          100              /** pixels */
#define T4_VU_OFFSET            40               /** pixels */

namespace T4GUI {
    
typedef std::map<int, Vu*> VuMap;
VuMap   vu_map;
GLuint  shader_id = 0;         ///< floating point shader

void _vu_set(int id, Vu *vu) {
    vu_map[id] = vu;
    printf("vu[%d] ", id);
}

Vu *_vu_get(int id) {
    VuMap::iterator it = vu_map.find(id);
    return (it == vu_map.end()) ? NULL : it->second;
}

Vu *_vu_now() {
    return _vu_get(glutGetWindow());
}
///
/// default texture shader for displaying floating-point
///
void _compile_shader() {
    static const char *code =
        "!!ARBfp1.0\n"
        "TEX result.color, fragment.texcoord, texture[0], 2D; \n"
        "END";

    if (shader_id) return;    ///< already compiled
    
    printf("\tShader...");
    glGenProgramsARB(1, &shader_id);
    glBindProgramARB(GL_FRAGMENT_PROGRAM_ARB, shader_id);
    glProgramStringARB(
        GL_FRAGMENT_PROGRAM_ARB, GL_PROGRAM_FORMAT_ASCII_ARB,
        (GLsizei)strlen(code), (GLubyte*)code);
   
    GLint xpos;
    glGetIntegerv(GL_PROGRAM_ERROR_POSITION_ARB, &xpos); /// CUDA GL extension
    if (xpos != -1) {
        const GLubyte *errmsg = glGetString(GL_PROGRAM_ERROR_STRING_ARB);
        fprintf(stderr, "Shader error at: %d\n%s\n",  (int)xpos, errmsg);
    }
    printf("compiled\n");
}

void _close_and_switch_vu() {
    int id = glutGetWindow();
    Vu *vu = _vu_get(id);
    glutDestroyWindow(id);
    
    hipGraphicsUnregisterResource(vu->pbo); GPU_CHK();
    vu_map.erase(id);                        /// * erase by key
    printf("\tvu[%d] released...", id);
    
    if (vu_map.size() > 0) {
        id = vu_map.rbegin()->first;
        glutSetWindow(id);                   /// * use another window
        printf("vu[%d] now active\n", id);
    }
    else printf("no avtive vu, shutting down...\n");
}

void _shutdown() {
    if (vu_map.size() > 0) return;
    
//    glDeleteProgramsARB(1, &shader_id);      /// remove shader
}

void _paint(int w, int h) {
    // Common display code path
    glClear(GL_COLOR_BUFFER_BIT);
    glTexSubImage2D(
        GL_TEXTURE_2D, 0, 0, 0, w, h,
        GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBegin(GL_TRIANGLES);
    glTexCoord2f(0, 0);       /// texture coordinates:
    glVertex2f(-1, -1);       ///     (0,0) lower left
    glTexCoord2f(2, 0);       ///     (1,1) upper right
    glVertex2f(+3, -1);
    glTexCoord2f(0, 2);
    glVertex2f(-1, +3);
    glEnd();
    glFinish();
    
    glutSwapBuffers();
    glutReportErrors();
}

void _mouse(int button, int state, int x, int y) {
    /// button: GLUT_LEFT_BUTTON, GLUT_MIDDLE_BUTTON, GLUT_RIGHT_BUTTON
    /// state:  GLUT_UP=1, GLUT_DOWN=0
    /// x,y: mouse location in window relative coordinates
    switch (button) {
    case GLUT_LEFT_BUTTON:
    case GLUT_MIDDLE_BUTTON:
    case GLUT_RIGHT_BUTTON:
        Vu *vu = _vu_now();
        if (vu) vu->mouse(button, state, x, y);
        break;
    }
}

void _keyboard(unsigned char k, int /*x*/, int /*y*/) {
    switch (k) {
    case 27:     // ESC
    case 'q':
    case 'Q': _close_and_switch_vu(); break;
    default: 
        Vu *vu = _vu_now();
        if (vu) vu->keyboard(k);
        break;
    }
}

void _display() {
    Vu  *vu = _vu_now();
    if (!vu) return;
    
    TColor *d_dst = NULL;
    size_t bsz;

    hipGraphicsMapResources(1, &vu->pbo, 0);   GPU_CHK();  /// lock
    hipGraphicsResourceGetMappedPointer(
        (void**)&d_dst, &bsz, vu->pbo);         GPU_CHK();

    vu->display(d_dst);         /// update buffer content
    
    hipGraphicsUnmapResources(1, &vu->pbo, 0); GPU_CHK();  /// unlock
    
    _paint(vu->X, vu->Y);
}

void _refresh(int) {
    if (!glutGetWindow()) return;
    
    glutPostRedisplay();       /// mark current window for refresh
    glutTimerFunc(T4_VU_REFRESH_DELAY, _refresh, 0);
}

void _bind_texture(Vu *vu) {
    GLuint gl_pbo, gl_tex;
    GLuint format = GL_RGBA8, depth = GL_RGBA;
    /*
    /// See OpenGL Core 3.2 internal format
    switch (vu->N) {
    case 1:  format = GL_R8;    depth = GL_RED;  break;
    case 2:  format = GL_RG8;   depth = GL_RG;   break;
    case 3:  format = GL_RGB8;  depth = GL_RGB;  break;
    default: format = GL_RGBA8; depth = GL_RGBA;
    }
    */
    printf("\tTexture");
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &gl_tex);
    glBindTexture(GL_TEXTURE_2D, gl_tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S,     GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T,     GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, format,
                 vu->X, vu->Y, 0, depth, GL_UNSIGNED_BYTE, NULL);
    printf("[%d] created\n", gl_tex);

    printf("\tPBO");
    int bsz = vu->X * vu->Y * sizeof(uchar4);
    glGenBuffers(1, &gl_pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, bsz, vu->h_tex, GL_STREAM_COPY);
    // While a PBO is registered to CUDA, it can't be used
    // as the destination for OpenGL drawing calls.
    // But in our particular case OpenGL is used
    // to display the content of the PBO, specified by CUDA kernels,
    // so we need to register/unregister it (once only).
    hipGraphicsGLRegisterBuffer(
        &vu->pbo, gl_pbo, cudaGraphicsMapFlagsWriteDiscard);
    GPU_CHK();
    printf("[%d] created\n", gl_pbo);
}

extern "C" int gui_init(int *argc, char **argv) {
    printf("\nGLUT...");
    glutInit(argc, argv);                /// * consumes X11 input parameters
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    printf("initialized\n");
    
    return 0;
}

extern "C" int gui_add(Vu &vu) {
    int z = T4_VU_OFFSET * vu_map.size();
    glutInitWindowPosition(T4_VU_X_CENTER + z - (vu.X / 2), T4_VU_Y_CENTER + z);
    glutInitWindowSize(vu.X, vu.Y);
    ///
    /// create window for img
    ///
    printf("\tWindow...");
    int id = glutCreateWindow(T4_APP_NAME); /// * create named window (as current)
    _vu_set(id, &vu);
    ///
    /// * set callbacks (for current window, i.e. id)
    ///
    glutDisplayFunc(_display);
    glutKeyboardFunc(_keyboard);
    glutMouseFunc(_mouse);
    glutTimerFunc(T4_VU_REFRESH_DELAY, _refresh, 0);
    glutCloseFunc(_shutdown);
    printf("created\n");

    _bind_texture(&vu);
//    _compile_shader();                      /// load float shader
    
    return 0;
}

extern "C" int gui_loop() {
    glutMainLoop();
    return 0;
}

} // namespace T4GUI
