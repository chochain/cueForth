/** -*- c++ -*- 
 * @File
 * @brief - tensorForth GUI - static, OpenGL in freeglut
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include <map>
#include "gui.h"

#define REFRESH_DELAY     10              /** ms */
#define BUFFER_DATA(i)    ((char*)0 + i)

namespace T4GUI {
    
typedef std::map<int, Vu*> VuMap;
VuMap   vu_map;
GLuint  shader_id = 0;         ///< floating point shader

void _vu_set(int id, Vu *vu) {
    vu_map[id] = vu;
}

Vu *_vu_get() {
    int id = glutGetWindow();
    VuMap::iterator vu = vu_map.find(id);
    return (vu == vu_map.end()) ? NULL : vu->second;
}

// shader for displaying floating-point texture
void _compile_shader() {
    static const char *code =
        "!!ARBfp1.0\n"
        "TEX result.color, fragment.texcoord, texture[0], 2D; \n"
        "END";

    if (shader_id) return;    ///< already compiled
    
    printf("\tShader...");
    glGenProgramsARB(1, &shader_id);
    glBindProgramARB(GL_FRAGMENT_PROGRAM_ARB, shader_id);
    glProgramStringARB(
        GL_FRAGMENT_PROGRAM_ARB, GL_PROGRAM_FORMAT_ASCII_ARB,
        (GLsizei)strlen(code), (GLubyte*)code);
    
    GLint xpos;
    glGetIntegerv(GL_PROGRAM_ERROR_POSITION_ARB, &xpos);
    if (xpos != -1) {
        const GLubyte *errmsg = glGetString(GL_PROGRAM_ERROR_STRING_ARB);
        fprintf(stderr, "Shader error at: %d\n%s\n",  (int)xpos, errmsg);
    }
    printf("compiled\n");
}

void _cleanup() {
    Vu *vu = _vu_get();
    hipGraphicsUnregisterResource(vu->pbo); GPU_CHK();
    
    if (vu_map.size()==0) {
        glDeleteProgramsARB(1, &shader_id);   /// remove shader
    }
}

void _gl_codepath(int w, int h) {
    // Common display code path
    glClear(GL_COLOR_BUFFER_BIT);
    glTexSubImage2D(
        GL_TEXTURE_2D, 0, 0, 0, w, h, GL_RGBA,
        GL_UNSIGNED_BYTE, BUFFER_DATA(0));
    glBegin(GL_TRIANGLES);
    glTexCoord2f(0, 0);       /// texture coordinates:
    glVertex2f(-1, -1);       ///     (0,0) lower left
    glTexCoord2f(2, 0);       ///     (1,1) upper right
    glVertex2f(+3, -1);
    glTexCoord2f(0, 2);
    glVertex2f(-1, +3);
    glEnd();
    glFinish();
}

void _keyboard(unsigned char k, int /*x*/, int /*y*/) {
    switch (k) {
    case 27:     // ESC
    case 'q':
    case 'Q': glutDestroyWindow(glutGetWindow()); return;
    default: 
        Vu *vu = _vu_get();
        if (vu) vu->keyboard(k);
        break;
    }
}

void _display() {
    Vu  *vu = _vu_get();
    if (!vu) return;
    
    TColor *d_dst = NULL;
    size_t num_bytes;

    hipGraphicsMapResources(1, &vu->pbo, 0);   GPU_CHK();
    hipGraphicsResourceGetMappedPointer(
        (void **)&d_dst, &num_bytes, vu->pbo);  GPU_CHK();

    vu->display(d_dst);
    
    hipGraphicsUnmapResources(1, &vu->pbo, 0); GPU_CHK();
    _gl_codepath(vu->W, vu->H);
    
    glutSwapBuffers();
    glutReportErrors();
}

void _refresh(int) {
    if (glutGetWindow()) {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, _refresh, 0);
    }
}

void _bind_texture(Vu *vu) {
    int    buf_sz = vu->W * vu->H * 4;
    GLuint gl_pbo, gl_tex;

    printf("\tTexture...");
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &gl_tex);
    glBindTexture(GL_TEXTURE_2D, gl_tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S,     GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T,     GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8,
                 vu->W, vu->H, 0, GL_RGBA, GL_UNSIGNED_BYTE, vu->h_src);
    printf("created\n");

    printf("\tPBO...");
    glGenBuffers(1, &gl_pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, buf_sz, vu->h_src, GL_STREAM_COPY);
    // While a PBO is registered to CUDA, it can't be used
    // as the destination for OpenGL drawing calls.
    // But in our particular case OpenGL is used
    // to display the content of the PBO, specified by CUDA kernels,
    // so we need to register/unregister it (once only).
    hipGraphicsGLRegisterBuffer(
        &vu->pbo, gl_pbo, cudaGraphicsMapFlagsWriteDiscard);
    GPU_CHK();
    printf("created\n");
}

extern "C" int gui_init(int *argc, char **argv) {
    printf("\nGLUT...");
    glutInit(argc, argv);                /// * consumes X11 input parameters
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    printf("initialized\n");
    
    return 0;
}

extern "C" int gui_add(Vu *vu) {
    int z = 40 * vu_map.size();
    glutInitWindowPosition(600 + z - (vu->W / 2), 100 + z);
    glutInitWindowSize(vu->W, vu->H);
    ///
    /// create window for img
    ///
    printf("\tWindow...");
    int id = glutCreateWindow(T4_APP_NAME); /// * create named window (as current)
    _vu_set(id, vu);
    ///
    /// * set callbacks (for current window, i.e. id)
    ///
    glutDisplayFunc(_display);
    glutKeyboardFunc(_keyboard);
    glutTimerFunc(REFRESH_DELAY, _refresh, 0);
    glutCloseFunc(_cleanup);
    printf("created\n");

    _bind_texture(vu);
    _compile_shader();                      /// load float shader
    
    return 0;
}

extern "C" int gui_loop() {
    glutMainLoop();
    return 0;
}

} // namespace T4GUI
