/*! @file
  @brief
  cueForth value definitions non-optimized
*/
#include <iostream>          // cin, cout
using namespace std;

#include "sstream.h"         // CUDA streams
#include "eforth.h"          // eForth core
#include "cueforth.h"        // wrapper

// forward declaration for implementation
extern "C" __KERN__ void mmu_init(void *ptr, U32 sz);

__GPU__ ForthVM *vm_pool[MIN_VM_COUNT];
__GPU__ Istream *istr;
__GPU__ Ostream *ostr;

__KERN__ void
eforth_init(U8 *ibuf, U8 *obuf) {
    if (threadIdx.x!=0 || blockIdx.x!=0) return;

    istr = new Istream((char*)ibuf);
    ostr = new Ostream((char*)obuf);

    for (int i=0; i<MIN_VM_COUNT; i++) {
        vm_pool[i] = new ForthVM(*istr, *ostr);          // instantiate new Forth VMs
        vm_pool[i]->init();                              // initialize dictionary
    }
    return;
}

__KERN__ void
eforth_exec() {
    if (threadIdx.x!=0) return;

    vm_pool[blockIdx.x]->outer();

    return;
}

CueForth::CueForth() {}
CueForth::~CueForth() {
    if (_obuf) _free(_obuf);
    if (_ibuf) _free(_ibuf);
    if (_heap) _free(_heap);
    hipDeviceReset();
}

__HOST__ void*
CueForth::_malloc(int sz, int type)
{
    void *mem;

    // TODO: to add texture memory
    switch (type) {
    case 0:     hipMalloc(&mem, sz); break;            // allocate device memory
    default:    hipMallocManaged(&mem, sz);            // managed (i.e. paged) memory
    }
    if (hipSuccess != hipGetLastError()) return NULL;

    return mem;
}

__HOST__ void
CueForth::_free(void *mem) {
    hipFree(mem);
}

__HOST__ int
CueForth::setup(int step, int trace) {
    hipDeviceReset();

    _heap = (U8*)_malloc(CUEF_HEAP_SIZE, 1);                // allocate main block (i.e. RAM)
    if (!_heap)  return -10;
    _ibuf = (U8*)_malloc(CUEF_IBUF_SIZE, 1);                // allocate main block (i.e. RAM)
    if (!_ibuf)  return -11;
    _obuf = (U8*)_malloc(CUEF_OBUF_SIZE, 1);                // allocate output buffer
    if (!_obuf)  return -12;

    //mmu_init<<<1,1>>>(mem, CUEF_HEAP_SIZE);               // setup memory management
    eforth_init<<<1,1>>>(_ibuf, _obuf);                     // setup basic classes  (TODO: => ROM)
    GPU_SYNC();

    U32 sz0, sz1;
    hipDeviceGetLimit((size_t *)&sz0, hipLimitStackSize);
    hipDeviceSetLimit(hipLimitStackSize, (size_t)sz0*4);
    hipDeviceGetLimit((size_t *)&sz1, hipLimitStackSize);

    return 0;
}

__HOST__ int
CueForth::run() {
	eforth_exec<<<1,1>>>();
    return 0;
}

__HOST__ void
CueForth::teardown(int sig) {}
///
/// main program
///
int main(int argc, char**argv) {
    CueForth *f = new CueForth();
    cout << CUEF_VERSION << " initializing..." << endl;
    f->setup();

    cout << CUEF_VERSION << " starting..." << endl;
    f->run();
    GPU_SYNC();

    cout << CUEF_VERSION << " done." << endl;
    f->teardown();
}
