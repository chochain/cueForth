/*! @file
  @brief
  cueForth value definitions non-optimized
*/
#include <iostream>          // cin, cout
#include <signal.h>
using namespace std;

#include "cuef_config.h"
#include "aio.h"             // CUDA async IO
#include "eforth.h"          // eForth core
#include "cueforth.h"        // wrapper

__GPU__    ForthVM *vm_pool[MIN_VM_COUNT];
///
/// instantiate VMs
/// TODO: use shared memory
///
__KERN__ void
cueforth_init(Istream *istr, Ostream *ostr, Dict *dict) {
    int b = blockIdx.x;
    if (threadIdx.x!=0) return;

    vm_pool[b] = new ForthVM(istr, ostr, dict);  // instantiate VM
    if (b==0) vm_pool[0]->init();                // initialize dictionary
}
///
/// check VM status
/// TODO: Dynamic Parallel
///
__KERN__ void
cueforth_busy(int *busy) {
    if (blockIdx.x != 0 || threadIdx.x != 0) return;

    *busy = 0;
    for (int i=0; i<MIN_VM_COUNT; i++) {
        if (vm_pool[i]->status == VM_RUN) {
            *busy = 1;
            break;
        }
    }
}
///
///
#include <stdio.h>
__KERN__ void
cueforth_exec() {
    const char *s[] = {"READY", "RUN", "WAITING", "STOPPED"};
    if (threadIdx.x!=0) return;

    ForthVM *vm = vm_pool[blockIdx.x];
    if (vm->status == VM_RUN) vm->outer();
    else                       printf("VM[%d] %s\n", blockIdx.x, s[vm->status]);
}

CueForth::CueForth(bool trace) {
    dict = new Dict();
    aio  = new AIO(dict, trace);            // TODO: aio not dict dependent
    hipMalloc((void**)&busy, sizeof(int));
    GPU_CHK();

    cueforth_init<<<MIN_VM_COUNT, 1>>>(aio->istream(), aio->ostream(), dict);
    GPU_CHK();

    //dict->dump(cout, 0, 120*0x10);        // dump memory from host
    //dict->words(cout);                    // dump dictionary from host
}
CueForth::~CueForth() {
    delete aio;
    hipFree(busy);
    hipDeviceReset();
}

__HOST__ int
CueForth::is_running() {
    int h_busy;
    //LOCK();                 // TODO: lock on vm_pool
    cueforth_busy<<<1, 1>>>(busy);
    GPU_SYNC();
    //UNLOCK();               // TODO:

    hipMemcpy(&h_busy, busy, sizeof(int), hipMemcpyDeviceToHost);

    return h_busy;
}

__HOST__ int
CueForth::run() {
    while (is_running()) {
        if (aio->readline()) {        // feed from host console to managed input buffer
            cueforth_exec<<<1,1>>>(); // TODO: multiple VM destination, shared memory
            GPU_CHK();
            aio->flush();             // flush output buffer
        }
        dict->dump(cout, 0, 0x30);
        yield();
    }
    return 0;
}

__HOST__ void
CueForth::teardown(int sig) {}
///
/// main program
///
void sigsegv_handler(int sig, siginfo_t *si, void *arg) {
    cout << "Exception caught at: " << si->si_addr << endl;
    exit(1);
}

void sigtrap() {
    struct sigaction sa;
    memset(&sa, 0, sizeof(struct sigaction));
    sigemptyset(&sa.sa_mask);
    sa.sa_sigaction = sigsegv_handler;
    sa.sa_flags     = SA_SIGINFO;
    sigaction(SIGSEGV, &sa, NULL);
}

int main(int argc, char**argv) {
    sigtrap();

    cout << CUEF_VERSION << " init" << endl;
    CueForth *f = new CueForth(CUEF_DEBUG);

    cout << CUEF_VERSION << " start" << endl;
    f->run();

    cout << CUEF_VERSION << " done." << endl;
    f->teardown();
}
