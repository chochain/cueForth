/*! @file
  @brief
  cueForth value definitions non-optimized
*/
#include <iostream>          // cin, cout
using namespace std;

#include "cuef_config.h"
#include "aio.h"             // CUDA async IO
#include "eforth.h"          // eForth core
#include "cueforth.h"        // wrapper

// forward declaration for implementation
extern "C" __KERN__ void mmu_init(void *ptr, U32 sz);

__GPU__ __managed__ ForthVM *vm_pool[MIN_VM_COUNT];

__KERN__ void
eforth_init(Istream *istr, Ostream *ostr) {
    if (threadIdx.x!=0 || blockIdx.x!=0) return;

    for (int i=0; i<MIN_VM_COUNT; i++) {
        vm_pool[i] = new ForthVM(istr, ostr);     // instantiate new Forth VMs
        vm_pool[i]->init();                       // initialize dictionary
    }
}

__KERN__ void
eforth_exec() {
    if (threadIdx.x!=0) return;

    ForthVM *vm = vm_pool[blockIdx.x];
    while (vm->status == VM_RUN) {
        vm->outer();
    }
}

CueForth::CueForth() {}
CueForth::~CueForth() {
    if (_obuf) _free(_obuf);
    if (_ibuf) _free(_ibuf);
    if (_heap) _free(_heap);
    hipDeviceReset();
}

__HOST__ void*
CueForth::_malloc(int sz, int type)
{
    void *mem;

    // TODO: to add texture memory
    switch (type) {
    case 0:     hipMalloc(&mem, sz); break;            // allocate device memory
    default:    hipMallocManaged(&mem, sz);            // managed (i.e. paged) memory
    }
    if (hipSuccess != hipGetLastError()) return NULL;

    return mem;
}

__HOST__ void
CueForth::_free(void *mem) {
    hipFree(mem);
}

__HOST__ int
CueForth::setup(int step, int trace) {
    hipDeviceReset();

    _heap = (U8*)_malloc(CUEF_HEAP_SIZE, 1);                // allocate main block (i.e. RAM)
    if (!_heap)  return -10;
    _ibuf = (U8*)_malloc(CUEF_IBUF_SIZE, 1);                // allocate main block (i.e. RAM)
    if (!_ibuf)  return -11;
    _obuf = (U8*)_malloc(CUEF_OBUF_SIZE, 1);                // allocate output buffer
    if (!_obuf)  return -12;

    aio = new AIO((char*)_ibuf, (char*)_obuf);

    //mmu_init<<<1,1>>>(mem, CUEF_HEAP_SIZE);               // setup memory management
    eforth_init<<<1,1>>>(aio->istream(), aio->ostream());
    GPU_SYNC();

    U32 sz0, sz1;
    hipDeviceGetLimit((size_t *)&sz0, hipLimitStackSize);
    hipDeviceSetLimit(hipLimitStackSize, (size_t)sz0*4);
    hipDeviceGetLimit((size_t *)&sz1, hipLimitStackSize);

    return 0;
}

__HOST__ int
CueForth::is_running() {
	int r = 0;
	//LOCK();                 // TODO: lock on vm_pool
	for (int i=0; i<MIN_VM_COUNT; i++) {
		if (vm_pool[i]->status != VM_STOP) r = 1;
	}
	//UNLOCK();               // TODO:
	return r;
}

__HOST__ int
CueForth::run() {
	while (is_running()) {
		cin.getline((char*)_ibuf, CUEF_IBUF_SIZE);
		if (*_ibuf) {
			eforth_exec<<<1,1>>>();
			GPU_SYNC();
			aio->flush();
		}
		yield();
	}
    return 0;
}

__HOST__ void
CueForth::teardown(int sig) {}
///
/// main program
///
int main(int argc, char**argv) {
    CueForth *f = new CueForth();
    cout << CUEF_VERSION << " initializing..." << endl;
    f->setup();

    cout << CUEF_VERSION << " starting..." << endl;
    f->run();

    cout << CUEF_VERSION << " done." << endl;
    f->teardown();
}
