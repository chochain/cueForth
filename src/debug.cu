#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @file
 * @brief System class - tensorForth Debug/Tracer implementation
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include <iomanip>
#include "debug.h"
///
///@name memory macros to reduce verbosity
///@{
#define MEM(a)   ((U8*)&mu->_pmem[a])         /** memory pointer by offset           */
#define DIDX     (mu->_didx)                  /** number of dictionary entries       */
#define DICT(w)  (mu->_dict[w])               /** dictionary entry                   */
#define XT0      ((UFP)DICT(0).xt)            /** base of lambda functions (i.e. xt) */
///@}
///@name Primitive words to help printing
///@{
Code prim[] = {
    Code(";",    EXIT),  Code("next ", NEXT), Code("loop ", LOOP), Code("lit",   LIT),
    Code("var",  VAR),   Code("str",   STR),  Code("dotq",  DOTQ), Code("bran ", BRAN),
    Code("0bran",ZBRAN), Code("for  ", FOR),  Code("do",    DO),   Code("key",   KEY)
};
///@}
///
/// AIO takes managed memory blocks as input and output buffers
/// which can be access by both device and host
///
__HOST__ void
Debug::ss_dump(IU id, int n, int base) {
    static char buf[34];                  ///< static buffer
    auto rdx = [](DU v, int b) {          ///< display v by radix
        DU t, f = modf(v, &t);            ///< integral, fraction
        if (ABS(f) > DU_EPS) {
            sprintf(buf, "%0.6g", v);
            return buf;
        }
        int i = 33;  buf[i]='\0';         /// * C++ can do only base=8,10,16
        int dec = b==10;
        U32 n   = dec ? (U32)(ABS(v)) : (U32)(v);  ///< handle negative
        do {                              ///> digit-by-digit
            U8 d = (U8)MOD(n,b);  n /= b;
            buf[--i] = d > 9 ? (d-10)+'a' : d+'0';
        } while (n && i);
        if (dec && v < DU0) buf[--i]='-';
        return &buf[i];
    };
    h_ostr &fout = io->fout;
    DU *ss = mu->vmss(id);                ///< retrieve VM SS
    for (int i=0; i < n; i++) {
        fout << rdx(*ss++, base) << ' ';
    }
    /// TODO << TOS
    fout << "-> ok" << std::endl;
}
__HOST__ int
Debug::p2didx(Param *p) {
    UFP xt0 = XT0;
    IU  pfa = p->ioff;
    for (int i = DIDX - 1; i > 0; --i) {
        Code &c  = DICT(i);
        bool hit = p->udf
            ? (c.udf  && pfa==c.pfa)
            : (!c.udf && pfa==(IU)((UFP)c.xt - xt0));
        if (hit) return i;
    }
    return -1;                                     /// * not found
}
__HOST__ int
Debug::to_s(IU w, int base) {
    Param *p = (Param*)MEM(DICT(w).pfa);
    return to_s(p, 0, base);
}
__HOST__ int
Debug::to_s(Param *p, int nv, int base) {
    bool pm = p->op != MAX_OP;                     ///< is prim
    int  w  = pm ? p->op : p2didx(p);              ///< fetch word index by pfa
    if (w < 0) return -1;                          ///> loop guard
    
    h_ostr &fout = io->fout;
    Code   &code = DICT(w);
    
    fout << std::endl << "  ";                     /// * indent
    if (io->trace) {                               /// * header
        fout << std::hex
             << std::setfill('0') << "( "
              << std::setw(4) << ((U8*)p - MEM(0)) ///> addr
             << std::setfill(' ') << '['
             << std::setw(4) << w << "] )"         ///> word ref
             << std::setbase(base);
    }
    if (!pm) {                                     ///> built-in
        char name[256];                            ///< name buffer on host
        d2h_strcpy(name, code.name);               /// * copy string from device
        fout << name << "  ";
        return 0;
    }
    U8 *ip = (U8*)(p+1);                           ///< pointer to data
    switch (w) {
    case LIT:  io->show(*(DU*)ip);                  break;
    case STR:  fout << "s\" " << (char*)ip << '"';  break;
    case DOTQ: fout << ".\" " << (char*)ip << '"';  break;
    case VAR:
        for (int i=0; i < nv; i+=sizeof(DU)) {
            fout << *(DU*)(ip + i) << ' ';
        }
        /* no break */
    default: fout << prim[w].name; break;
    }
    switch (w) {
    case NEXT: case LOOP:
    case BRAN: case ZBRAN:                   ///> display jmp target
        fout << " \\ $" << std::hex
             << std::setfill('0') << std::setw(4) << p->ioff;
        break;
    default: fout << std::setfill(' ') << std::setw(-1);          ///> restore format
    }
    return
        w==EXIT ||                           /// * end of word
        (w==LIT && p->exit) ||               /// * constant
        (w==VAR && !p->ioff);                /// * variable
}
///
/// display dictionary word (wastefully one byte at a time)
///
__HOST__ void
Debug::words(int base) {
    const int WIDTH = 60;
    h_ostr &fout = io->fout;
    fout << std::dec;
    char name[256];
    for (int i=1, sz=0; i < DIDX; i++) {
        d2h_strcpy(name, DICT(i).name);
        fout << "  " << name;
        sz += strlen(name) + 2;

        if (sz > WIDTH) {
            fout << ENDL; sz = 0;
        }
    }
    fout << std::setbase(base) << std::endl;
}
///
/// Forth pmem memory dump
/// TODO: dynamic parallel
///
#define C2H(c) { buf[x++] = i2h[(c)>>4]; buf[x++] = i2h[(c)&0xf]; }
#define IU2H(i){ C2H((i)>>8); C2H((i)&0xff); }
__HOST__ void
Debug::mem_dump(IU p0, int sz, int base) {
    const char i2h[] = "0123456789abcdef";
    h_ostr &fout = io->fout;
    char buf[80];
    fout << std::hex << std::setfill('0');
    for (IU i=ALIGN16(p0); i<=ALIGN16(p0+sz); i+=16) {
        int x = 0;
        buf[x++] = '\n'; IU2H(i); buf[x++] = ':'; buf[x++] = ' ';  // "%04x: "
        for (IU j=0; j<16; j++) {
            //U8 c = *(((U8*)&_dict[0])+i+j) & 0x7f;               // to dump _dict
            U8 c = *MEM(i+j);
            C2H(c);                                                // "%02x "
            c &= 0x7f;                                             // mask off high bit
            buf[x++] = ' ';
            if (j%4==3) buf[x++] = ' ';
            buf[59+j]= (c==0x7f||c<0x20) ? '.' : c;                // %c
        }
        buf[75] = '\0';
        fout << buf;
    }
    fout << std::setfill(' ') << std::setbase(base) << std::endl;
}

#define NFA(w) (DICT(w).pfa - ALIGN(strlen(DICT(w).name)))
__HOST__ void
Debug::see(IU w, int base) {
    h_ostr &fout = io->fout;
    Code   &c    = DICT(w);
    fout << ": " << c.name << ENDL;
    if (!c.udf) {
        fout << " ( built-ins ) ;" << std::endl;
        return;
    }
    auto nvar = [this](IU i0, IU ioff, U8 *ip) {       /// * calculate # of elements
        if (ioff) return MEM(ioff) - ip - sizeof(IU);  /// create...does>
        IU pfa0 = DICT(i0).pfa;
        IU nfa1 = (i0+1) < DIDX ? NFA(i0+1) : mu->_midx;
        return (nfa1 - pfa0 - sizeof(IU));             ///> variable, create ,
    };
    U8 *ip = MEM(c.pfa);                               ///< PFA pointer
    while (1) {
        Param *p = (Param*)ip;
        int   nv = p->op==VAR ? nvar(w, p->ioff, ip) : 0;  ///< VAR number of elements
        if (to_s(p, nv, base) != 0) break;                 ///< display Parameter
        ///
        /// advance ip to next Param
        ///
        ip += sizeof(IU);
        switch (p->op) {                     ///> extra bytes to skip
        case LIT: ip += sizeof(DU);             break;
        case VAR: ip = MEM(p->ioff);            break;  ///> create/does
        case STR: case DOTQ: ip += p->ioff;     break;
        }
    }
    fout << std::endl;
}
///====================================================================
///
///> System statistics - for heap, stack, external memory debugging
///
__HOST__ void
Debug::dict_dump(int base) {
    h_ostr &fout = io->fout;
    UFP xt0 = XT0;
    char name[256];
    fout << "Built-in Dictionary: _XT0="
         << std::hex << xt0 << std::setfill('0') << ENDL;
    for (int i=0; i < DIDX; i++) {
        Code &c = DICT(i);
        IU  ip = c.udf ? c.pfa : (IU)(((UFP)c.xt & MSK_XT) - xt0);
        d2h_strcpy(name, (char*)c.name);
        fout << std::dec << std::setw(4) << i << '|'
             << std::hex << std::setw(3) << i << " :"
             << std::setw(6) << ip
             << (c.udf ? 'u' : ' ')
			 << (c.imm ? '*' : ' ') << ' '
             << name << std::endl;
    }
    fout << std::setbase(base) << std::setfill(' ') << std::setw(-1);
}

__HOST__ void Debug::self_tests() {
//    dict_dump(10);
//    words();
//    mem_dump(0, 256, 10);
    ss_dump(0, 3, 10);
}
