#include "hip/hip_runtime.h"
/** -*- c++ -*- 
 * @File
 * @brief - tensorForth GUI - Image Viewer base class
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "vu.h"

__HOST__
Vu::Vu(Ndata &nd, int x, int y) :
    ndata(nd), X(x ? x : ndata.W), Y(y ? y : ndata.H) {
    if (sizeof(uchar4) != 4) {
        fprintf(stderr, "ERR: Bad uchar4 size = %ld\n", sizeof(uchar4));
        exit(-1);
    }
    if (X == ndata.W && Y == ndata.H && ndata.C == 4) {
        h_tex = (uchar4*)ndata.data;       /// * pass thru, no buffer needed
        return;
    }
    ///
    /// malloc GL texture block (in uchar4 format)
    ///
    size_t bsz = X * Y * sizeof(uchar4);   ///< texture block size

    h_tex = (uchar4*)malloc(bsz);          ///< alloc texture block
    if (!h_tex) {
        fprintf(stderr, "Vu.h_tex malloc %ld bytes failed\n", bsz);
        exit(-1);
    }
}

__HOST__ int
Vu::init_host_tex() {
    int    C  = ndata.C;
    U8     *s = ndata.data;
    uchar4 *t = h_tex;
    for (int i = 0; i < Y; i++) {
        for (int j = 0; j < X; j++, t++, s+=C) {
            t->x = *s;
            t->y = C < 2 ? *s   : *(s+1);
            t->z = C < 3 ? *s   : *(s+2);
            t->w = C < 4 ? 0xff : *(s+3);
        }
    }
    return 0;                                ///< has malloc
}

__HOST__ void
Vu::tex_dump() {
#if CC_DEBUG
    if (!h_tex) return;
    uchar4 *p = h_tex;
    for (int i = 0; i < 10; i++) {
        printf("\n");
        for (int j = 0; j < 4; j++, p++) {
            printf("[%02x,%02x,%02x,%02x] ", p->x, p->y, p->z, p->w);
        }
    }
    printf("\n");
#endif // CC_DEBUG
}

__HOST__ void
Vu::init_cuda_tex() {
    int pitch = sizeof(uchar4) * X;
    
    hipChannelFormatDesc fmt = hipCreateChannelDesc<uchar4>();
    CUX(hipMallocArray(&d_ary, &fmt, X, Y));
    CUX(hipMemcpy2DToArray(
        d_ary, 0, 0, h_tex, pitch, pitch, Y, hipMemcpyHostToDevice));

    hipResourceDesc res;
    memset(&res, 0, sizeof(hipResourceDesc));
    res.resType           = hipResourceTypeArray;
    res.res.array.array   = d_ary;

    hipTextureDesc desc;
    memset(&desc, 0, sizeof(hipTextureDesc));
    desc.normalizedCoords = false;
    desc.filterMode       = hipFilterModeLinear;
    desc.addressMode[0]   = hipAddressModeWrap;
    desc.addressMode[1]   = hipAddressModeWrap;
    desc.readMode         = hipReadModeNormalizedFloat;
  
    CUX(hipCreateTextureObject(&cu_tex, &res, &desc, NULL));
}

__HOST__ void
Vu::free_tex() {
    if (!h_tex) return;
    
    CUX(hipDestroyTextureObject(cu_tex));   /// * release texture object
    CUX(hipFreeArray(d_ary));               /// * free device texture memory

    // free(h_tex);  /// * free host texture memory, TODO: => core dump?
}



