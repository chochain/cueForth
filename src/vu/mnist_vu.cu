#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - tensorForth MNIST Dataset Vu (Texture builder)
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include "mnist_vu.h"
///
/// MNIST GUI texture builder
///
__HOST__ int
MnistVu::init_host_tex() {
    auto fit = [this](int z0, unsigned char *src) {
        uchar4 *t = &h_tex[z0];
        for (int i = dset.H - 1; i >= 0; i--) {      // y top-down flip
            for (int j = 0; j < dset.W; j++, src++) {
                int z = j + i * dset.W * NX;
                t[z].x = t[z].y = t[z].z = *src;
                t[z].w = 0xff;
            }
        }
    };
    for (int y = 0, y1 = NY-1; y < NY; y++, y1--) { // y1 top-down flip
        printf("\n");
        for (int x = 0; x < NX; x++) {
            int z   = x + y * NX;
            U8 *src = dset[z];
            int z0  = (x + y1 * dset.H * NX) * dset.W;
            fit(z0, src);
            printf("%1d", dset.label[z]);
        }
    }
    printf("\n");
    return 0;
}



