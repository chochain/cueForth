/** -*- c++ -*-
 * @file
 * @brief - GEMM tester
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include <iostream>          // cin, cout
#include <signal.h>

//===========================================================================================
//
// CUTLASS includes needed for single-precision GEMM kernel
//
// Defines cutlass::gemm::device::Gemm, the generic Gemm computation template class.
#include "cutlass/gemm/device/gemm.h"
#include "../src/ten4_types.h"
#include "../src/tensor.h"
#include "../src/opt.h"

typedef DU FP;
typedef hipError_t (*gemm_op)(Tensor &A, Tensor &B, Tensor &C, FP alpha, FP beta);

void benchmark(gemm_op op, Tensor &A, Tensor &B, Tensor &C, FP alpha, FP beta) {
    hipEvent_t events[2];
    float       runtime_ms;
    for (auto & event : events) {
        hipEventCreate(&event);
        GPU_CHK();
    }
    hipEventRecord(events[0]);

    op(A, B, C, alpha, beta);
    GPU_CHK();

    // Wait for work on the device to complete.
    hipEventRecord(events[1]);
    hipEventSynchronize(events[1]);
    // Measure eresultd runtime
    hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    printf(" => %f ms\n", runtime_ms);

    // Cleanup
    for (auto event : events) {
        (void)hipEventDestroy(event);
    }
}

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassSgemmNN(Tensor &A, Tensor &B, Tensor &C, FP alpha, FP beta) {
    int M = C.H(), N = C.W(), K = A.W();
    printf("Cutlass.GEMM M=%d, N=%d, K=%d ", M, N, K);
    using LO   = cutlass::layout::ColumnMajor;
    using Gemm = cutlass::gemm::device::Gemm<FP, LO, FP, LO, FP, LO>;
    Gemm::Arguments args(
        {M, N, K},               // Gemm Problem dimensions
        {(FP const*)A.data, M},  // Tensor-ref for source matrix A
        {(FP const*)B.data, K},  // Tensor-ref for source matrix B
        {(FP*)C.data, M},        // Tensor-ref for source matrix C
        {(FP*)C.data, M},        // Tensor-ref for destination matrix D (may be different memory than source C matrix)
        {alpha, beta}            // Scalars used in the Epilogue
    );
    // Define a CUTLASS GEMM type
    Gemm cutlass_gemm;
    cutlass::Status status = cutlass_gemm(args);
    //
    // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
    //
    return (status==cutlass::Status::kSuccess) ? hipSuccess : hipErrorUnknown;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// The source code after this point in the file is generic CUDA using the CUDA Runtime API
// and simple CUDA kernels to initialize matrices and compute the general matrix product.
//
///////////////////////////////////////////////////////////////////////////////////////////////////
/// Naive reference GEMM computation.
__KERN__ void ReferenceGemm_kernel(
    int M, int N, int K,
    FP *A, FP *B, FP *C,   /* MxK, KxN, MxN */
    FP  alpha, FP beta)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < N && j < M) {
        FP acc = 0;
        for (int k = 0; k < K; ++k) {
            acc += A[k + j * K] * B[i + k * N];      /* row major */
        }
        C[i + j * N] = alpha * acc + beta * C[i + j * N];
    }
}
///////////////////////////////////////////////////////////////////////////////////////////////////
/// Reference GEMM computation.
hipError_t ReferenceGemm(Tensor &A, Tensor &B, Tensor &C, FP alpha, FP beta) {
    int M = A.H(), N = B.W(), K = A.W();
    printf("Ref.GEMM M=%d, N=%d, K=%d", M, N, K);
    dim3 block(16, 16);   /* 256 threads */
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);

    ReferenceGemm_kernel<<<grid, block>>>(M, N, K, (FP*)A.data, (FP*)B.data, (FP*)C.data, alpha, beta);

    return hipGetLastError();
}

///////////////////////////////////////////////////////////////////////////////////////////////////
/// Allocate several matrices in GPU device memory and call a single-precision
/// CUTLASS GEMM kernel.
hipError_t TestCutlassGemm(int M, int N, int K, FP alpha, FP beta) {
    // Define pointers to matrices in GPU device memory.
    //
    // matrix allocation, fill, random
    //
    Tensor tensor_A(M, K); tensor_A.fill(0.0).random(0);
    Tensor tensor_B(K, N); tensor_B.fill(0.0).random(17);
    //
    // reshape test
    //
    U32 sz = M * N;
    Tensor tensor_C(sz); tensor_C.reshape(M, N).fill(0.0).random(101);
    //
    // reset test
    //
    U8  *ref;
    hipMallocManaged((void**)&ref, (size_t)sz * sizeof(DU));
    GPU_CHK();
    Tensor tensor_R; tensor_R.reset(ref, sz).reshape(M, N).fill(0).random(101);
    //=============================================================================
    // Launch CUTLASS GEMM.
    //
    benchmark(CutlassSgemmNN, tensor_A, tensor_B, tensor_C, alpha, beta);
    //
    // Lanch reference GEMM
    //
    benchmark(ReferenceGemm, tensor_A, tensor_B, tensor_R, alpha, beta);
    //
    // Verify: copy to host and verify equivalence.
    //
    std::vector<FP> host_c(tensor_C.size, 0);
    std::vector<FP> host_r(tensor_R.size, 0);

    tensor_C.copy_to_host(host_c.data());
    tensor_R.copy_to_host(host_r.data());
    //
    // Test for bit equivalence of results.
    //
    if (host_c != host_r) {
        std::cerr << "results different." << std::endl;
        return hipErrorUnknown;
    }
    hipFree(ref);
    return hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// usage:  t_tensor <M> <N> <K> <alpha> <beta>
//
#define CUTLASS_OPTS 1

int main(int argc, const char **argv) {
    //
    // Parse the command line to obtain GEMM dimensions and scalar values.
    //
#if CUTLASS_OPTS
    Options opt;
    opt.parse(argc, argv);
    
    if (opt.help) {
        opt.check_devices(std::cout);
        opt.print_usage(std::cout);
        return 0;
    }
    hipError_t result = TestCutlassGemm(
        opt.problem_size.m(),     // GEMM M dimension
        opt.problem_size.n(),     // GEMM N dimension
        opt.problem_size.k(),     // GEMM K dimension
        opt.alpha.real(),         // alpha
        opt.beta.real()           // beta
        );
#else // CUTLASS_OPTS
    // GEMM problem dimensions.
    int problem[3] = { 1024, 512, 2048 };

    for (int i = 1; i < argc && i < 4; ++i) {
        std::stringstream ss(arg[i]);
        ss >> problem[i - 1];
    }
    // Scalars used for linear scaling the result of the matrix product.
    float scalars[2] = { 1, 0 };
    for (int i = 4; i < argc && i < 6; ++i) {
        std::stringstream ss(arg[i]);
        ss >> scalars[i - 4];
    }
    //
    // Run the CUTLASS GEMM test.
    //
    hipError_t result = TestCutlassGemm(
        problem[0],     // GEMM M dimension
        problem[1],     // GEMM N dimension
        problem[2],     // GEMM K dimension
        scalars[0],     // alpha
        scalars[1]      // beta
        );
#endif // CUTLASS_OPTS
    if (result == hipSuccess) {
        std::cout << "Passed." << std::endl;
    }
    // Exit.
    return result == hipSuccess ? 0 : -1;
}
