#include "hip/hip_runtime.h"
/** -*- c++ -*-
 * @File
 * @brief - tensorForth Image Loader with STB
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>
#include "t_imgvu.h"

ImgLoader &ImgLoader::load() {
    printf("Loading %s", d_fn);
    
    stbi_set_flip_vertically_on_load(true);
    h_data = stbi_load(d_fn, &W, &H, &C, STBI_rgb); 
//    h_data = stbi_load(d_fn, &W, &H, &C, STBI_rgb_alpha);
//    C = 4;           // plus alpha
    if (!h_data) {
        printf(" => failed\n");
        exit(-1);
    }
    printf(" => [%d,%d,%d] loaded\n", H, W, C);
    
    return *this;
}

__GPU__ __INLINE__ TColor tex2color(float r, float g, float b, float a) {
    return
        ((int)(a * 255.0f) << 24) |
        ((int)(b * 255.0f) << 16) |
        ((int)(g * 255.0f) << 8)  |
        ((int)(r * 255.0f) << 0);
}
__KERN__ void k_img_copy(TColor *dst, int W, int H, CuTexObj tex, bool flip) {
    const int j = threadIdx.x + blockDim.x * blockIdx.x;
    const int i = threadIdx.y + blockDim.y * blockIdx.y;
    // Add half of a texel to always address exact texel centers
    const float x = flip ? (float)(W - j) - 0.5f : (float)j + 0.5f;
    const float y = (float)i + 0.5f;

    if (j < W && i < H) {
        float4 v = tex2D<float4>(tex, x, y);
        dst[j + i * W] = tex2color(v.x, v.y, v.z, 0);
    }
}

ImgVu::ImgVu(Dataset &ds) : Vu(ds) {
    uchar4 *p = h_tex;
    for (int i = 0; i < 10; i++) {
        printf("\n");
        for (int j = 0; j < 4; j++, p++) {
            printf("[%02x,%02x,%02x,%02x] ", p->x, p->y, p->z, p->w);
        }
    }
    printf("\n");
}

void ImgVu::_img_copy(TColor *d_dst) {
    dim3 blk(T4_WARP_SZ, T4_WARP_SZ, 1);
    dim3 grd(TGRID(X, Y, 1, blk));

    k_img_copy<<<grd,blk>>>(d_dst, X, Y, cu_tex, false);
    GPU_CHK();
}
void ImgVu::_img_flip(TColor *d_dst) {
    dim3 blk(T4_WARP_SZ, T4_WARP_SZ, 1);
    dim3 grd(TGRID(X, Y, 1, blk));

    k_img_copy<<<grd,blk>>>(d_dst, X, Y, cu_tex, true);
    GPU_CHK();
}

static const char *list_keys =
    "\nStarting GLUT main loop...\n"
    "Press [0] to view flipped image\n"
    "Press [1] to view original image\n"
    "Press [q] to exit\n";

int main(int argc, char **argv) {
#if defined(__linux__)
    setenv("DISPLAY", ":0", 0);
#endif
    printf("Starting %s...\n\n", argv[0]);

    hipSetDevice(0);

    if (gui_init(&argc, argv)) return -1;

    ImgLoader &ldr = (*new ImgLoader("./data/cat_n_dog.jpg")).load();
    ImgVu &vu0 = *new ImgVu(ldr);
    ImgVu &vu1 = *new ImgVu(ldr);
    gui_add(vu0);
    gui_add(vu1);
    
    printf("%s", list_keys);
    
    return gui_loop();
}
