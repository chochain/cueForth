#include "hip/hip_runtime.h"
/*!
 * @file - ten4.cu
 * @brief - tensorForth mmu#tensor tests
 *
 * <pre>Copyright (C) 2022- GreenII, this file is distributed under BSD 3-Clause License.</pre>
 */
#include <iostream>          // cin, cout
#include <signal.h>
using namespace std;

#include "../src/ten4_config.h"
#include "../src/ten4_types.h"
#include "../src/mmu.h"

__KERN__ void k_GEMM(
    int M, int N, int K,
    DU *A, DU *B, DU *C,   /* MxK, KxN, MxN */
    DU alpha, DU beta)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < N && j < M) {
        DU acc = 0;
        for (int k = 0; k < K; ++k) {
            acc += A[k + j * K] * B[i + k * N];      /* row major */
        }
        C[i + j * N] = alpha * acc + beta * C[i + j * N];
    }
}
//
// dynamic parallel
//
__KERN__ void
test_mmu_gemm(
    MMU *mmu, int khz,
    U16 M, U16 N, U16 K, DU alpha, DU beta) {
    if (threadIdx.x != 0 || blockIdx.x != 0) return;

    Tensor &A = mmu->tensor(M, K); A.fill(0).random(0);
    Tensor &B = mmu->tensor(K, N); B.fill(0).random(17);
    Tensor &C = mmu->tensor(M, N); C.fill(0).random(101);

    int m = C.H(), n = C.W(), k = A.W();
    printf("\nRef.GEMM M=%d, N=%d, K=%d", m, n, k);
    
    dim3 block(16, 16);         /* 256 threads */
    dim3 grid((n + block.x - 1) / block.x, (m + block.y - 1) / block.y);

    clock_t start = clock();
    k_GEMM<<<grid, block>>>(
        m, n, k,
        (DU*)A.data, (DU*)B.data, (DU*)C.data,
        alpha, beta);
    hipDeviceSynchronize();
    printf(" (k_GEMM in %0.2f ms @ %d khz)", ((float)(clock() - start))/khz, khz);
}

void benchmark(MMU *mmu, int khz) {
    hipEvent_t events[2];
    float       runtime_ms;
    for (auto & event : events) {
        hipEventCreate(&event);
        GPU_CHK();
    }
    hipEventRecord(events[0]);
    
    test_mmu_gemm<<<1,1>>>(mmu, khz, 1024, 512, 2048, 1.5, 1.5);
    GPU_CHK();

    // Wait for work on the device to complete.
    hipEventRecord(events[1]);
    hipEventSynchronize(events[1]);
    // Measure eresultd runtime
    hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    printf(" => %0.2f ms\n", runtime_ms);

    // Cleanup
    for (auto event : events) {
        (void)hipEventDestroy(event);
    }
}

int main(int argc, char**argv) {
    int device = 0;
    int khz    = 0;
    hipDeviceGetAttribute(&khz, hipDeviceAttributeClockRate, device);
    GPU_CHK();

    MMU *mmu   = new MMU();
    benchmark(mmu, khz);

    return 0;
}

    
